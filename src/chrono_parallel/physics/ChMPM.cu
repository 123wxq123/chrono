#include "hip/hip_runtime.h"
#include "chrono_parallel/physics/ChMPM.h"
#include "chrono_parallel/physics/MPMUtils.h"
using namespace chrono;

struct Bounds {
    real minimum[3];
    real maximum[3];
};

struct Settings {
    real dt, radius, inv_radius, bin_edge;
    real inv_bin_edge, max_vel, mu, lambda;
    real hardening_coefficient, theta_c, theta_s, alpha;
    real youngs_modulus, poissons_ratio;
    int num_particles;
    int num_nodes;
    real mass;
    real p1, p2, p3;
    int num_iterations;
    int bins_per_axis_x;
    int bins_per_axis_y;
    int bins_per_axis_z;
};

CUDA_CONSTANT Settings system_settings;
CUDA_CONSTANT Bounds system_bounds;

/////// BB Constants
__device__ real alpha = 0.0001;
__device__ real dot_ms_ms = 0;
__device__ real dot_ms_my = 0;
__device__ real dot_my_my = 0;
__device__ real gdiff = 1;

CUDA_CONSTANT real a_min = 1e-13;
CUDA_CONSTANT real a_max = 1e13;
CUDA_CONSTANT real neg_BB1_fallback = 0.11;
CUDA_CONSTANT real neg_BB2_fallback = 0.12;

#define LOOP_TWO_RING_GPU(X)                                                                         \
    const real bin_edge = system_settings.bin_edge;                                                  \
    const real inv_bin_edge = 1.f / bin_edge;                                                        \
                                                                                                     \
    const int cx = GridCoord(xi.x, inv_bin_edge, system_bounds.minimum[0]);                          \
    const int cy = GridCoord(xi.y, inv_bin_edge, system_bounds.minimum[1]);                          \
    const int cz = GridCoord(xi.z, inv_bin_edge, system_bounds.minimum[2]);                          \
                                                                                                     \
    for (int k = cz - 2; k <= cz + 2; ++k) {                                                         \
        for (int j = cy - 2; j <= cy + 2; ++j) {                                                     \
            for (int i = cx - 2; i <= cx + 2; ++i) {                                                 \
                vec3 bins_per_axis(system_settings.bins_per_axis_x, system_settings.bins_per_axis_y, \
                                   system_settings.bins_per_axis_z);                                 \
                const int current_node = GridHash(i, j, k, bins_per_axis);                           \
                real3 current_node_location;                                                         \
                current_node_location.x = i * bin_edge + system_bounds.minimum[0];                   \
                current_node_location.y = j * bin_edge + system_bounds.minimum[1];                   \
                current_node_location.z = k * bin_edge + system_bounds.minimum[2];                   \
                X                                                                                    \
            }                                                                                        \
        }                                                                                            \
    }

//////========================================================================================================================================================================
////
CUDA_GLOBAL void kComputeBounds(const real3* pos,  // input
                                real3* lower,      // output
                                real3* upper       // output
                                ) {
    typedef hipcub::BlockReduce<real3, num_threads_per_block> BlockReduce;

    __shared__ typename BlockReduce::TempStorage temp_storage;

    const int block_start = blockDim.x * blockIdx.x;
    const int num_valid = min(system_settings.num_particles - block_start, blockDim.x);

    const int index = block_start + threadIdx.x;

    if (index < system_settings.num_particles) {
        real3 data = pos[index];

        real3 blockUpper = BlockReduce(temp_storage).Reduce(data, real3Max(), num_valid);

        // sync threads because second reduce uses same temp storage as first
        __syncthreads();

        real3 blockLower = BlockReduce(temp_storage).Reduce(data, real3Min(), num_valid);

        if (threadIdx.x == 0) {
            // write out block results, expanded by the radius
            AtomicMax(upper, blockUpper);
            AtomicMin(lower, blockLower);
        }
    }
}
////========================================================================================================================================================================
CUDA_GLOBAL void kRasterize(const real3* sorted_pos,  // input
                            const real3* sorted_vel,  // input
                            real* grid_mass,          // output
                            real* grid_vel) {         // output
    const int p = blockIdx.x * blockDim.x + threadIdx.x;
    if (p < system_settings.num_particles) {
        const real3 xi = sorted_pos[p];
        const real3 vi = sorted_vel[p];
        LOOP_TWO_RING_GPU(                                                                     //
            real weight = N(xi - current_node_location, inv_bin_edge) * system_settings.mass;  //
            atomicAdd(&grid_mass[current_node], weight);                                       //
            AtomicAdd(&((real3*)grid_vel)[current_node], weight * real3(vi));                  //
            )
    }
}
CUDA_GLOBAL void kRasterize(const real3* sorted_pos,  // input
                            real* grid_mass) {        // output
    const int p = blockIdx.x * blockDim.x + threadIdx.x;
    if (p < system_settings.num_particles) {
        const real3 xi = sorted_pos[p];
        LOOP_TWO_RING_GPU(                                                                     //
            real weight = N(xi - current_node_location, inv_bin_edge) * system_settings.mass;  //
            atomicAdd(&grid_mass[current_node], weight);                                       //
            )
    }
}
//
////========================================================================================================================================================================
//
CUDA_GLOBAL void kNormalizeWeights(real* grid_mass,   // input
                                   real* grid_vel) {  // output
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < system_settings.num_nodes) {
        real n_mass = grid_mass[i];
        if (n_mass > C_EPSILON) {
            grid_vel[i * 3 + 0] /= n_mass;
            grid_vel[i * 3 + 1] /= n_mass;
            grid_vel[i * 3 + 2] /= n_mass;
        }
    }
}
//////========================================================================================================================================================================
////
CUDA_GLOBAL void kComputeParticleVolumes(const real3* sorted_pos,  // input
                                         real* grid_mass,          // output
                                         real* volume) {
    const int p = blockIdx.x * blockDim.x + threadIdx.x;
    if (p < system_settings.num_particles) {
        const real3 xi = sorted_pos[p];
        real particle_density = 0;
        LOOP_TWO_RING_GPU(                                              //
            real weight = N(xi - current_node_location, inv_bin_edge);  //
            particle_density += grid_mass[current_node] * weight;       //
            atomicAdd(&grid_mass[current_node], weight);                //
            )
        particle_density /= bin_edge * bin_edge * bin_edge;
        volume[p] = system_settings.mass / particle_density;
    }
}

CUDA_GLOBAL void kRhs(const real3* sorted_pos,     // input
                      const Mat33* marker_Fe_hat,  // input
                      const Mat33* marker_Fe,      // input
                      const Mat33* marker_Fp,      // input
                      const real* marker_volume,   // input
                      real* rhs,                   // output
                      real* volume) {
    const int p = blockIdx.x * blockDim.x + threadIdx.x;
    if (p < system_settings.num_particles) {
        const real3 xi = sorted_pos[p];

        Mat33 PED =
            Potential_Energy_Derivative_Deviatoric(marker_Fe_hat[p], marker_Fp[p], system_settings.mu,
                                                   system_settings.lambda, system_settings.hardening_coefficient);
        Mat33 vPEDFepT = marker_volume[p] * MultTranspose(PED, marker_Fe[p]);
        real JE = Determinant(marker_Fe[p]);  //
        real JP = Determinant(marker_Fp[p]);

        LOOP_TWO_RING_GPU(                                                  //
            real3 d_weight = dN(xi - current_node_location, inv_bin_edge);  //
            real3 force = system_settings.dt * (vPEDFepT * d_weight) / (JE * JP);

            rhs[current_node * 3 + 0] -= force.x;  //
            rhs[current_node * 3 + 1] -= force.y;  //
            rhs[current_node * 3 + 2] -= force.z;  //
            )
    }
}

CUDA_GLOBAL void kMultiplyA(const real3* sorted_pos,  // input
                            const real* v_array,
                            const real* old_vel_node_mpm,
                            const Mat33* marker_Fe_hat,  // input
                            const Mat33* marker_Fe,      // input
                            const Mat33* marker_Fp,      // input
                            const real* marker_volume,   // input
                            real* result_array) {
    const int p = blockIdx.x * blockDim.x + threadIdx.x;
    if (p < system_settings.num_particles) {
        const real3 xi = sorted_pos[p];
        Mat33 delta_F(0);
        LOOP_TWO_RING_GPU(  //
            real3 vnew(v_array[current_node * 3 + 0], v_array[current_node * 3 + 1], v_array[current_node * 3 + 2]);
            real3 vold(old_vel_node_mpm[current_node * 3 + 0], old_vel_node_mpm[current_node * 3 + 1],
                       old_vel_node_mpm[current_node * 3 + 2]);
            real3 v0 = vold + vnew;                                   //
            real3 v1 = dN(xi - current_node_location, inv_bin_edge);  //
            delta_F += OuterProduct(v0, v1);                          //
            )

        real plastic_determinant = Determinant(marker_Fp[p]);
        real J = Determinant(marker_Fe_hat[p]);
        real current_mu = system_settings.mu * Exp(system_settings.hardening_coefficient * (1.0 - plastic_determinant));
        real current_lambda =
            system_settings.lambda * Exp(system_settings.hardening_coefficient * (1.0 - plastic_determinant));
        Mat33 Fe_hat_inv_transpose = InverseTranspose(marker_Fe_hat[p]);

        real dJ = J * InnerProduct(Fe_hat_inv_transpose, delta_F);
        Mat33 dF_inverse_transposed = -Fe_hat_inv_transpose * Transpose(delta_F) * Fe_hat_inv_transpose;
        Mat33 dJF_inverse_transposed = dJ * Fe_hat_inv_transpose + J * dF_inverse_transposed;
        Mat33 RD = Rotational_Derivative(marker_Fe_hat[p], delta_F);

        Mat33 volume_Ap_Fe_transpose =
            marker_volume[p] * (2 * current_mu * (delta_F - RD) + (current_lambda * J * dJ) * Fe_hat_inv_transpose +
                                (current_lambda * (J - 1.0)) * dJF_inverse_transposed) *
            Transpose(marker_Fe[p]);
        {
            LOOP_TWO_RING_GPU(  //
                real3 res = volume_Ap_Fe_transpose * dN(xi - current_node_location, inv_bin_edge);
                atomicAdd(&result_array[current_node * 3 + 0], res.x);
                atomicAdd(&result_array[current_node * 3 + 1], res.y);
                atomicAdd(&result_array[current_node * 3 + 2], res.z););
        }
    }
}
CUDA_GLOBAL void kMultiplyB(const real* v_array,
                            const real* old_vel_node_mpm,
                            const real* node_mass,
                            const real* offset_array,
                            real* result_array) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < system_settings.num_nodes) {
        real mass = node_mass[i];
        if (mass > C_EPSILON) {
            result_array[i * 3 + 0] =
                (mass * (v_array[i * 3 + 0] + old_vel_node_mpm[i * 3 + 0]) + result_array[i * 3 + 0]) -
                offset_array[i * 3 + 0];
            result_array[i * 3 + 1] =
                (mass * (v_array[i * 3 + 1] + old_vel_node_mpm[i * 3 + 1]) + result_array[i * 3 + 1]) -
                offset_array[i * 3 + 1];
            result_array[i * 3 + 2] =
                (mass * (v_array[i * 3 + 2] + old_vel_node_mpm[i * 3 + 2]) + result_array[i * 3 + 2]) -
                offset_array[i * 3 + 2];
        }
    }
}

void ChMPM::Bounds(const real kernel_radius, std::vector<real3>& positions) {
    num_mpm_markers = positions.size();
    max_bounding_point = real3(-FLT_MAX);
    min_bounding_point = real3(FLT_MAX);

    hipMemcpyAsync(lower_bound, &min_bounding_point, sizeof(real3), hipMemcpyHostToDevice);
    hipMemcpyAsync(upper_bound, &max_bounding_point, sizeof(real3), hipMemcpyHostToDevice);

    kComputeBounds<<<CONFIG(num_mpm_markers)>>>(pos.data_d,    //
                                                lower_bound,   //
                                                upper_bound);  //

    hipMemcpy(&min_bounding_point, lower_bound, sizeof(real3), hipMemcpyDeviceToHost);
    hipMemcpy(&max_bounding_point, upper_bound, sizeof(real3), hipMemcpyDeviceToHost);

    max_bounding_point = max_bounding_point + kernel_radius * 8;
    min_bounding_point = min_bounding_point - kernel_radius * 6;

    hipMemcpyToSymbolAsync(HIP_SYMBOL(system_bounds), &min_bounding_point, sizeof(real3), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbolAsync(HIP_SYMBOL(system_bounds), &max_bounding_point, sizeof(real3), sizeof(real3), hipMemcpyHostToDevice);

    bin_edge = kernel_radius * 2;
    real3 bpa = real3(max_bounding_point - min_bounding_point) / bin_edge;
    bins_per_axis.x = bpa.x;
    bins_per_axis.y = bpa.y;
    bins_per_axis.z = bpa.z;
    inv_bin_edge = real(1.) / bin_edge;
    num_mpm_nodes = bins_per_axis.x * bins_per_axis.y * bins_per_axis.z;

    printf("max_bounding_point [%f %f %f]\n", max_bounding_point.x, max_bounding_point.y, max_bounding_point.z);
    printf("min_bounding_point [%f %f %f]\n", min_bounding_point.x, min_bounding_point.y, min_bounding_point.z);
    printf("Compute DOF [%d %d %d] [%f] %d %d\n", bins_per_axis.x, bins_per_axis.y, bins_per_axis.z, bin_edge,
           num_mpm_nodes, num_mpm_markers);
}
//
void ChMPM::Initialize(const real marker_mass, const real radius, std::vector<real3>& positions) {
    num_mpm_markers = positions.size();
    mass = marker_mass;
    kernel_radius = radius;

    cudaCheck(hipMalloc(&lower_bound, sizeof(real3)));
    cudaCheck(hipMalloc(&upper_bound, sizeof(real3)));

    pos.data_h = positions;
    pos.copyHostToDevice();
    Bounds(kernel_radius, positions);
    marker_volume.resize(num_mpm_markers);
    node_mass.resize(num_mpm_nodes);
    node_mass.set(0);

    kRasterize<<<CONFIG(num_mpm_markers)>>>(pos.data_d,         // input
                                            node_mass.data_d);  // output

    kComputeParticleVolumes<<<CONFIG(num_mpm_markers)>>>(pos.data_d,        // input
                                                         node_mass.data_d,  // output
                                                         marker_volume.data_d);
}
void ChMPM::Multiply(gpu_vector<real>& input, gpu_vector<real>& output, gpu_vector<real>& r) {
    //    int size = input.size();
    //    kMultiplyA<<<CONFIG(size)>>>(sorted_pos,  // input
    //                                 input.data_d, old_vel_node_mpm.data_d,
    //                                 marker_Fe_hat.data_d,  // input
    //                                 marker_Fe.data_d,      // input
    //                                 marker_Fp.data_d,      // input
    //                                 marker_volume.data_d,  // input
    //                                 output.data_d);
    //
    //    kMultiplyB<<<CONFIG(size)>>>(input, old_vel_node_mpm.data_d, node_mass.data_d, r.data_d, result_array.data_d);
}
template <bool inner>
CUDA_GLOBAL void kResetGlobals(int size) {
    if (inner) {
        dot_ms_ms = 0;
        dot_ms_my = 0;
        dot_my_my = 0;
    } else {
        alpha = 0.0001;
        gdiff = 1.0 / pow(size, 2.0);
    }
}

template <bool even>
CUDA_GLOBAL void kUpdateAlpha(int num_items, real* ml_p, real* ml, real* mg_p, real* mg) {
    typedef hipcub::BlockReduce<real, num_threads_per_block> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    const int block_start = blockDim.x * blockIdx.x;
    const int num_valid = min(num_items - block_start, blockDim.x);

    const int tid = block_start + threadIdx.x;
    if (tid < num_items) {
        real data, block_sum;
        real ms = ml_p[tid] - ml[tid];
        real my = mg_p[tid] - mg[tid];

        if (even) {
            data = ms * ms;
            block_sum = BlockReduce(temp_storage).Reduce(data, hipcub::Sum(), num_valid);
            if (threadIdx.x == 0) {
                atomicAdd(&dot_ms_ms, block_sum);
            }
        } else {
            data = my * my;
            block_sum = BlockReduce(temp_storage).Reduce(data, hipcub::Sum(), num_valid);
            if (threadIdx.x == 0) {
                atomicAdd(&dot_my_my, block_sum);
            }
        }
        __syncthreads();
        data = ms * my;
        block_sum = BlockReduce(temp_storage).Reduce(data, hipcub::Sum(), num_valid);

        if (threadIdx.x == 0) {
            atomicAdd(&dot_ms_my, block_sum);
        }
    }
}

template <bool even>
CUDA_GLOBAL void kAlpha() {
    if (even) {
        if (dot_ms_my <= 0) {
            alpha = neg_BB1_fallback;
        } else {
            alpha = Min(a_max, Max(a_min, dot_ms_ms / dot_ms_my));
        }
    } else {
        if (dot_ms_my <= 0) {
            alpha = neg_BB2_fallback;
        } else {
            alpha = Min(a_max, Max(a_min, dot_ms_my / dot_my_my));
        }
    }
}

CUDA_GLOBAL void kCompute_ml_p(int num_items, real* ml, real* mg, real* ml_p) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_items) {
        ml_p[i] = ml[i] + alpha * mg[i];
    }
}
CUDA_GLOBAL void kResidual(int num_items, real* mg, real* dot_g_proj_norm) {
    typedef hipcub::BlockReduce<real, num_threads_per_block> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    const int block_start = blockDim.x * blockIdx.x;
    const int num_valid = min(num_items - block_start, blockDim.x);
    real data, block_sum;
    const int tid = block_start + threadIdx.x;
    if (tid < num_items) {
        data = gdiff * mg[tid] / (-gdiff);
        block_sum = BlockReduce(temp_storage).Reduce(data, hipcub::Sum(), num_valid);

        if (threadIdx.x == 0) {
            atomicAdd(&dot_g_proj_norm[0], block_sum);
        }
    }
}
void ChMPM::BBSolver(gpu_vector<real>& r, gpu_vector<real>& delta_v) {
    const uint size = rhs.size();

    temp.resize(size);
    ml.resize(size);
    mg.resize(size);
    mg_p.resize(size);
    ml_candidate.resize(size);
    ms.resize(size);
    my.resize(size);
    mdir.resize(size);
    ml_p.resize(size);

    temp = 0;
    ml = 0;
    mg = 0;
    mg_p = 0;
    ml_candidate = 0;
    ms = 0;
    my = 0;
    mdir = 0;
    ml_p = 0;

    real sigma_min = 0.1;
    real sigma_max = 0.9;
    real alpha = 0.0001;
    real gmma = 1e-4;

    real lastgoodres = 10e30;
    real lastgoodfval = 10e30;

    // Kernel 1
    Multiply(delta_v, mg, r);
    //
    ml = delta_v;
    ml_candidate = delta_v;

    // real mf_p = 0;

    gpu_vector<real> dot_g_proj_norm(1);
    kResetGlobals<false><<<CONFIG(1)>>>(size);

    for (int current_iteration = 0; current_iteration < 40; current_iteration++) {
        kResetGlobals<true><<<CONFIG(1)>>>(size);

        kCompute_ml_p<<<CONFIG(size)>>>(size, ml.data_d, mg.data_d, ml_p.data_d);
        Multiply(ml_p, mg_p, r);
        ml = ml_p;
        mg = mg_p;

        if (current_iteration % 2 == 0) {
            kUpdateAlpha<true><<<CONFIG(size)>>>(size, ml_p.data_d, ml.data_d, mg_p.data_d, mg.data_d);
            kAlpha<true><<<CONFIG(1)>>>();
        } else {
            kUpdateAlpha<false><<<CONFIG(size)>>>(size, ml_p.data_d, ml.data_d, mg_p.data_d, mg.data_d);
            kAlpha<false><<<CONFIG(1)>>>();
        }

        kResidual<<<CONFIG(size)>>>(size, mg.data_d, dot_g_proj_norm.data_d);
        dot_g_proj_norm.copyDeviceToHost();
        real g_proj_norm = Sqrt(dot_g_proj_norm.data_h[0]);
        if (g_proj_norm < lastgoodres) {
            lastgoodres = g_proj_norm;
            // objective_value = mf_p;
            ml_candidate = ml;
        }
    }

    delta_v = ml_candidate;
}

void ChMPM::Solve(const real kernel_radius, std::vector<real3>& positions, std::vector<real3>& velocities) {
    num_mpm_markers = positions.size();

    Bounds(kernel_radius, positions);

    // ========================================================================================
    kRasterize<<<CONFIG(num_mpm_markers)>>>(pos.data_d,        // input
                                            vel.data_d,        // input
                                            node_mass.data_d,  // output
                                            grid_vel.data_d    // output
                                            );

    kRasterize<<<CONFIG(num_mpm_nodes)>>>(pos.data_d,       // input
                                          node_mass.data_d  // output
                                          );

    old_vel_node_mpm = grid_vel;

    kRhs<<<CONFIG(num_mpm_markers)>>>(pos.data_d,            // input
                                      marker_Fe_hat.data_d,  // input
                                      marker_Fe.data_d,      // input
                                      marker_Fp.data_d,      // input
                                      marker_volume.data_d,  // input
                                      rhs.data_d,            // output
                                      marker_volume.data_d);
    delta_v.resize(num_mpm_nodes);
    delta_v.set(0);

    BBSolver(rhs, delta_v);

    // Solver Kernels
    // Resize
}
