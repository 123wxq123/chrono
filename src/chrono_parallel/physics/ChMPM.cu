#include "hip/hip_runtime.h"
#include "chrono_parallel/physics/ChMPM.cuh"
#include "chrono_parallel/physics/MPMUtils.h"
#include "chrono_parallel/ChCudaHelper.cuh"
#include "chrono_parallel/ChGPUVector.cuh"
#include "thirdparty/hipcub/hipcub.hpp"
namespace chrono {

real3 min_bounding_point;
real3 max_bounding_point;

vec3 bins_per_axis;

std::vector<int> particle_node_mapping;
std::vector<int> node_particle_mapping;
std::vector<int> node_start_index;
std::vector<int> particle_number;
uint num_mpm_nodes_active;
std::vector<Mat33> volume_Ap_Fe_transpose;

// GPU Things
real3* lower_bound;
real3* upper_bound;

gpu_vector<real3> pos, vel;
gpu_vector<real> node_mass;
gpu_vector<real> marker_volume;
gpu_vector<real> grid_vel, delta_v;
gpu_vector<real> rhs;
gpu_vector<Mat33> marker_Fe, marker_Fe_hat, marker_Fp, marker_delta_F;
gpu_vector<real> old_vel_node_mpm;
gpu_vector<real> temp, ml, mg, mg_p, ml_candidate, ms, my, mdir, ml_p;

void MPM_UpdateState();
void MPM_ComputeBounds();
void MPM_BBSolver(gpu_vector<real>& rhs, gpu_vector<real>& delta_v);

struct Bounds {
    real minimum[3];
    real maximum[3];
};

MPM_Settings host_settings;

CUDA_CONSTANT MPM_Settings device_settings;
CUDA_CONSTANT Bounds system_bounds;

/////// BB Constants
__device__ real alpha = 0.0001;
__device__ real dot_ms_ms = 0;
__device__ real dot_ms_my = 0;
__device__ real dot_my_my = 0;
__device__ real gdiff = 1;

CUDA_CONSTANT real a_min = 1e-13;
CUDA_CONSTANT real a_max = 1e13;
CUDA_CONSTANT real neg_BB1_fallback = 0.11;
CUDA_CONSTANT real neg_BB2_fallback = 0.12;

#define LOOP_TWO_RING_GPU(X)                                                             \
    const real bin_edge = device_settings.bin_edge;                                      \
    const real inv_bin_edge = 1.f / bin_edge;                                            \
                                                                                         \
    const int cx = GridCoord(xi.x, inv_bin_edge, system_bounds.minimum[0]);              \
    const int cy = GridCoord(xi.y, inv_bin_edge, system_bounds.minimum[1]);              \
    const int cz = GridCoord(xi.z, inv_bin_edge, system_bounds.minimum[2]);              \
    vec3 bins_per_axis(device_settings.bins_per_axis_x, device_settings.bins_per_axis_y, \
                       device_settings.bins_per_axis_z);                                 \
    for (int i = cx - 2; i <= cx + 2; ++i) {                                             \
        for (int j = cy - 2; j <= cy + 2; ++j) {                                         \
            for (int k = cz - 2; k <= cz + 2; ++k) {                                     \
                const int current_node = GridHash(i, j, k, bins_per_axis);               \
                real3 current_node_location;                                             \
                current_node_location.x = i * bin_edge + system_bounds.minimum[0];       \
                current_node_location.y = j * bin_edge + system_bounds.minimum[1];       \
                current_node_location.z = k * bin_edge + system_bounds.minimum[2];       \
                X                                                                        \
            }                                                                            \
        }                                                                                \
    }

//////========================================================================================================================================================================
////
CUDA_GLOBAL void kComputeBounds(const real3* pos,  // input
                                real3* lower,      // output
                                real3* upper       // output
                                ) {
    typedef hipcub::BlockReduce<real3, num_threads_per_block> BlockReduce;

    __shared__ typename BlockReduce::TempStorage temp_storage;

    const int block_start = blockDim.x * blockIdx.x;
    const int num_valid = min(device_settings.num_mpm_markers - block_start, blockDim.x);

    const int index = block_start + threadIdx.x;
    if (index < device_settings.num_mpm_markers) {
        real3 data = pos[index];

        real3 blockUpper = BlockReduce(temp_storage).Reduce(data, real3Max(), num_valid);

        // sync threads because second reduce uses same temp storage as first
        __syncthreads();

        real3 blockLower = BlockReduce(temp_storage).Reduce(data, real3Min(), num_valid);

        if (threadIdx.x == 0) {
            // write out block results, expanded by the radius
            AtomicMax(upper, blockUpper);
            AtomicMin(lower, blockLower);
        }
    }
}
////========================================================================================================================================================================
CUDA_GLOBAL void kRasterize(const real3* sorted_pos,  // input
                            const real3* sorted_vel,  // input
                            real* grid_mass,          // output
                            real* grid_vel) {         // output
    const int p = blockIdx.x * blockDim.x + threadIdx.x;
    if (p < device_settings.num_mpm_markers) {
        const real3 xi = sorted_pos[p];
        const real3 vi = sorted_vel[p];
        LOOP_TWO_RING_GPU(                                                                     //
            real weight = N(xi - current_node_location, inv_bin_edge) * device_settings.mass;  //
            atomicAdd(&grid_mass[current_node], weight);                                       //
            atomicAdd(&grid_vel[current_node * 3 + 0], weight * vi.x);
            atomicAdd(&grid_vel[current_node * 3 + 1], weight * vi.y);
            atomicAdd(&grid_vel[current_node * 3 + 2], weight * vi.z);
            // AtomicAdd(&((real3*)grid_vel)[current_node * 3], weight * real3(vi));  //
            )
    }
}
CUDA_GLOBAL void kRasterize(const real3* sorted_pos,  // input
                            real* grid_mass) {        // output
    const int p = blockIdx.x * blockDim.x + threadIdx.x;
    if (p < device_settings.num_mpm_markers) {
        const real3 xi = sorted_pos[p];
        LOOP_TWO_RING_GPU(                                                                     //
            real weight = N(xi - current_node_location, inv_bin_edge) * device_settings.mass;  //
            atomicAdd(&grid_mass[current_node], weight);                                       //
            )
    }
}
//
////========================================================================================================================================================================
//
CUDA_GLOBAL void kNormalizeWeights(real* grid_mass,   // input
                                   real* grid_vel) {  // output
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < device_settings.num_mpm_nodes) {
        real n_mass = grid_mass[i];
        if (n_mass > C_EPSILON) {
            grid_vel[i * 3 + 0] /= n_mass;
            grid_vel[i * 3 + 1] /= n_mass;
            grid_vel[i * 3 + 2] /= n_mass;
        }
    }
}
//////========================================================================================================================================================================
////
CUDA_GLOBAL void kComputeParticleVolumes(const real3* sorted_pos,  // input
                                         real* grid_mass,          // output
                                         real* volume) {
    const int p = blockIdx.x * blockDim.x + threadIdx.x;
    if (p < device_settings.num_mpm_markers) {
        const real3 xi = sorted_pos[p];
        real particle_density = 0;
        LOOP_TWO_RING_GPU(                                              //
            real weight = N(xi - current_node_location, inv_bin_edge);  //
            particle_density += grid_mass[current_node] * weight;       //
            )
        // Inverse density to remove division
        particle_density = (bin_edge * bin_edge * bin_edge) / particle_density;
        volume[p] = device_settings.mass * particle_density;
    }
}
CUDA_GLOBAL void kFeHat(const real3* sorted_pos,  // input
                        const Mat33* marker_Fe,   // input
                        const real* grid_vel,     // input
                        Mat33* marker_Fe_hat) {   // output

    const int p = blockIdx.x * blockDim.x + threadIdx.x;
    if (p < device_settings.num_mpm_markers) {
        const real3 xi = sorted_pos[p];
        marker_Fe_hat[p] = Mat33(1.0);
        Mat33 Fe_hat_t(1.0);
        LOOP_TWO_RING_GPU(                                                             //
            real3 vel(grid_vel[i * 3 + 0], grid_vel[i * 3 + 1], grid_vel[i * 3 + 2]);  //
            real3 kern = dN(xi - current_node_location, inv_bin_edge);                 //
            Fe_hat_t += OuterProduct(device_settings.dt * vel, kern);)
        marker_Fe_hat[p] = Fe_hat_t * marker_Fe[p];
    }
}
CUDA_GLOBAL void kRhs(const real3* sorted_pos,     // input
                      const Mat33* marker_Fe_hat,  // input
                      const Mat33* marker_Fe,      // input
                      const Mat33* marker_Fp,      // input
                      const real* marker_volume,   // input
                      real* rhs,                   // output
                      real* volume) {
    const int p = blockIdx.x * blockDim.x + threadIdx.x;
    if (p < device_settings.num_mpm_markers) {
        const real3 xi = sorted_pos[p];

        Mat33 PED =
            Potential_Energy_Derivative_Deviatoric(marker_Fe_hat[p], marker_Fp[p], device_settings.mu,
                                                   device_settings.lambda, device_settings.hardening_coefficient);

        // Print(PED, "PED");

        Mat33 vPEDFepT = marker_volume[p] * MultTranspose(PED, marker_Fe[p]);
        real JE = Determinant(marker_Fe[p]);  //
        real JP = Determinant(marker_Fp[p]);

        LOOP_TWO_RING_GPU(                                                  //
            real3 d_weight = dN(xi - current_node_location, inv_bin_edge);  //
            real3 force = device_settings.dt * (vPEDFepT * d_weight) / (JE * JP);

            atomicAdd(&rhs[current_node * 3 + 0], -force.x);
            atomicAdd(&rhs[current_node * 3 + 1], -force.y);
            atomicAdd(&rhs[current_node * 3 + 2], -force.z);

            //            printf("rhs: [%f %f %f] %f %f\n", d_weight.x, d_weight.y, d_weight.z, device_settings.dt, (JE
            //            * JP));

            )
    }
}

CUDA_GLOBAL void kMultiplyA(const real3* sorted_pos,  // input
                            const real* v_array,
                            const real* old_vel_node_mpm,
                            const Mat33* marker_Fe_hat,  // input
                            const Mat33* marker_Fe,      // input
                            const Mat33* marker_Fp,      // input
                            const real* marker_volume,   // input
                            real* result_array) {
    const int p = blockIdx.x * blockDim.x + threadIdx.x;
    if (p < device_settings.num_mpm_markers) {
        const real3 xi = sorted_pos[p];
        Mat33 delta_F(0);
        {
            LOOP_TWO_RING_GPU(  //
                real3 vnew(v_array[current_node * 3 + 0], v_array[current_node * 3 + 1], v_array[current_node * 3 + 2]);
                real3 vold(old_vel_node_mpm[current_node * 3 + 0], old_vel_node_mpm[current_node * 3 + 1],
                           old_vel_node_mpm[current_node * 3 + 2]);
                real3 v0 = vold + vnew;                                   //
                real3 v1 = dN(xi - current_node_location, inv_bin_edge);  //
                delta_F += OuterProduct(v0, v1);                          //
                )
        }
        // Mat33 A = delta_F;
        //        printf("%s %d: [%f,%f,%f,%f,%f,%f,%f,%f,%f]\n", "vold", p, A[0], A[1], A[2], A[4], A[5], A[6], A[8],
        //        A[9],
        //               A[10]);

        delta_F = delta_F * marker_Fe[p];

        real plastic_determinant = Determinant(marker_Fp[p]);
        real current_mu = device_settings.mu * Exp(device_settings.hardening_coefficient * (1.0 - plastic_determinant));
        Mat33 RD = Rotational_Derivative(marker_Fe_hat[p], delta_F);
        Mat33 volume_Ap_Fe_transpose = marker_volume[p] * MultTranspose(2 * current_mu * (delta_F - RD), marker_Fe[p]);
        {
            LOOP_TWO_RING_GPU(  //
                real3 res = volume_Ap_Fe_transpose * dN(xi - current_node_location, inv_bin_edge);
                atomicAdd(&result_array[current_node * 3 + 0], res.x);
                atomicAdd(&result_array[current_node * 3 + 1], res.y);
                atomicAdd(&result_array[current_node * 3 + 2], res.z););
        }
    }
}
CUDA_GLOBAL void kMultiplyB(const real* v_array,
                            const real* old_vel_node_mpm,
                            const real* node_mass,
                            real* result_array) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < device_settings.num_mpm_nodes) {
        real mass = node_mass[i];
        if (mass > 0) {
            result_array[i * 3 + 0] += mass * (v_array[i * 3 + 0] + old_vel_node_mpm[i * 3 + 0]);
            result_array[i * 3 + 1] += mass * (v_array[i * 3 + 1] + old_vel_node_mpm[i * 3 + 1]);
            result_array[i * 3 + 2] += mass * (v_array[i * 3 + 2] + old_vel_node_mpm[i * 3 + 2]);
        }
    }
}

void MPM_ComputeBounds() {
    max_bounding_point = real3(-FLT_MAX);
    min_bounding_point = real3(FLT_MAX);

    hipMemcpyAsync(lower_bound, &min_bounding_point, sizeof(real3), hipMemcpyHostToDevice);
    hipMemcpyAsync(upper_bound, &max_bounding_point, sizeof(real3), hipMemcpyHostToDevice);

    kComputeBounds<<<CONFIG(host_settings.num_mpm_markers)>>>(pos.data_d,    //
                                                              lower_bound,   //
                                                              upper_bound);  //

    hipMemcpy(&min_bounding_point, lower_bound, sizeof(real3), hipMemcpyDeviceToHost);
    hipMemcpy(&max_bounding_point, upper_bound, sizeof(real3), hipMemcpyDeviceToHost);

    min_bounding_point.x = host_settings.kernel_radius * Round(min_bounding_point.x / host_settings.kernel_radius);
    min_bounding_point.y = host_settings.kernel_radius * Round(min_bounding_point.y / host_settings.kernel_radius);
    min_bounding_point.z = host_settings.kernel_radius * Round(min_bounding_point.z / host_settings.kernel_radius);

    max_bounding_point.x = host_settings.kernel_radius * Round(max_bounding_point.x / host_settings.kernel_radius);
    max_bounding_point.y = host_settings.kernel_radius * Round(max_bounding_point.y / host_settings.kernel_radius);
    max_bounding_point.z = host_settings.kernel_radius * Round(max_bounding_point.z / host_settings.kernel_radius);

    max_bounding_point = max_bounding_point + host_settings.kernel_radius * 8;
    min_bounding_point = min_bounding_point - host_settings.kernel_radius * 6;

    hipMemcpyToSymbolAsync(HIP_SYMBOL(system_bounds), &min_bounding_point, sizeof(real3), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbolAsync(HIP_SYMBOL(system_bounds), &max_bounding_point, sizeof(real3), sizeof(real3), hipMemcpyHostToDevice);

    host_settings.bin_edge = host_settings.kernel_radius * 2;
    real3 bpa = real3(max_bounding_point - min_bounding_point) / host_settings.bin_edge;

    bins_per_axis.x = bpa.x;
    bins_per_axis.y = bpa.y;
    bins_per_axis.z = bpa.z;

    host_settings.inv_bin_edge = real(1.) / host_settings.bin_edge;
    host_settings.num_mpm_nodes = bins_per_axis.x * bins_per_axis.y * bins_per_axis.z;
    host_settings.bins_per_axis_x = bins_per_axis.x;
    host_settings.bins_per_axis_y = bins_per_axis.y;
    host_settings.bins_per_axis_z = bins_per_axis.z;

    cudaCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(device_settings), &host_settings, sizeof(MPM_Settings)));

    printf("max_bounding_point [%f %f %f]\n", max_bounding_point.x, max_bounding_point.y, max_bounding_point.z);
    printf("min_bounding_point [%f %f %f]\n", min_bounding_point.x, min_bounding_point.y, min_bounding_point.z);
    printf("Compute DOF [%d %d %d] [%f] %d %d\n", bins_per_axis.x, bins_per_axis.y, bins_per_axis.z,
           host_settings.bin_edge, host_settings.num_mpm_nodes, host_settings.num_mpm_markers);
}
//

void Multiply(gpu_vector<real>& input, gpu_vector<real>& output) {
    int size = input.size();

    //    old_vel_node_mpm.copyDeviceToHost();
    // pos.copyDeviceToHost();

    //    for (int i = 0; i < host_settings.num_mpm_nodes; i++) {
    //        printf("pd: %d [%.20f %.20f %.20f]\n", i, pos.data_h[i * 3 + 0], pos.data_h[i * 3 + 1], pos.data_h[i * 3 +
    //        2]);
    //    }
    //    for (int i = 0; i < host_settings.num_mpm_markers; i++) {
    //        printf("pd: %d [%.20f %.20f %.20f]\n", i, pos.data_h[i].x, pos.data_h[i].y, pos.data_h[i].z);
    //    }
    kMultiplyA<<<CONFIG(size)>>>(pos.data_d,    // input
                                 input.data_d,  //
                                 old_vel_node_mpm.data_d,
                                 marker_Fe_hat.data_d,  // input
                                 marker_Fe.data_d,      // input
                                 marker_Fp.data_d,      // input
                                 marker_volume.data_d,  // input
                                 output.data_d);
    //    output.copyDeviceToHost();
    //
    //    for (int i = 0; i < host_settings.num_mpm_nodes; i++) {
    //        printf("Nd: %d [%.20f %.20f %.20f]\n", i, output[i * 3 + 0], output[i * 3 + 1], output[i * 3 + 2]);
    //    }

    kMultiplyB<<<CONFIG(size)>>>(input.data_d, old_vel_node_mpm.data_d, node_mass.data_d, output.data_d);
}

CUDA_GLOBAL void kSubtract(int size, real* x, real* y) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        // printf("sg: %d %f\n", i, y[i]);
        y[i] = y[i] - x[i];
    }
}

template <bool inner>
CUDA_GLOBAL void kResetGlobals(int size) {
    if (inner) {
        dot_ms_ms = 0;
        dot_ms_my = 0;
        dot_my_my = 0;
    } else {
        alpha = 0.0001;
        gdiff = 1.0 / pow(size, 2.0);
        // printf("gdiff, alpha, [%.20f %f] \n", gdiff, alpha);
    }
}

template <bool even>
CUDA_GLOBAL void kUpdateAlpha(int num_items, real* ml_p, real* ml, real* mg_p, real* mg) {
    typedef hipcub::BlockReduce<real, num_threads_per_block> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    const int block_start = blockDim.x * blockIdx.x;
    const int num_valid = min(num_items - block_start, blockDim.x);

    const int tid = block_start + threadIdx.x;
    if (tid < num_items) {
        real data, block_sum;
        real ms = ml_p[tid] - ml[tid];
        real my = mg_p[tid] - mg[tid];

        if (even) {
            data = ms * ms;
            block_sum = BlockReduce(temp_storage).Reduce(data, hipcub::Sum(), num_valid);
            if (threadIdx.x == 0) {
                atomicAdd(&dot_ms_ms, block_sum);
            }
        } else {
            data = my * my;
            block_sum = BlockReduce(temp_storage).Reduce(data, hipcub::Sum(), num_valid);
            if (threadIdx.x == 0) {
                atomicAdd(&dot_my_my, block_sum);
            }
        }
        __syncthreads();
        data = ms * my;
        block_sum = BlockReduce(temp_storage).Reduce(data, hipcub::Sum(), num_valid);

        if (threadIdx.x == 0) {
            atomicAdd(&dot_ms_my, block_sum);
        }
    }
}

template <bool even>
CUDA_GLOBAL void kAlpha() {
    if (even) {
        if (dot_ms_my <= 0) {
            alpha = neg_BB1_fallback;
        } else {
            alpha = Min(a_max, Max(a_min, dot_ms_ms / dot_ms_my));
        }
    } else {
        if (dot_ms_my <= 0) {
            alpha = neg_BB2_fallback;
        } else {
            alpha = Min(a_max, Max(a_min, dot_ms_my / dot_my_my));
        }
    }
    // printf("alpha: %f %f %f %f \n", alpha, dot_ms_ms, dot_ms_my, dot_my_my);
}

CUDA_GLOBAL void kCompute_ml_p(int num_items, real* ml, real* mg, real* ml_p) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_items) {
        ml_p[i] = ml[i] - alpha * mg[i];
        // printf("mlps : [%f %f %f]\n", ml_p[i], ml[i], mg[i]);
    }
}
CUDA_GLOBAL void kResidual(int num_items, real* mg, real* dot_g_proj_norm) {
    typedef hipcub::BlockReduce<real, num_threads_per_block> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    const int block_start = blockDim.x * blockIdx.x;
    const int num_valid = min(num_items - block_start, blockDim.x);
    real data, block_sum;
    const int tid = block_start + threadIdx.x;
    if (tid < num_items) {
        data = mg[tid] * mg[tid];

        block_sum = BlockReduce(temp_storage).Reduce(data, hipcub::Sum(), num_valid);

        if (threadIdx.x == 0) {
            atomicAdd(&dot_g_proj_norm[0], block_sum);
        }
        // printf("resid [%f %f]\n", mg[tid], dot_g_proj_norm[0]);
    }
}
void MPM_BBSolver(gpu_vector<real>& r, gpu_vector<real>& delta_v) {
    const uint size = r.size();
    gpu_vector<real> dot_g_proj_norm(1);
    ml.resize(size);
    mg.resize(size);
    mg_p.resize(size);
    ml_candidate.resize(size);
    mdir.resize(size);
    ml_p.resize(size);

    cudaCheck(hipPeekAtLastError());
    cudaCheck(hipDeviceSynchronize());

    mg = 0;
    mg_p = 0;
    ml = delta_v;
    ml_candidate = delta_v;

    cudaCheck(hipPeekAtLastError());
    cudaCheck(hipDeviceSynchronize());

    real lastgoodres = 10e30;

    cudaCheck(hipPeekAtLastError());
    cudaCheck(hipDeviceSynchronize());
    // Kernel 1

    Multiply(ml, mg);

    kSubtract<<<CONFIG(size)>>>(size, r.data_d, mg.data_d);

    mg_p = mg;

    cudaCheck(hipPeekAtLastError());
    cudaCheck(hipDeviceSynchronize());
    kResetGlobals<false><<<1, 1>>>(size);

    for (int current_iteration = 0; current_iteration < host_settings.num_iterations; current_iteration++) {
        kResetGlobals<true><<<1, 1>>>(size);

        kCompute_ml_p<<<CONFIG(size)>>>(size, ml.data_d, mg.data_d, ml_p.data_d);
        mg_p = 0;
        Multiply(ml_p, mg_p);
        kSubtract<<<CONFIG(size)>>>(size, r.data_d, mg_p.data_d);

        if (current_iteration % 2 == 0) {
            kUpdateAlpha<true><<<CONFIG(size)>>>(size, ml_p.data_d, ml.data_d, mg_p.data_d, mg.data_d);
            kAlpha<true><<<1, 1>>>();
        } else {
            kUpdateAlpha<false><<<CONFIG(size)>>>(size, ml_p.data_d, ml.data_d, mg_p.data_d, mg.data_d);
            kAlpha<false><<<1, 1>>>();
        }

        ml = ml_p;
        mg = mg_p;

        dot_g_proj_norm = 0;

        kResidual<<<CONFIG(size)>>>(size, mg.data_d, dot_g_proj_norm.data_d);
        dot_g_proj_norm.copyDeviceToHost();
        real g_proj_norm = Sqrt(dot_g_proj_norm.data_h[0]);
        printf("[%f %f]\n", g_proj_norm, dot_g_proj_norm.data_h[0]);
        if (g_proj_norm < lastgoodres) {
            lastgoodres = g_proj_norm;
            ml_candidate = ml;
        }
    }

    delta_v = ml_candidate;
}
CUDA_GLOBAL void kIncrementVelocity(real* delta_v, real* grid_vel) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < device_settings.num_mpm_nodes) {
        grid_vel[i * 3 + 0] += delta_v[i * 3 + 0];
        grid_vel[i * 3 + 1] += delta_v[i * 3 + 1];
        grid_vel[i * 3 + 2] += delta_v[i * 3 + 2];
    }
}

CUDA_GLOBAL void kUpdateParticleVelocity(real* grid_vel,
                                         real* old_vel_node_mpm,
                                         real3* pos_marker,
                                         real3* vel_marker,
                                         Mat33* marker_Fe,
                                         Mat33* marker_Fp) {
    const int p = blockIdx.x * blockDim.x + threadIdx.x;
    if (p < device_settings.num_mpm_markers) {
        const real3 xi = pos_marker[p];
        real3 V_flip = vel_marker[p];
        real3 V_pic = real3(0.0);

        Mat33 velocity_gradient(0);

        LOOP_TWO_RING_GPU(                                              //
            real weight = N(xi - current_node_location, inv_bin_edge);  //
            real3 g_vel(grid_vel[current_node * 3 + 0], grid_vel[current_node * 3 + 1], grid_vel[current_node * 3 + 2]);
            V_pic += g_vel * weight;                                                                 //
            V_flip.x += (g_vel.x - old_vel_node_mpm[current_node * 3 + 0]) * weight;                 //
            V_flip.y += (g_vel.y - old_vel_node_mpm[current_node * 3 + 1]) * weight;                 //
            V_flip.z += (g_vel.z - old_vel_node_mpm[current_node * 3 + 2]) * weight;                 //
            velocity_gradient += OuterProduct(g_vel, dN(xi - current_node_location, inv_bin_edge));  //
            )
        real3 new_vel = (1.0 - alpha) * V_pic + alpha * V_flip;

        real speed = Length(new_vel);
        if (speed > device_settings.max_velocity) {
            new_vel = new_vel * device_settings.max_velocity / speed;
        }
        vel_marker[p] = new_vel;

        Mat33 Fe_tmp = (Mat33(1.0) + device_settings.dt * velocity_gradient) * marker_Fe[p];
        Mat33 F_tmp = Fe_tmp * marker_Fp[p];
        Mat33 U, V;
        real3 E;
        SVD(Fe_tmp, U, E, V);
        real3 E_clamped;

        E_clamped.x = Clamp(E.x, 1.0 - device_settings.theta_c, 1.0 + device_settings.theta_s);
        E_clamped.y = Clamp(E.y, 1.0 - device_settings.theta_c, 1.0 + device_settings.theta_s);
        E_clamped.z = Clamp(E.z, 1.0 - device_settings.theta_c, 1.0 + device_settings.theta_s);

        marker_Fe[p] = U * MultTranspose(Mat33(E_clamped), V);
        // Inverse of Diagonal E_clamped matrix is 1/E_clamped
        marker_Fp[p] = V * MultTranspose(Mat33(1.0 / E_clamped), U) * F_tmp;
    }
}

void MPM_Solve(MPM_Settings& settings, std::vector<real3>& positions, std::vector<real3>& velocities) {
    host_settings = settings;

    pos.data_h = positions;
    pos.copyHostToDevice();

    vel.data_h = velocities;
    vel.copyHostToDevice();

    cudaCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(device_settings), &host_settings, sizeof(MPM_Settings)));

    MPM_ComputeBounds();

    node_mass.resize(host_settings.num_mpm_nodes);
    node_mass = 0;

    grid_vel.resize(host_settings.num_mpm_nodes * 3);
    grid_vel = 0;

    // ========================================================================================
    kRasterize<<<CONFIG(host_settings.num_mpm_markers)>>>(pos.data_d,        // input
                                                          vel.data_d,        // input
                                                          node_mass.data_d,  // output
                                                          grid_vel.data_d    // output
                                                          );

    kNormalizeWeights<<<CONFIG(host_settings.num_mpm_nodes)>>>(node_mass.data_d,  // output
                                                               grid_vel.data_d);

    old_vel_node_mpm.resize(host_settings.num_mpm_nodes * 3);
    old_vel_node_mpm = grid_vel;

    //    cudaCheck(hipPeekAtLastError());
    //    cudaCheck(hipDeviceSynchronize());

    rhs.resize(host_settings.num_mpm_nodes * 3);
    rhs = 0;

    kFeHat<<<CONFIG(host_settings.num_mpm_markers)>>>(pos.data_d, marker_Fe.data_d, grid_vel.data_d,
                                                      marker_Fe_hat.data_d);

    kRhs<<<CONFIG(host_settings.num_mpm_markers)>>>(pos.data_d,            // input
                                                    marker_Fe_hat.data_d,  // input
                                                    marker_Fe.data_d,      // input
                                                    marker_Fp.data_d,      // input
                                                    marker_volume.data_d,  // input
                                                    rhs.data_d,            // output
                                                    marker_volume.data_d);

    //    rhs.copyDeviceToHost();
    //
    //    for (int i = 0; i < host_settings.num_mpm_nodes; i++) {
    //        printf("Rd: %d [%.20f %.20f %.20f]\n", i, rhs.data_h[i * 3 + 0], rhs.data_h[i * 3 + 1], rhs.data_h[i * 3 +
    //        2]);
    //    }

    delta_v.resize(host_settings.num_mpm_nodes * 3);
    delta_v = 0;

    MPM_BBSolver(rhs, delta_v);

    kIncrementVelocity<<<CONFIG(host_settings.num_mpm_nodes)>>>(delta_v.data_d, grid_vel.data_d);
    kUpdateParticleVelocity<<<CONFIG(host_settings.num_mpm_markers)>>>(
        grid_vel.data_d, old_vel_node_mpm.data_d, pos.data_d, vel.data_d, marker_Fe.data_d, marker_Fp.data_d);
    vel.copyDeviceToHost();

    velocities = vel.data_h;
}

CUDA_GLOBAL void kInitFeFp(Mat33* marker_Fe, Mat33* marker_Fp) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < device_settings.num_mpm_markers) {
        marker_Fe[i] = Mat33(1);
        marker_Fp[i] = Mat33(1);
    }
}

void MPM_Initialize(MPM_Settings& settings, std::vector<real3>& positions) {
    host_settings = settings;

    cudaCheck(hipMalloc(&lower_bound, sizeof(real3)));
    cudaCheck(hipMalloc(&upper_bound, sizeof(real3)));

    pos.data_h = positions;
    pos.copyHostToDevice();

    cudaCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(device_settings), &host_settings, sizeof(MPM_Settings)));

    MPM_ComputeBounds();
    marker_volume.resize(host_settings.num_mpm_markers);
    node_mass.resize(host_settings.num_mpm_nodes);
    node_mass = 0;

    kRasterize<<<CONFIG(host_settings.num_mpm_markers)>>>(pos.data_d,         // input
                                                          node_mass.data_d);  // output

    kComputeParticleVolumes<<<CONFIG(host_settings.num_mpm_markers)>>>(pos.data_d,             // input
                                                                       node_mass.data_d,       // input
                                                                       marker_volume.data_d);  // output

    marker_Fe.resize(host_settings.num_mpm_markers);
    marker_Fe_hat.resize(host_settings.num_mpm_markers);
    marker_Fp.resize(host_settings.num_mpm_markers);
    marker_delta_F.resize(host_settings.num_mpm_markers);

    kInitFeFp<<<CONFIG(host_settings.num_mpm_markers)>>>(marker_Fe.data_d,   // output
                                                         marker_Fp.data_d);  // output

    cudaCheck(hipPeekAtLastError());
    cudaCheck(hipDeviceSynchronize());
}
}
