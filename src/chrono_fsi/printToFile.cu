#include "hip/hip_runtime.h"
#include <string.h>
#include <stdio.h>
#include <sstream>
#include <fstream>
#include <thrust/reduce.h>
#include "printToFile.cuh"
#include "custom_cutil_math.h"
#include "SPHCudaUtils.h"
using namespace std;

//*******************************************************************************************************************************
void PrintCartesianData_MidLine(
		const thrust::host_vector<Real4> & rho_Pres_CartH,
		const thrust::host_vector<Real4> & vel_VelMag_CartH,
		const int3 & cartesianGridDims,
		const SimParams & paramsH) {
	int3 gridCenter = mI3(cartesianGridDims.x / 2, cartesianGridDims.y / 2, cartesianGridDims.z / 2);
	stringstream midLineProfile;
	for (int k = 0; k < cartesianGridDims.z; k ++) {
		//Assuming flow in x Direction, walls on Z direction, periodic on y direction
		int index = (cartesianGridDims.x * cartesianGridDims.y) * k + cartesianGridDims.x * gridCenter.y + gridCenter.x;
		Real3 v = mR3(vel_VelMag_CartH[index]);
		Real3 rp = mR3(rho_Pres_CartH[index]);
//		midLineProfile << v.x << ", " << v.y << ", " << v.z << ", " << length(v) << ", " << rp.x << ", " << rp.y << endl;
		midLineProfile << v.x << ", " ;
	}
	midLineProfile << endl;
	static int count = 0;
	ofstream midLineData;
	if (count == 0) {
		midLineData.open("MidLineData.txt");
	} else {
		midLineData.open("MidLineData.txt", ios::app);
	}
	count ++;
	midLineData << midLineProfile.str();
	midLineData.close();
}
//*******************************************************************************************************************************
void PrintToFile_SPH(
		const thrust::device_vector<Real3> & posRadD,
		const thrust::device_vector<Real4> & velMasD,
		const thrust::device_vector<Real4> & rhoPresMuD,
		const thrust::host_vector<int3> & referenceArray,

		const SimParams paramsH,
		const Real realTime,
		int tStep,
		int stepSave) {


	thrust::host_vector<Real3> posRadH = posRadD;
	thrust::host_vector<Real4> velMasH = velMasD;
	thrust::host_vector<Real4> rhoPresMuH = rhoPresMuD;

// ######## the commented sections need to be fixed. you need cartesian data by calling SphSystemGpu.MapSPH_ToGrid
////////-+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++//comcom
//	ofstream fileNameCartesianTotal;
//	thrust::host_vector<Real4> rho_Pres_CartH(1);
//	thrust::host_vector<Real4> vel_VelMag_CartH(1);
//	Real resolution = 2 * paramsH.HSML;
//	int3 cartesianGridDims;
//	int tStepCartesianTotal = 1000000;
//	int tStepCartesianSlice = 100000;
//	int tStepPoiseuilleProf = 1000; //tStepCartesianSlice;
//
//	int stepCalcCartesian = min(tStepCartesianTotal, tStepCartesianSlice);
//	stepCalcCartesian = min(stepCalcCartesian, tStepPoiseuilleProf);
//
//	if (tStep % stepCalcCartesian == 0) {
//		MapSPH_ToGrid(resolution, cartesianGridDims, rho_Pres_CartH, vel_VelMag_CartH, posRadD, velMasD, rhoPresMuD,
//				referenceArray[referenceArray.size() - 1].y, paramsH);
//	}
//	if (tStep % tStepCartesianTotal == 0) {
//		if (tStep / tStepCartesianTotal == 0) {
//			fileNameCartesianTotal.open("dataCartesianTotal.txt");
//			fileNameCartesianTotal<<"variables = \"x\", \"y\", \"z\", \"Vx\", \"Vy\", \"Vz\", \"Velocity Magnitude\", \"Rho\", \"Pressure\"\n";
//		} else {
//			fileNameCartesianTotal .open("dataCartesianTotal.txt", ios::app);
//		}
//		fileNameCartesianTotal<<"zone I = "<<cartesianGridDims.x<<", J = "<<cartesianGridDims.y<<", K = "<<cartesianGridDims.z<<endl;
//		stringstream ssCartesianTotal;
//		for (int k = 0; k < cartesianGridDims.z; k++) {
//			for (int j = 0; j < cartesianGridDims.y; j++) {
//				for (int i = 0; i < cartesianGridDims.x; i++) {
//					int index = i + j * cartesianGridDims.x + k * cartesianGridDims.x * cartesianGridDims.y;
//					Real3 gridNodeLoc = resolution * mR3(i, j, k) + paramsH.worldOrigin;
//					ssCartesianTotal<<gridNodeLoc.x<<", "<< gridNodeLoc.y<<", "<< gridNodeLoc.z<<", "<<
//							vel_VelMag_CartH[index].x<<", "<< vel_VelMag_CartH[index].y<<", "<< vel_VelMag_CartH[index].z<<", "<< vel_VelMag_CartH[index].w<<", "<<
//							rho_Pres_CartH[index].x<<", "<< rho_Pres_CartH[index].y<<endl;
//				}
//			}
//		}
//		fileNameCartesianTotal<<ssCartesianTotal.str();
//		fileNameCartesianTotal.close();
//	}
//////////-+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++ //comcom
//	ofstream fileNameCartesianMidplane;
//	if (tStep % tStepCartesianSlice == 0) {
//		if (tStep / tStepCartesianSlice == 0) {
//			fileNameCartesianMidplane.open("dataCartesianMidplane.txt");
//			fileNameCartesianMidplane<<"variables = \"x\", \"z\", \"Vx\", \"Vy\", \"Vz\", \"Velocity Magnitude\", \"Rho\", \"Pressure\"\n";
//		} else {
//			fileNameCartesianMidplane .open("dataCartesianMidplane.txt", ios::app);
//		}
//		fileNameCartesianMidplane<< "zone I = "<<cartesianGridDims.x<<", J = "<<cartesianGridDims.z<<"\n";
//		int j = cartesianGridDims.y / 2;
//		stringstream ssCartesianMidplane;
//		for (int k = 0; k < cartesianGridDims.z; k++) {
//			for (int i = 0; i < cartesianGridDims.x; i++) {
//				int index = i + j * cartesianGridDims.x + k * cartesianGridDims.x * cartesianGridDims.y;
//				Real3 gridNodeLoc = resolution * mR3(i, j, k) + paramsH.worldOrigin;
//				ssCartesianMidplane<<gridNodeLoc.x<<", "<< gridNodeLoc.z<<", "<< vel_VelMag_CartH[index].x<<", "<<
//						vel_VelMag_CartH[index].y<<", "<< vel_VelMag_CartH[index].z<<", "<< vel_VelMag_CartH[index].w<<", "<< rho_Pres_CartH[index].x<<", "<<
//						rho_Pres_CartH[index].y<<endl;
//			}
//		}
//		fileNameCartesianMidplane<<ssCartesianMidplane.str();
//		fileNameCartesianMidplane.close();
//	}
//	rho_Pres_CartH.clear();
//////////-+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++comcom
//	ofstream fileVelocityProfPoiseuille;
//	if (tStep % tStepPoiseuilleProf == 0) {
//		if (tStep / tStepPoiseuilleProf == 0) {
//			fileVelocityProfPoiseuille.open("dataVelProfile.txt");
//			fileVelocityProfPoiseuille<< "variables = \"Z(m)\", \"Vx(m/s)\"\n";
//
//		} else {
//			fileVelocityProfPoiseuille.open("dataVelProfile.txt", ios::app);
//		}
//		fileVelocityProfPoiseuille<<"zone T=\"t = "<< realTime <<"\""endl;
//		stringstream ssVelocityProfPoiseuille;
//		int j = cartesianGridDims.y / 2;
//		int i = cartesianGridDims.x / 2;
//		for (int k = 0; k < cartesianGridDims.z; k++) {
//			int index = i + j * cartesianGridDims.x + k * cartesianGridDims.x * cartesianGridDims.y;
//			Real3 gridNodeLoc = resolution * mR3(i, j, k) + paramsH.worldOrigin;
//			if (gridNodeLoc.z > 1 * paramsH.sizeScale && gridNodeLoc.z < 2 * paramsH.sizeScale) {
//				ssVelocityProfPoiseuille<<gridNodeLoc.z<<", "<< vel_VelMag_CartH[index].x<<endl;
//			}
//		}
//		fileVelocityProfPoiseuille<<ssVelocityProfPoiseuille.str();
//		fileVelocityProfPoiseuille.close();
//	}
//	vel_VelMag_CartH.clear();
//////////-+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++comcom


	ofstream fileNameFluidParticles;
	ofstream fileNameBoundaries;
	ofstream fileNameFluidBoundaries;

	int tStepsPovFiles = stepSave;//25;//1000;//2000;
	if (tStep % tStepsPovFiles == 0) {
#ifdef _WIN32
			system("mkdir povFiles");
#else
			system("mkdir -p povFiles");
#endif
		if (tStep / tStepsPovFiles == 0) {
					//linux. In windows, it is System instead of system (to invoke a command in the command line)
			system("rm povFiles/*.csv");
		}
		char fileCounter[5];
		int dumNumChar = sprintf(fileCounter, "%d", int(tStep / tStepsPovFiles) );

		char nameFluid[255];
		sprintf(nameFluid, "povFiles/fluid");
		strcat(nameFluid, fileCounter);
		strcat(nameFluid, ".csv");
		char nameBoundary[255];
		sprintf(nameBoundary, "povFiles/boundary");
		strcat(nameBoundary, fileCounter);
		strcat(nameBoundary, ".csv");
		char nameFluidBoundaries[255];
		sprintf(nameFluidBoundaries, "povFiles/fluid_boundary");
		strcat(nameFluidBoundaries, fileCounter);
		strcat(nameFluidBoundaries, ".csv");
	//*****************************************************
		fileNameFluidParticles.open(nameFluid);
		stringstream ssFluidParticles;
		for (int i = referenceArray[0].x; i < referenceArray[0].y; i++) {
			Real3 pos = posRadH[i];
			Real3 vel = mR3(velMasH[i]);
			Real4 rP = rhoPresMuH[i];
			Real velMag = length(vel);
			ssFluidParticles<< pos.x<<", "<< pos.y<<", "<< pos.z<<", "<< vel.x<<", "<< vel.y<<", "<< vel.z<<", "<< velMag<<", "<< rP.x<<", "<< rP.y<<", "<< rP.w<<", "<<endl;
		}
		fileNameFluidParticles<<ssFluidParticles.str();
		fileNameFluidParticles.close();
	//*****************************************************
//		fileNameBoundaries.open(nameBoundary);
//		stringstream ssBoundary;
//		for (int i = referenceArray[1].x; i < referenceArray[1].y; i++) {
//			Real3 pos = posRadH[i];
//			Real3 vel = mR3(velMasH[i]);
//			Real4 rP = rhoPresMuH[i];
//			Real velMag = length(vel);
//			ssBoundary<<pos.x<<", "<< pos.y<<", "<< pos.z<<", "<< vel.x<<", "<< vel.y<<", "<< vel.z<<", "<< velMag<<", "<< rP.x<<", "<< rP.y<<", "<< rP.w<<", "<<endl;
//		}
//		fileNameBoundaries << ssBoundary.str();
//		fileNameBoundaries.close();
	//*****************************************************
		fileNameFluidBoundaries.open(nameFluidBoundaries);
		stringstream ssFluidBoundaryParticles;
	//		ssFluidBoundaryParticles.precision(20);
		for (int i = referenceArray[0].x; i < referenceArray[1].y; i++) {
			Real3 pos = posRadH[i];
			Real3 vel = mR3(velMasH[i]);
			Real4 rP = rhoPresMuH[i];
			Real velMag = length(vel);
			//if (pos.y > .0002 && pos.y < .0008)
			ssFluidBoundaryParticles<< pos.x<<", "<< pos.y<<", "<< pos.z<<", "<< vel.x<<", "<< vel.y<<", "<< vel.z<<", "<< velMag<<", "<< rP.x<<", "<< rP.y<<", "<< rP.z<<", "<< rP.w<<", "<<endl;
		}
		fileNameFluidBoundaries<<ssFluidBoundaryParticles.str();
		fileNameFluidBoundaries.close();
	//*****************************************************
	}
	posRadH.clear();
	velMasH.clear();
	rhoPresMuH.clear();
}

//*******************************************************************************************************************************

void PrintToFile(
		const thrust::device_vector<Real3> & posRadD,
		const thrust::device_vector<Real4> & velMasD,
		const thrust::device_vector<Real4> & rhoPresMuD,
		const thrust::host_vector<int3> & referenceArray,
		const SimParams paramsH,
		Real realTime,
		int tStep,
		int stepSave) {
	// print fluid stuff
	PrintToFile_SPH(posRadD, velMasD, rhoPresMuD, referenceArray, paramsH, realTime, tStep, stepSave);
}
//*******************************************************************************************************************************
