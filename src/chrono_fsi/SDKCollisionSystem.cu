#include "hip/hip_runtime.h"
/*
 * SDKCollisionSystem.cu
 *
 *  Created on: Mar 2, 2013
 *      Author: Arman Pazouki
 */
#include <thrust/sort.h>
#include "chrono_fsi/SDKCollisionSystem.cuh"


//#include "extraOptionalFunctions.cuh"
//#include "SDKCollisionSystemAdditional.cuh"

/**
 * @brief calcGridHash
 * @details  See SDKCollisionSystem.cuh
 */
__device__ int3 calcGridPos(Real3 p) {
	int3 gridPos;
	gridPos.x = floor((p.x - paramsD.worldOrigin.x) / paramsD.cellSize.x);
	gridPos.y = floor((p.y - paramsD.worldOrigin.y) / paramsD.cellSize.y);
	gridPos.z = floor((p.z - paramsD.worldOrigin.z) / paramsD.cellSize.z);
	return gridPos;
}

/**
 * @brief calcGridHash
 * @details  See SDKCollisionSystem.cuh
 */
__device__ uint calcGridHash(int3 gridPos) {
	gridPos.x -= ((gridPos.x >= paramsD.gridSize.x) ? paramsD.gridSize.x : 0);
	gridPos.y -= ((gridPos.y >= paramsD.gridSize.y) ? paramsD.gridSize.y : 0);
	gridPos.z -= ((gridPos.z >= paramsD.gridSize.z) ? paramsD.gridSize.z : 0);

	gridPos.x += ((gridPos.x < 0) ? paramsD.gridSize.x : 0);
	gridPos.y += ((gridPos.y < 0) ? paramsD.gridSize.y : 0);
	gridPos.z += ((gridPos.z < 0) ? paramsD.gridSize.z : 0);

	return __umul24(__umul24(gridPos.z, paramsD.gridSize.y), paramsD.gridSize.x)
			+ __umul24(gridPos.y, paramsD.gridSize.x) + gridPos.x;
}

/**
 * @brief calcGridHash
 * @details  See SDKCollisionSystem.cuh
 */
__device__ inline Real4 DifVelocityRho(const Real3& dist3, const Real& d,
		const Real3& velMasA, const Real3& vel_XSPH_A, const Real3& velMasB,
		const Real3& vel_XSPH_B, const Real4& rhoPresMuA,
		const Real4& rhoPresMuB, Real multViscosity) {
	Real epsilonMutualDistance = .01f;
	Real3 gradW = GradW(dist3);

	// Real vAB_Dot_rAB = dot(velMasA - velMasB, dist3);

	//	//*** Artificial viscosity type 1.1
	//	Real alpha = .001;
	//	Real c_ab = 10 * paramsD.v_Max; //Ma = .1;//sqrt(7.0f * 10000 / ((rhoPresMuA.x + rhoPresMuB.x) / 2.0f));
	//	//Real h = paramsD.HSML;
	//	Real rho = .5f * (rhoPresMuA.x + rhoPresMuB.x);
	//	Real nu = alpha * paramsD.HSML * c_ab / rho;

	//	//*** Artificial viscosity type 1.2
	//	Real nu = 22.8f * paramsD.mu0 / 2.0f / (rhoPresMuA.x * rhoPresMuB.x);
	//	Real3 derivV = -paramsD.markerMass * (
	//		rhoPresMuA.y / (rhoPresMuA.x * rhoPresMuA.x) + rhoPresMuB.y / (rhoPresMuB.x * rhoPresMuB.x)
	//		- nu * vAB_Dot_rAB / ( d * d + epsilonMutualDistance * paramsD.HSML * paramsD.HSML )
	//		) * gradW;
	//	return mR4(derivV,
	//		rhoPresMuA.x * paramsD.markerMass / rhoPresMuB.x * dot(vel_XSPH_A - vel_XSPH_B, gradW));

	//*** Artificial viscosity type 2
	Real rAB_Dot_GradW = dot(dist3, gradW);
	Real rAB_Dot_GradW_OverDist = rAB_Dot_GradW
			/ (d * d + epsilonMutualDistance * paramsD.HSML * paramsD.HSML);
	Real3 derivV = -paramsD.markerMass
			* (rhoPresMuA.y / (rhoPresMuA.x * rhoPresMuA.x)
					+ rhoPresMuB.y / (rhoPresMuB.x * rhoPresMuB.x)) * gradW
			+ paramsD.markerMass * (8.0f * multViscosity) * paramsD.mu0
					* pow(rhoPresMuA.x + rhoPresMuB.x, Real(-2))
					* rAB_Dot_GradW_OverDist * (velMasA - velMasB);
	Real derivRho = rhoPresMuA.x * paramsD.markerMass / rhoPresMuB.x
			* dot(vel_XSPH_A - vel_XSPH_B, gradW);
	//	Real zeta = 0;//.05;//.1;
	//	Real derivRho = rhoPresMuA.x * paramsD.markerMass * invrhoPresMuBx * (dot(vel_XSPH_A - vel_XSPH_B, gradW)
	//			+ zeta * paramsD.HSML * (10 * paramsD.v_Max) * 2 * (rhoPresMuB.x / rhoPresMuA.x - 1) *
	// rAB_Dot_GradW_OverDist
	//			);
	return mR4(derivV, derivRho);

	//	//*** Artificial viscosity type 1.3
	//	Real rAB_Dot_GradW = dot(dist3, gradW);
	//	Real3 derivV = -paramsD.markerMass * (rhoPresMuA.y / (rhoPresMuA.x * rhoPresMuA.x) + rhoPresMuB.y / (rhoPresMuB.x *
	// rhoPresMuB.x)) * gradW
	//		+ paramsD.markerMass / (rhoPresMuA.x * rhoPresMuB.x) * 2.0f * paramsD.mu0 * rAB_Dot_GradW / ( d * d +
	// epsilonMutualDistance * paramsD.HSML * paramsD.HSML ) * (velMasA - velMasB);
	//	return mR4(derivV,
	//		rhoPresMuA.x * paramsD.markerMass / rhoPresMuB.x * dot(vel_XSPH_A - vel_XSPH_B, gradW));
}
//--------------------------------------------------------------------------------------------------------------------------------
__device__ inline Real3 DifVelocity_SSI_DEM(const Real3& dist3, const Real& d,
		const Real3& velMasA, const Real3& velMasB) {
	// printf("** DifVelocity_SSI_DEM\n");
	Real l = paramsD.MULT_INITSPACE * paramsD.HSML - d;  // penetration distance
	if (l < 0) {
		return mR3(0);
	}
	Real kS = .00006; // 6;//3; //50; //1000.0; //392400.0;	//spring. 50 worked almost fine. I am using 30 to be
					  // sure!
	Real kD = 40; // 20;//40.0;//20.0; //420.0;				//damping coef. // 40 is good don't change it.
	Real3 n = dist3 / d;  // unit vector B to A
	Real m_eff = 0.5 * paramsD.markerMass; //(mA * mB) / (mA + mB);
	Real3 force = (/*pow(paramsD.sizeScale, Real(3)) * */kS * l
			- kD * m_eff * dot(velMasA - velMasB, n)) * n; // relative velocity at contact is simply assumed as the relative vel of the centers. If you are
														   // updating the rotation, this should be modified.
	return force / paramsD.markerMass;  // return dV/dT same as SPH
}
//--------------------------------------------------------------------------------------------------------------------------------
__device__ inline Real3 DifVelocity_SSI_Lubrication(const Real3& dist3,
		const Real& d, const Real3& velMasA, const Real3& velMasB) {
	// printf("** DifVelocity_SSI_Lubrication\n");
	Real Delta_c = paramsD.HSML;
	Real s = d - paramsD.MULT_INITSPACE * paramsD.HSML;
	if (s > Delta_c)
		return mR3(0);

	Real Delta_i = .1 * Delta_c;
	Real mult = 0;
	if (s > Delta_i) {
		mult = 1 / s - 1 / Delta_c;
	} else {
		mult = 1 / Delta_i - 1 / Delta_c;
	}
	Real3 n = dist3 / d;  // unit vector B to A
	Real3 force = -(mult * 1.5 * PI * paramsD.mu0 * paramsD.HSML * paramsD.HSML)
			* dot(velMasA - velMasB, n) * n;
	return force / paramsD.markerMass;
}
//--------------------------------------------------------------------------------------------------------------------------------
// collide a particle against all other particles in a given cell
__device__ Real3 deltaVShare(int3 gridPos, uint index, Real3 posRadA,
		Real3 velMasA, Real4 rhoPresMuA, Real3* sortedPosRad,
		Real3* sortedVelMas, Real4* sortedRhoPreMu, uint* cellStart,
		uint* cellEnd) {
	uint gridHash = calcGridHash(gridPos);
	// get start of bucket for this cell
	Real3 deltaV = mR3(0.0f);

	uint startIndex = FETCH(cellStart, gridHash);
	if (startIndex != 0xffffffff) {  // cell is not empty
		// iterate over particles in this cell
		uint endIndex = FETCH(cellEnd, gridHash);

		for (uint j = startIndex; j < endIndex; j++) {
			if (j != index) {  // check not colliding with self
				Real3 posRadB = FETCH(sortedPosRad, j);
				Real3 dist3 = Distance(posRadA, posRadB);
				Real d = length(dist3);
				if (d > RESOLUTION_LENGTH_MULT * paramsD.HSML)
					continue;
				Real4 rhoPresMuB = FETCH(sortedRhoPreMu, j);
				if (rhoPresMuB.w > -.1)
					continue; //# B must be fluid (A was checked originally and it is fluid at this point), accoring to
				// colagrossi (2003), the other phase (i.e. rigid) should not be considered)
				Real multRho = 2.0f / (rhoPresMuA.x + rhoPresMuB.x);
				Real3 velMasB = FETCH(sortedVelMas, j);
				deltaV += paramsD.markerMass * (velMasB - velMasA) * W3(d)
						* multRho;
			}
		}
	}
	return deltaV;
}
//--------------------------------------------------------------------------------------------------------------------------------
// collide a particle against all other particles in a given cell
__device__ void BCE_modification_Share(
		Real4& deltaVDenom,  // in and out
		Real4& deltaRP, int& isAffected, int3 gridPos, uint index,
		Real3 posRadA, Real3* sortedPosRad, Real3* sortedVelMas,
		Real4* sortedRhoPreMu, uint* cellStart, uint* cellEnd) {
	uint gridHash = calcGridHash(gridPos);
	// get start of bucket for this cell
	uint startIndex = FETCH(cellStart, gridHash);
	if (startIndex != 0xffffffff) {  // cell is not empty
		// iterate over particles in this cell
		uint endIndex = FETCH(cellEnd, gridHash);

		for (uint j = startIndex; j < endIndex; j++) {
			Real3 posRadB = FETCH(sortedPosRad, j);
			Real3 dist3 = Distance(posRadA, posRadB);
			Real d = length(dist3);
			if (d > RESOLUTION_LENGTH_MULT * paramsD.HSML)
				continue;
			Real4 rhoPresMuB = FETCH(sortedRhoPreMu, j);

			Real Wd = W3(d);
			Real3 velMasB = FETCH(sortedVelMas, j);
			//			deltaVDenom += mR4(
			//					paramsD.markerMass / rhoPresMuB.x * velMasB * Wd,
			//					paramsD.markerMass / rhoPresMuB.x * Wd);
			//			deltaVDenom += mR4(
			//					velMasB * Wd,
			//					Wd);

			if (rhoPresMuB.w < -.1) { // only fluid pressure is used to update BCE pressure see Eq 27 of Adami, 2012 paper

				isAffected = (Wd > W3(1.99 * paramsD.HSML));

				deltaVDenom += mR4(velMasB * Wd, Wd);

				deltaRP += mR4(rhoPresMuB.x * dist3 * Wd, // Arman: check if dist3 or -dist3
				rhoPresMuB.y * Wd);
			}
		}
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
// modify pressure for body force
__device__ __inline__ void modifyPressure(Real4& rhoPresMuB,
		const Real3& dist3Alpha) {
	// body force in x direction
	rhoPresMuB.y =
			(dist3Alpha.x > 0.5 * paramsD.boxDims.x) ?
					(rhoPresMuB.y - paramsD.deltaPress.x) : rhoPresMuB.y;
	rhoPresMuB.y =
			(dist3Alpha.x < -0.5 * paramsD.boxDims.x) ?
					(rhoPresMuB.y + paramsD.deltaPress.x) : rhoPresMuB.y;
	// body force in x direction
	rhoPresMuB.y =
			(dist3Alpha.y > 0.5 * paramsD.boxDims.y) ?
					(rhoPresMuB.y - paramsD.deltaPress.y) : rhoPresMuB.y;
	rhoPresMuB.y =
			(dist3Alpha.y < -0.5 * paramsD.boxDims.y) ?
					(rhoPresMuB.y + paramsD.deltaPress.y) : rhoPresMuB.y;
	// body force in x direction
	rhoPresMuB.y =
			(dist3Alpha.z > 0.5 * paramsD.boxDims.z) ?
					(rhoPresMuB.y - paramsD.deltaPress.z) : rhoPresMuB.y;
	rhoPresMuB.y =
			(dist3Alpha.z < -0.5 * paramsD.boxDims.z) ?
					(rhoPresMuB.y + paramsD.deltaPress.z) : rhoPresMuB.y;
}
//--------------------------------------------------------------------------------------------------------------------------------
// collide a particle against all other particles in a given cell
__device__ Real4 collideCell(int3 gridPos, uint index, Real3 posRadA,
		Real3 velMasA, Real3 vel_XSPH_A, Real4 rhoPresMuA, Real3* sortedPosRad,
		Real3* sortedVelMas, Real3* vel_XSPH_Sorted_D, Real4* sortedRhoPreMu,
		uint* cellStart, uint* cellEnd) {

	uint gridHash = calcGridHash(gridPos);
	// get start of bucket for this cell
	Real3 derivV = mR3(0.0f);
	Real derivRho = 0.0f;

	uint startIndex = FETCH(cellStart, gridHash);
	if (startIndex != 0xffffffff) {  // cell is not empty
		// iterate over particles in this cell
		uint endIndex = FETCH(cellEnd, gridHash);

		for (uint j = startIndex; j < endIndex; j++) {
			if (j != index) {  // check not colliding with self
				Real3 posRadB = FETCH(sortedPosRad, j);
				Real3 dist3Alpha = posRadA - posRadB;
				Real3 dist3 = Distance(posRadA, posRadB);
				Real d = length(dist3);
				if (d > RESOLUTION_LENGTH_MULT * paramsD.HSML)
					continue;

				Real4 rhoPresMuB = FETCH(sortedRhoPreMu, j);
				if ((fabs(rhoPresMuB.w - rhoPresMuA.w) < .1)
						&& rhoPresMuA.w > -.1) {
					continue;
				}
				Real3 velMasB = FETCH(sortedVelMas, j);
				modifyPressure(rhoPresMuB, dist3Alpha);
				Real multViscosit = 1;
				Real4 derivVelRho = mR4(0.0f);
				Real3 vel_XSPH_B = FETCH(vel_XSPH_Sorted_D, j);
				derivVelRho = DifVelocityRho(dist3, d, velMasA, vel_XSPH_A,
						velMasB, vel_XSPH_B, rhoPresMuA, rhoPresMuB,
						multViscosit);
				derivV += mR3(derivVelRho);
				derivRho += derivVelRho.w;
			}
		}
	}

	// ff1
	//	if (rhoPresMuA.w > 0) printf("force value %f %f %f\n", 1e20*derivV.x, 1e20*derivV.y, 1e20*derivV.z);
	return mR4(derivV, derivRho);
} //--------------------------------------------------------------------------------------------------------------------------------
// collide a particle against all other particles in a given cell
__device__ __inline__ void stressCell(Real3& devS3, Real3& volS3, int3 gridPos,
		uint index, Real3 posRadA, Real3 velMasA, Real4 rhoPresMuA,
		Real3* sortedPosRad, Real3* sortedVelMas, Real4* sortedRhoPreMu,
		uint* cellStart, uint* cellEnd) {

	uint gridHash = calcGridHash(gridPos);
	// get start of bucket for this cell
	Real3 derivV = mR3(0.0f);

	uint startIndex = FETCH(cellStart, gridHash);
	if (startIndex != 0xffffffff) {  // cell is not empty
		// iterate over particles in this cell
		uint endIndex = FETCH(cellEnd, gridHash);

		for (uint j = startIndex; j < endIndex; j++) {
			if (j != index) {  // check not colliding with self
				Real3 posRadB = FETCH(sortedPosRad, j);
				Real3 dist3Alpha = posRadA - posRadB;
				Real3 dist3 = Distance(posRadA, posRadB);
				Real d = length(dist3);
				if (d > RESOLUTION_LENGTH_MULT * paramsD.HSML)
					continue;

				Real3 velMasB = FETCH(sortedVelMas, j);
				Real4 rhoPresMuB = FETCH(sortedRhoPreMu, j);

				Real3 vr = velMasB - velMasA;
				Real3 gradW = GradW(dist3);

				// Randles and Libersky, 1996
				devS3 += -paramsD.mu0 * paramsD.markerMass / rhoPresMuB.x
						*
						mR3(vr.x * gradW.y + vr.y * gradW.x,
								vr.x * gradW.z + vr.z * gradW.x,
								vr.y * gradW.z + vr.z * gradW.y);
				volS3 += -paramsD.mu0 * paramsD.markerMass / rhoPresMuB.x * 4.0
						/ 3.0
						* mR3(vr.x * gradW.x, vr.y * gradW.y, vr.z * gradW.z);
			}
		}
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
// collide a particle against all other particles in a given cell
__device__ void collideCellDensityReInit(Real& densityShare, Real& denominator,
		int3 gridPos, uint index, Real3 posRadA, Real3* sortedPosRad,
		Real3* sortedVelMas, Real4* sortedRhoPreMu, uint* cellStart,
		uint* cellEnd) {

	//?c2 printf("grid pos %d %d %d \n", gridPos.x, gridPos.y, gridPos.z);
	uint gridHash = calcGridHash(gridPos);
	// get start of bucket for this cell
	Real densityShare2 = 0.0f;
	Real denominator2 = 0.0f;

	uint startIndex = FETCH(cellStart, gridHash);
	if (startIndex != 0xffffffff) {  // cell is not empty
		// iterate over particles in this cell
		uint endIndex = FETCH(cellEnd, gridHash);

		for (uint j = startIndex; j < endIndex; j++) {
			if (j != index) {  // check not colliding with self
				Real3 posRadB = FETCH(sortedPosRad, j);
				Real4 rhoPreMuB = FETCH(sortedRhoPreMu, j);
				Real3 dist3 = Distance(posRadA, posRadB);
				Real d = length(dist3);
				if (d > RESOLUTION_LENGTH_MULT * paramsD.HSML)
					continue;
				Real partialDensity = paramsD.markerMass * W3(d); // optimize it ?$
				densityShare2 += partialDensity;
				denominator2 += partialDensity / rhoPreMuB.x;
				// if (fabs(W3(d)) < .00000001) {printf("good evening, distance %f %f %f\n", dist3.x, dist3.y, dist3.z);
				// printf("posRadA %f %f %f, posRadB, %f %f %f\n", posRadA.x, posRadA.y, posRadA.z, posRadB.x, posRadB.y,
				// posRadB.z);
				//}
			}
		}
	}
	densityShare += densityShare2;
	denominator += denominator2;
}
//--------------------------------------------------------------------------------------------------------------------------------
// collide a particle against all other particles in a given cell
__device__ void projectTheClosestFluidMarker(Real3& distRhoPress, int3 gridPos,
		uint index, Real3 posRadA, Real3* sortedPosRad, Real4* sortedRhoPreMu,
		uint* cellStart, uint* cellEnd) {

	//?c2 printf("grid pos %d %d %d \n", gridPos.x, gridPos.y, gridPos.z);
	uint gridHash = calcGridHash(gridPos);
	// get start of bucket for this cell
	uint startIndex = FETCH(cellStart, gridHash);
	if (startIndex != 0xffffffff) {  // cell is not empty
		// iterate over particles in this cell
		uint endIndex = FETCH(cellEnd, gridHash);

		for (uint j = startIndex; j < endIndex; j++) {
			if (j == index)
				continue;
			Real4 rhoPreMuB = FETCH(sortedRhoPreMu, j);
			if (rhoPreMuB.w > -.1)
				continue;  // we don't care about the closest non-fluid marker
			Real3 posRadB = FETCH(sortedPosRad, j);
			Real3 dist3 = Distance(posRadA, posRadB);
			Real d = length(dist3);
			if (distRhoPress.x > d) {
				distRhoPress = mR3(d, rhoPreMuB.x, rhoPreMuB.y);
			}
		}
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
// collide a particle against all other particles in a given cell
__device__ void calcOnCartesianShare(Real3& v_share, Real4& rp_share,
		int3 gridPos, Real4 gridNodePos4, Real3* sortedPosRad,
		Real3* sortedVelMas, Real4* sortedRhoPreMu, uint* cellStart,
		uint* cellEnd) {

	//?c2 printf("grid pos %d %d %d \n", gridPos.x, gridPos.y, gridPos.z);
	uint gridHash = calcGridHash(gridPos);
	// get start of bucket for this cell
	uint startIndex = FETCH(cellStart, gridHash);
	if (startIndex != 0xffffffff) {  // cell is not empty
		// iterate over particles in this cell
		uint endIndex = FETCH(cellEnd, gridHash);

		for (uint j = startIndex; j < endIndex; j++) {
			Real3 posRadB = FETCH(sortedPosRad, j);
			Real3 velMasB = FETCH(sortedVelMas, j);
			Real4 rhoPreMuB = FETCH(sortedRhoPreMu, j);
			Real3 dist3 = Distance(gridNodePos4, posRadB);
			Real d = length(dist3);
			Real mult = paramsD.markerMass / rhoPreMuB.x * W3(d);
			v_share += mult * velMasB;  // optimize it ?$
			rp_share += mult * mR4(rhoPreMuB.x, rhoPreMuB.y, 0, 0);
		}
	}
}

/**
 * @brief calcHashD
 * @details
 * 		 1. Get particle index. Determine by the block and thread we are in.
 * 		 2. From x,y,z position determine which bin it is in.
 * 		 3. Calculate hash from bin index.
 * 		 4. Store hash and particle index associated with it.
 *
 * @param gridMarkerHash
 * @param gridMarkerIndex
 * @param posRad
 * @param numAllMarkers
 */
__global__ void calcHashD(uint* gridMarkerHash,   // output
		uint* gridMarkerIndex,  // output
		Real3* posRad,          // input: positions
		Real4* rp, uint numAllMarkers) {

	/* Calculate the index of where the particle is stored in posRad. */
	uint index = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
	if (index >= numAllMarkers)
		return;

	Real3 p = posRad[index];

	if (!(isfinite(p.x) && isfinite(p.y) && isfinite(p.z))) {
		printf("Error! particle position is NAN: thrown from SDKCollisionSystem.cu, calcHashD !\n");
	}

	/* Check particle is inside the domain. */
	Real3 boxCorner = paramsD.worldOrigin;
	if (p.x < boxCorner.x || p.y < boxCorner.y || p.z < boxCorner.z) {
		printf("Out of Min Boundary\n");
		return;
	}
	boxCorner = paramsD.worldOrigin + paramsD.boxDims;
	if (p.x > boxCorner.x || p.y > boxCorner.y || p.z > boxCorner.z) {
		printf(
				"Out of max Boundary, point %f %f %f, type %f, boundary max: %f %f %f \n",
				p.x, p.y, p.z, rp->w, boxCorner.x, boxCorner.y, boxCorner.z);
		return;
	}

	/* Get x,y,z bin index in grid */
	int3 gridPos = calcGridPos(p);
	/* Calculate a hash from the bin index */
	uint hash = calcGridHash(gridPos);

	/* Store grid hash */
	gridMarkerHash[index] = hash;
	/* Store particle index associated to the hash we stored in gridMarkerHash */
	gridMarkerIndex[index] = index;
}

/**
 * @brief reorderDataAndFindCellStartD
 * @details See SDKCollisionSystem.cuh for more info
 */
__global__ void reorderDataAndFindCellStartD(uint* cellStart, // output: cell start index
		uint* cellEnd,        // output: cell end index
		Real3* sortedPosRad,  // output: sorted positions
		Real3* sortedVelMas,  // output: sorted velocities
		Real4* sortedRhoPreMu, uint* gridMarkerHash, // input: sorted grid hashes
		uint* gridMarkerIndex,      // input: sorted particle indices
		uint* mapOriginalToSorted, // mapOriginalToSorted[originalIndex] = originalIndex
		Real3* oldPosRad,           // input: sorted position array
		Real3* oldVelMas,           // input: sorted velocity array
		Real4* oldRhoPreMu, uint numAllMarkers) {
	extern __shared__ uint sharedHash[];  // blockSize + 1 elements
	/* Get the particle index the current thread is supposed to be looking at. */
	uint index = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
	uint hash;
	/* handle case when no. of particles not multiple of block size */
	if (index < numAllMarkers) {
		hash = gridMarkerHash[index];
		/* Load hash data into shared memory so that we can look at neighboring particle's hash
		 * value without loading two hash values per thread
		 */
		sharedHash[threadIdx.x + 1] = hash;

		if (index > 0 && threadIdx.x == 0) {
			/* first thread in block must load neighbor particle hash */
			sharedHash[0] = gridMarkerHash[index - 1];
		}
	}

	__syncthreads();

	if (index < numAllMarkers) {
		/* If this particle has a different cell index to the previous particle then it must be
		 * the first particle in the cell, so store the index of this particle in the cell. As it
		 * isn't the first particle, it must also be the cell end of the previous particle's cell
		 */
		if (index == 0 || hash != sharedHash[threadIdx.x]) {
			cellStart[hash] = index;
			if (index > 0)
				cellEnd[sharedHash[threadIdx.x]] = index;
		}

		if (index == numAllMarkers - 1) {
			cellEnd[hash] = index + 1;
		}

		/* Now use the sorted index to reorder the pos and vel data */
		uint originalIndex = gridMarkerIndex[index];  // map sorted to original
		mapOriginalToSorted[index] = index;	// will be sorted outside. Alternatively, you could have mapOriginalToSorted[originalIndex] = index; without need to sort. But that is not thread safe
		Real3 posRad = FETCH(oldPosRad, originalIndex); // macro does either global read or texture fetch
		Real3 velMas = FETCH(oldVelMas, originalIndex); // see particles_kernel.cuh
		Real4 rhoPreMu = FETCH(oldRhoPreMu, originalIndex);

		if (!(isfinite(posRad.x) && isfinite(posRad.y)
				&& isfinite(posRad.z))) {
			printf("Error! particle position is NAN: thrown from SDKCollisionSystem.cu, reorderDataAndFindCellStartD !\n");
		}
		if (!(isfinite(velMas.x) && isfinite(velMas.y)
				&& isfinite(velMas.z))) {
			printf("Error! particle velocity is NAN: thrown from SDKCollisionSystem.cu, reorderDataAndFindCellStartD !\n");
		}
		if (!(isfinite(rhoPreMu.x) && isfinite(rhoPreMu.y)
				&& isfinite(rhoPreMu.z) && isfinite(rhoPreMu.w))) {
			printf("Error! particle rhoPreMu is NAN: thrown from SDKCollisionSystem.cu, reorderDataAndFindCellStartD !\n");
		}
		sortedPosRad[index] = posRad;
		sortedVelMas[index] = velMas;
		sortedRhoPreMu[index] = rhoPreMu;
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void newVel_XSPH_D(Real3* vel_XSPH_Sorted_D,  // output: new velocity
		Real3* sortedPosRad,       // input: sorted positions
		Real3* sortedVelMas,       // input: sorted velocities
		Real4* sortedRhoPreMu, uint* gridMarkerIndex, // input: sorted particle indices
		uint* cellStart, uint* cellEnd, uint numAllMarkers) {
	uint index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	if (index >= numAllMarkers)
		return;

	// read particle data from sorted arrays

	Real4 rhoPreMuA = FETCH(sortedRhoPreMu, index);
	Real3 velMasA = FETCH(sortedVelMas, index);
	if (rhoPreMuA.w > -0.1) { // v_XSPH is calculated only for fluid markers. Keep unchanged if not fluid.
		vel_XSPH_Sorted_D[index] = velMasA;
		return;
	}

	Real3 posRadA = FETCH(sortedPosRad, index);
	Real3 deltaV = mR3(0);

	// get address in grid
	int3 gridPos = calcGridPos(posRadA);

	/// if (gridPos.x == paramsD.gridSize.x-1) printf("****aha %d %d\n", gridPos.x, paramsD.gridSize.x);

	// examine neighbouring cells
	for (int z = -1; z <= 1; z++) {
		for (int y = -1; y <= 1; y++) {
			for (int x = -1; x <= 1; x++) {
				int3 neighbourPos = gridPos + mI3(x, y, z);
				deltaV += deltaVShare(neighbourPos, index, posRadA, velMasA,
						rhoPreMuA, sortedPosRad, sortedVelMas, sortedRhoPreMu,
						cellStart, cellEnd);
			}
		}
	}
	//   // write new velocity back to original unsorted location
	// sortedVel_XSPH[index] = velMasA + paramsD.EPS_XSPH * deltaV;

	// write new velocity back to original unsorted location
	// uint originalIndex = gridMarkerIndex[index];
	Real3 vXSPH = velMasA + paramsD.EPS_XSPH * deltaV;
	if (!(isfinite(vXSPH.x) && isfinite(vXSPH.y)
			&& isfinite(vXSPH.z))) {
		printf("Error! particle vXSPH is NAN: thrown from SDKCollisionSystem.cu, newVel_XSPH_D !\n");
	}
	vel_XSPH_Sorted_D[index] = vXSPH;
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void new_BCE_VelocityPressure(
		Real3* sortedVelMas_ModifiedBCE,    // input: sorted velocities
		Real4* sortedRhoPreMu_ModifiedBCE,  // input: sorted velocities
		Real3* sortedPosRad,                // input: sorted positions
		Real3* sortedVelMas,                // input: sorted velocities
		Real4* sortedRhoPreMu, uint* cellStart, uint* cellEnd,
		uint numAllMarkers) {
	uint index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	if (index >= numAllMarkers)
		return;
	Real4 rhoPreMuA = FETCH(sortedRhoPreMu, index);
	if (rhoPreMuA.w < -0.1) {  // keep unchanged if fluid
		return;
	}

	// read particle data from sorted arrays
	Real3 posRadA = FETCH(sortedPosRad, index);
	Real3 velMasA = FETCH(sortedVelMas, index);
	int isAffected = 0;

	Real4 deltaVDenom = mR4(0);
	Real4 deltaRP = mR4(0);

	// get address in grid
	int3 gridPos = calcGridPos(posRadA);

	/// if (gridPos.x == paramsD.gridSize.x-1) printf("****aha %d %d\n", gridPos.x, paramsD.gridSize.x);

	// examine neighbouring cells
	for (int z = -1; z <= 1; z++) {
		for (int y = -1; y <= 1; y++) {
			for (int x = -1; x <= 1; x++) {
				int3 neighbourPos = gridPos + mI3(x, y, z);
				BCE_modification_Share(deltaVDenom, deltaRP, isAffected,
						neighbourPos, index, posRadA, sortedPosRad,
						sortedVelMas, sortedRhoPreMu, cellStart, cellEnd);
			}
		}
	}

	if (isAffected) {
		Real3 modifiedBCE_v = 2 * velMasA - mR3(deltaVDenom) / deltaVDenom.w;
		sortedVelMas_ModifiedBCE[index] = modifiedBCE_v;

		Real pressure = (deltaRP.w + dot(paramsD.gravity, mR3(deltaRP)))
				/ deltaVDenom.w;  //(in fact:  (paramsD.gravity -
		// aW), but aW for moving rigids
		// is hard to calc. Assume aW is
		// zero for now
		Real density = InvEos(pressure);
		sortedRhoPreMu_ModifiedBCE[index] = mR4(density, pressure, rhoPreMuA.z,
				rhoPreMuA.w);
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void collideD(Real4* sortedDerivVelRho_fsi_D,  // output: new velocity
		Real3* sortedPosRad,  // input: sorted positions
		Real3* sortedVelMas,  // input: sorted velocities
		Real3* vel_XSPH_Sorted_D, Real4* sortedRhoPreMu,
		uint* cellStart, uint* cellEnd, uint numAllMarkers) {

	uint index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	if (index >= numAllMarkers)
		return;

	// read particle data from sorted arrays
	Real3 posRadA = FETCH(sortedPosRad, index);
	Real3 velMasA = FETCH(sortedVelMas, index);
	Real4 rhoPreMuA = FETCH(sortedRhoPreMu, index);

//	uint originalIndex = gridMarkerIndex[index];
	Real3 vel_XSPH_A = FETCH(vel_XSPH_Sorted_D, index);

	Real4 derivVelRho = sortedDerivVelRho_fsi_D[index];

	// get address in grid
	int3 gridPos = calcGridPos(posRadA);

	// examine neighbouring cells
	for (int x = -1; x <= 1; x++) {
		for (int y = -1; y <= 1; y++) {
			for (int z = -1; z <= 1; z++) {
				derivVelRho += collideCell(gridPos + mI3(x, y, z), index,
						posRadA, velMasA, vel_XSPH_A, rhoPreMuA, sortedPosRad,
						sortedVelMas, vel_XSPH_Sorted_D, sortedRhoPreMu,
						cellStart, cellEnd);
			}
		}
	}

	// write new velocity back to original unsorted location
	// *** let's tweak a little bit :)
	if (!(isfinite(derivVelRho.x) && isfinite(derivVelRho.y)
			&& isfinite(derivVelRho.z) && isfinite(derivVelRho.w))) {
		printf("Error! particle derivVelRho is NAN: thrown from SDKCollisionSystem.cu, collideD !\n");
	}
	sortedDerivVelRho_fsi_D[index] = derivVelRho;
}
//--------------------------------------------------------------------------------------------------------------------------------
// calculate particles stresses
__global__ void CalcBCE_Stresses_kernel(Real3* devStressD, Real3* volStressD,
		Real3* sortedPosRad, Real3* sortedVelMas, Real4* sortedRhoPreMu,
		uint* mapOriginalToSorted, uint* cellStart, uint* cellEnd, int numBCE) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numBCE) {
		return;
	}
	// Arman take care of this
	uint BCE_Index = index
			+ min(numObjectsD.startRigidMarkers, numObjectsD.startRigidMarkers); // updatePortion = [start, end] index of the update portion
	uint originalIndex = mapOriginalToSorted[BCE_Index]; // index in the sorted array

	// read particle data from sorted arrays
	Real3 posRadA = FETCH(sortedPosRad, originalIndex);
	Real3 velMasA = FETCH(sortedVelMas, originalIndex);
	Real4 rhoPreMuA = FETCH(sortedRhoPreMu, originalIndex);

	// get address in grid
	int3 gridPos = calcGridPos(posRadA);

	Real3 devS3 = mR3(0);
	Real3 volS3 = mR3(0);

	// examine neighbouring cells
	for (int x = -1; x <= 1; x++) {
		for (int y = -1; y <= 1; y++) {
			for (int z = -1; z <= 1; z++) {
				stressCell(devS3, volS3, gridPos + mI3(x, y, z), originalIndex,
						posRadA, velMasA, rhoPreMuA, sortedPosRad, sortedVelMas,
						sortedRhoPreMu, cellStart, cellEnd);
			}
		}
	}

	devStressD[index] = devS3;
	volStressD[index] = volS3;
}
//--------------------------------------------------------------------------------------------------------------------------------
// calculate particles stresses
__global__ void CalcBCE_MainStresses_kernel(Real4* mainStressD,
		Real3* devStressD, Real3* volStressD, int numBCE) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numBCE) {
		return;
	}

	Real3 devS3 = devStressD[index];
	Real3 volS3 = volStressD[index];
	Real4 mainS3 = mR4(0);
	mainS3.w = sqrt(
			.5
					* (pow(volS3.x - volS3.y, Real(2))
							+ pow(volS3.x - volS3.z, Real(2))
							+ pow(volS3.y - volS3.z, Real(2))
							+ 6
									* (devS3.x * devS3.x + devS3.y * devS3.y
											+ devS3.z * devS3.z)));

	mainStressD[index] = mainS3;
}
//--------------------------------------------------------------------------------------------------------------------------------
// without normalization
__global__ void ReCalcDensityD_F1(Real3* oldPosRad, Real3* oldVelMas,
		Real4* oldRhoPreMu, Real3* sortedPosRad, Real3* sortedVelMas,
		Real4* sortedRhoPreMu, uint* gridMarkerIndex, uint* cellStart,
		uint* cellEnd, uint numAllMarkers) {
	uint index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	if (index >= numAllMarkers)
		return;

	// read particle data from sorted arrays
	Real3 posRadA = FETCH(sortedPosRad, index);
	Real4 rhoPreMuA = FETCH(sortedRhoPreMu, index);

	if (rhoPreMuA.w > -.1)
		return;

	// get address in grid
	int3 gridPos = calcGridPos(posRadA);

	Real densityShare = 0.0f;
	Real denominator = 0.0f;
	// examine neighbouring cells
	for (int z = -1; z <= 1; z++) {
		for (int y = -1; y <= 1; y++) {
			for (int x = -1; x <= 1; x++) {
				int3 neighbourPos = gridPos + mI3(x, y, z);
				collideCellDensityReInit(densityShare, denominator,
						neighbourPos, index, posRadA, sortedPosRad,
						sortedVelMas, sortedRhoPreMu, cellStart, cellEnd);
			}
		}
	}
	// write new velocity back to original unsorted location
	uint originalIndex = gridMarkerIndex[index];

	Real newDensity = densityShare + paramsD.markerMass * W3(0); //?$ include the particle in its summation as well
	Real newDenominator = denominator
			+ paramsD.markerMass * W3(0) / rhoPreMuA.x;
	if (rhoPreMuA.w < 0) {
		//		rhoPreMuA.x = newDensity; // old version
		rhoPreMuA.x = newDensity / newDenominator;  // correct version
	}
	rhoPreMuA.y = Eos(rhoPreMuA.x, rhoPreMuA.w);
	oldRhoPreMu[originalIndex] = rhoPreMuA;
}
//--------------------------------------------------------------------------------------------------------------------------------
// without normalization
__global__ void ProjectDensityPressureToBCandBCE_D(Real4* oldRhoPreMu,
		Real3* sortedPosRad, Real4* sortedRhoPreMu, uint* gridMarkerIndex,
		uint* cellStart, uint* cellEnd, uint numAllMarkers) {
	uint index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	if (index >= numAllMarkers)
		return;

	// read particle data from sorted arrays
	Real3 posRadA = FETCH(sortedPosRad, index);
	Real4 rhoPreMuA = FETCH(sortedRhoPreMu, index);

	if (rhoPreMuA.w < -.1)
		return;

	// get address in grid
	int3 gridPos = calcGridPos(posRadA);

	Real3 distRhoPress =
	mR3((RESOLUTION_LENGTH_MULT + 2) * paramsD.HSML, rhoPreMuA.x, rhoPreMuA.y); //(large distance, rhoA, pA)
	// examine neighbouring cells
	for (int z = -1; z <= 1; z++) {
		for (int y = -1; y <= 1; y++) {
			for (int x = -1; x <= 1; x++) {
				int3 neighbourPos = gridPos + mI3(x, y, z);
				projectTheClosestFluidMarker(distRhoPress, neighbourPos, index,
						posRadA, sortedPosRad, sortedRhoPreMu, cellStart,
						cellEnd);
			}
		}
	}
	// write new velocity back to original unsorted location
	uint originalIndex = gridMarkerIndex[index];
	rhoPreMuA.x = distRhoPress.y;
	rhoPreMuA.y = distRhoPress.z;
	oldRhoPreMu[originalIndex] = rhoPreMuA;
}
//--------------------------------------------------------------------------------------------------------------------------------
// without normalization
__global__ void CalcCartesianDataD(Real4* rho_Pres_CartD,
		Real4* vel_VelMag_CartD, Real3* sortedPosRad, Real3* sortedVelMas,
		Real4* sortedRhoPreMu, uint* gridMarkerIndex, uint* cellStart,
		uint* cellEnd, int3 cartesianGridDims, Real resolution) {
	uint index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	if (index
			>= cartesianGridDims.x * cartesianGridDims.y * cartesianGridDims.z)
		return;

	int3 gridLoc;
	gridLoc.z = index / (cartesianGridDims.x * cartesianGridDims.y);
	gridLoc.y = (index % (cartesianGridDims.x * cartesianGridDims.y))
			/ cartesianGridDims.x;
	gridLoc.x = (index % (cartesianGridDims.x * cartesianGridDims.y))
			% cartesianGridDims.x;
	// alias cartesianGridDims = Dim,  you can say:   "index = (Dim.x * Dim.y) * gridLoc.z + Dim.x * gridLoc.y +
	// gridLoc.x"

	// get address in grid
	Real3 gridNodePos3 = mR3(gridLoc) * resolution + paramsD.worldOrigin;
	int3 gridPos = calcGridPos(gridNodePos3);

	Real3 vel_share = mR3(0.0f);
	Real4 rho_pres_share = mR4(0.0f);
	// examine neighbouring cells
	for (int z = -1; z <= 1; z++) {
		for (int y = -1; y <= 1; y++) {
			for (int x = -1; x <= 1; x++) {
				int3 neighbourPos = gridPos + mI3(x, y, z);
				calcOnCartesianShare(vel_share, rho_pres_share, neighbourPos,
				mR4(gridNodePos3), sortedPosRad, sortedVelMas, sortedRhoPreMu,
						cellStart, cellEnd);
			}
		}
	}
	// write new velocity back to original unsorted location
	//  uint originalIndex = gridMarkerIndex[index];

	// Real newDensity = densityShare + paramsD.markerMass * W3(0); //?$ include the particle in its summation as well
	// if (rhoPreMuA.w < -.1) { rhoPreMuA.x = newDensity; }
	// rhoPreMuA.y = Eos(rhoPreMuA.x, rhoPreMuA.w);
	//   oldRhoPreMu[originalIndex] = rhoPreMuA;
	/////printf("density %f\n", rhoPreMuA.x);
	/////printf("densityshare %f\n", densityShare);
	/////printf("gridPos x y z %d %d %d %f\n", gridPos.x, gridPos.y, gridPos.z, densityShare);
	rho_Pres_CartD[index] = rho_pres_share;
	vel_VelMag_CartD[index] = mR4(vel_share, length(vel_share));
}

//%%%%%%%%%%%%%%%%%%%%%%%%
//--------------------------------------------------------------------------------------------------------------------------------
//--------------------------------------------------------------------------------------------------------------------------------
// updates the fluid particles' properties, i.e. velocity, density, pressure, position
__global__ void UpdateFluidD(Real3* posRadD, Real3* velMasD, Real3* vel_XSPH_D,
		Real4* rhoPresMuD, Real4* derivVelRhoD, int2 updatePortion, Real dT) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	index += updatePortion.x; // updatePortion = [start, end] index of the update portion
	if (index >= updatePortion.y) {
		return;
	}
	Real4 derivVelRho = derivVelRhoD[index];
	Real4 rhoPresMu = rhoPresMuD[index];

	if (rhoPresMu.w < 0) {
		Real3 vel_XSPH = vel_XSPH_D[index];
		// 0** if you have rigid BCE, make sure to apply same tweaks to them, to satify action/reaction. Or apply tweak to
		// force in advance
		// 1*** let's tweak a little bit :)
		if (length(vel_XSPH) > paramsD.tweakMultV * paramsD.HSML / paramsD.dT
				&& paramsD.enableTweak) {
			vel_XSPH *= (paramsD.tweakMultV * paramsD.HSML / paramsD.dT)
					/ length(vel_XSPH);
			if (length(vel_XSPH)
					> 1.001 * paramsD.tweakMultV * paramsD.HSML / paramsD.dT) { // infinity
				if (paramsD.enableAggressiveTweak) {
					vel_XSPH = mR3(0);
				} else {
					printf("Error! Infinite vel_XSPH detected!\n");
				}
			}
		}
		// 1*** end tweak

		Real3 posRad = posRadD[index];
		Real3 updatedPositon = posRad + vel_XSPH * dT;
		posRadD[index] = updatedPositon;  // posRadD updated

		Real3 velMas = velMasD[index];
		Real3 updatedVelocity = velMas + mR3(derivVelRho) * dT;
		// 2*** let's tweak a little bit :)
		if (length(updatedVelocity)
				> paramsD.tweakMultV * paramsD.HSML / paramsD.dT
				&& paramsD.enableTweak) {
			updatedVelocity *= (paramsD.tweakMultV * paramsD.HSML / paramsD.dT)
					/ length(updatedVelocity);
			if (length(updatedVelocity)
					> 1.001 * paramsD.tweakMultV * paramsD.HSML / paramsD.dT) { // infinity
				if (paramsD.enableAggressiveTweak) {
					updatedVelocity = mR3(0);
				} else {
					printf("Error! Infinite updatedVelocity detected!\n");
				}
			}
		}
		// 2*** end tweak
		velMasD[index] = updatedVelocity;

	}
	// 3*** let's tweak a little bit :)
	if (fabs(derivVelRho.w) > paramsD.tweakMultRho * paramsD.rho0 / paramsD.dT
			&& paramsD.enableTweak) {
		derivVelRho.w *= (paramsD.tweakMultRho * paramsD.rho0 / paramsD.dT)
				/ fabs(derivVelRho.w);  // to take care of the sign as well
		if (fabs(derivVelRho.w)
				> 1.001 * paramsD.tweakMultRho * paramsD.rho0 / paramsD.dT) {
			if (paramsD.enableAggressiveTweak) {
				derivVelRho.w = 0;
			} else {
				printf("Error! Infinite derivRho detected!\n");
			}
		}
	}
	// 2*** end tweak
	Real rho2 = rhoPresMu.x + derivVelRho.w * dT; // rho update. (i.e. rhoPresMu.x), still not wriiten to global matrix
	rhoPresMu.y = Eos(rho2, rhoPresMu.w);
	rhoPresMu.x = rho2;
	rhoPresMuD[index] = rhoPresMu;  // rhoPresMuD updated
}
//--------------------------------------------------------------------------------------------------------------------------------
// updates the fluid particles' properties, i.e. velocity, density, pressure, position
__global__ void UpdateFluidD_init_LF(Real3* posRadD, Real3* velMasD_half,
		Real4* rhoPresMuD_half, Real4* derivVelRhoD, int2 updatePortion,
		Real dT) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	index += updatePortion.x; // updatePortion = [start, end] index of the update portion
	if (index >= updatePortion.y) {
		return;
	}

	Real4 derivVelRho = derivVelRhoD[index];
	Real3 velMas = velMasD_half[index];
	Real3 updatedVelocity = velMas + mR3(derivVelRho) * (0.5 * dT);
	velMasD_half[index] = updatedVelocity;  // velMasD_half updated

	Real3 posRad = posRadD[index];
	Real3 updatedPositon = posRad + updatedVelocity * dT;
	posRadD[index] = updatedPositon;  // posRadD updated

	Real4 rhoPresMu = rhoPresMuD_half[index];
	Real rho2 = rhoPresMu.x + derivVelRho.w * (0.5 * dT); // rho update. (i.e. rhoPresMu.x), still not wriiten to global matrix
	rhoPresMu.y = Eos(rho2, rhoPresMu.w);
	rhoPresMu.x = rho2;
	rhoPresMuD_half[index] = rhoPresMu;  // rhoPresMuD_half updated
}
//--------------------------------------------------------------------------------------------------------------------------------
// updates the fluid particles' properties, i.e. velocity, density, pressure, position
__global__ void UpdateFluidD_rho_vel_LF(Real3* velMasD, Real4* rhoPresMuD,
		Real3* velMasD_old, Real4* rhoPresMuD_old, Real4* derivVelRhoD,
		int2 updatePortion, Real dT) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	index += updatePortion.x; // updatePortion = [start, end] index of the update portion
	if (index >= updatePortion.y) {
		return;
	}

	Real4 derivVelRho = derivVelRhoD[index];
	Real3 velMas = velMasD_old[index];
	Real3 updatedVelocity = velMas + mR3(derivVelRho) * dT;
	// 2*** let's tweak a little bit :)
	//	if (length(updatedVelocity) > .1 * paramsD.HSML / dT  && paramsD.enableTweak) {
	//		updatedVelocity *= ( .1 * paramsD.HSML / dT ) / length(updatedVelocity);
	//		if (length(updatedVelocity) > 1.001) { // infinity
	//			if (paramsD.enableAggressiveTweak) {
	//				updatedVelocity = mR3(0);
	//			} else {
	//				printf("Error! Infinite updatedVelocity detected!\n");
	//			}
	//		}
	//	}
	// 2*** end tweak
	velMasD[index] = updatedVelocity; // velMasD_half updated

	Real4 rhoPresMu = rhoPresMuD_old[index];

	// 3*** let's tweak a little bit :)
	//	if (fabs(derivVelRho.w) > .002 * paramsD.rho0 / dT  && paramsD.enableTweak) {
	//		derivVelRho.w *= (.002 * paramsD.rho0 / dT) / fabs(derivVelRho.w); //to take care of the sign as well
	//		if (fabs(derivVelRho.w) > 00201 * paramsD.rho0 / dT) {
	//			if (paramsD.enableAggressiveTweak) {
	//				derivVelRho.w = 0;
	//			} else {
	//				printf("Error! Infinite derivRho detected!\n");
	//			}
	//		}
	//	}
	// 2*** end tweak
	Real rho2 = rhoPresMu.x + derivVelRho.w * dT; // rho update. (i.e. rhoPresMu.x), still not wriiten to global matrix
	rhoPresMu.y = Eos(rho2, rhoPresMu.w);
	rhoPresMu.x = rho2;
	rhoPresMuD[index] = rhoPresMu;  // rhoPresMuD_half updated
}
//--------------------------------------------------------------------------------------------------------------------------------
// updates the fluid particles' properties, i.e. velocity, density, pressure, position
__global__ void UpdateFluidD_EveryThing_LF(Real3* posRadD, Real3* velMasD_half,
		Real4* rhoPresMuD_half, Real4* derivVelRhoD, int2 updatePortion,
		Real dT) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	index += updatePortion.x; // updatePortion = [start, end] index of the update portion
	if (index >= updatePortion.y) {
		return;
	}

	Real4 derivVelRho = derivVelRhoD[index];
	Real3 velMas = velMasD_half[index];
	Real3 updatedVelocity = velMas + mR3(derivVelRho) * dT;
	// 2*** let's tweak a little bit :)
	//	if (length(updatedVelocity) > .1 * paramsD.HSML / dT  && paramsD.enableTweak) {
	//		updatedVelocity *= ( .1 * paramsD.HSML / dT ) / length(updatedVelocity);
	//		if (length(updatedVelocity) > 1.001) { // infinity
	//			if (paramsD.enableAggressiveTweak) {
	//				updatedVelocity = mR3(0);
	//			} else {
	//				printf("Error! Infinite updatedVelocity detected!\n");
	//			}
	//		}
	//	}
	// 2*** end tweak
	velMasD_half[index] = updatedVelocity;  // velMasD_half updated

	posRadD[index] += updatedVelocity * dT;  // posRadD updated

	Real4 rhoPresMu = rhoPresMuD_half[index];

	// 3*** let's tweak a little bit :)
	//	if (fabs(derivVelRho.w) > .002 * paramsD.rho0 / dT  && paramsD.enableTweak) {
	//		derivVelRho.w *= (.002 * paramsD.rho0 / dT) / fabs(derivVelRho.w); //to take care of the sign as well
	//		if (fabs(derivVelRho.w) > 00201 * paramsD.rho0 / dT) {
	//			if (paramsD.enableAggressiveTweak) {
	//				derivVelRho.w = 0;
	//			} else {
	//				printf("Error! Infinite derivRho detected!\n");
	//			}
	//		}
	//	}
	// 2*** end tweak
	Real rho2 = rhoPresMu.x + derivVelRho.w * dT; // rho update. (i.e. rhoPresMu.x), still not wriiten to global matrix
	rhoPresMu.y = Eos(rho2, rhoPresMu.w);
	rhoPresMu.x = rho2;
	rhoPresMuD_half[index] = rhoPresMu;  // rhoPresMuD_half updated
}

/**
 * @brief Copies the sortedVelXSPH to velXSPH according to indexing
 * @details [long description]
 *
 * @param vel_XSPH_D
 * @param vel_XSPH_Sorted_D Pointer to new sorted vel_XSPH vector
 * @param m_dGridMarkerIndex List of indeces used to sort vel_XSPH_D
 */

__global__ void CopySorted_vXSPH_dVdRho_to_original_kernel(Real3* vel_XSPH_D,
		Real4* derivVelRhoD,
		Real3* vel_XSPH_Sorted_D, Real4* sortedDerivVelRho_fsi_D,
		uint* mapOriginalToSorted) {
	uint index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	if (index >= numObjectsD.numAllMarkers)
		return;
	vel_XSPH_D[index] = vel_XSPH_Sorted_D[mapOriginalToSorted[index]];
	derivVelRhoD[index] = sortedDerivVelRho_fsi_D[mapOriginalToSorted[index]];
}

//--------------------------------------------------------------------------------------------------------------------------------
// updates the fluid particles' properties, i.e. velocity, density, pressure, position
__global__ void UpdateKernelBoundary(Real3* posRadD, Real3* velMasD,
		Real4* rhoPresMuD, Real4* derivVelRhoD, int2 updatePortion, Real dT) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	index += updatePortion.x; // updatePortion = [start, end] index of the update portion
	if (index >= updatePortion.y) {
		return;
	}

	Real4 derivVelRho = derivVelRhoD[index];
	Real4 rhoPresMu = rhoPresMuD[index];
	Real rho2 = rhoPresMu.x + derivVelRho.w * dT; // rho update. (i.e. rhoPresMu.x), still not wriiten to global matrix
	rhoPresMu.y = Eos(rho2, rhoPresMu.w);
	rhoPresMu.x = rho2;
	rhoPresMuD[index] = rhoPresMu;  // rhoPresMuD updated
}

//--------------------------------------------------------------------------------------------------------------------------------
// applies periodic BC along x
__global__ void ApplyPeriodicBoundaryXKernel(Real3* posRadD,
		Real4* rhoPresMuD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numObjectsD.numAllMarkers) {
		return;
	}
	Real4 rhoPresMu = rhoPresMuD[index];
	if (fabs(rhoPresMu.w) < .1) {
		return;
	}  // no need to do anything if it is a boundary particle
	Real3 posRad = posRadD[index];
	if (posRad.x > paramsD.cMax.x) {
		posRad.x -= (paramsD.cMax.x - paramsD.cMin.x);
		posRadD[index] = posRad;
		if (rhoPresMu.w < -.1) {
			rhoPresMu.y = rhoPresMu.y + paramsD.deltaPress.x;
			rhoPresMuD[index] = rhoPresMu;
		}
		return;
	}
	if (posRad.x < paramsD.cMin.x) {
		posRad.x += (paramsD.cMax.x - paramsD.cMin.x);
		posRadD[index] = posRad;
		if (rhoPresMu.w < -.1) {
			rhoPresMu.y = rhoPresMu.y - paramsD.deltaPress.x;
			rhoPresMuD[index] = rhoPresMu;
		}
		return;
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
// applies periodic BC along y
__global__ void ApplyPeriodicBoundaryYKernel(Real3* posRadD,
		Real4* rhoPresMuD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numObjectsD.numAllMarkers) {
		return;
	}
	Real4 rhoPresMu = rhoPresMuD[index];
	if (fabs(rhoPresMu.w) < .1) {
		return;
	}  // no need to do anything if it is a boundary particle
	Real3 posRad = posRadD[index];
	if (posRad.y > paramsD.cMax.y) {
		posRad.y -= (paramsD.cMax.y - paramsD.cMin.y);
		posRadD[index] = posRad;
		if (rhoPresMu.w < -.1) {
			rhoPresMu.y = rhoPresMu.y + paramsD.deltaPress.y;
			rhoPresMuD[index] = rhoPresMu;
		}
		return;
	}
	if (posRad.y < paramsD.cMin.y) {
		posRad.y += (paramsD.cMax.y - paramsD.cMin.y);
		posRadD[index] = posRad;
		if (rhoPresMu.w < -.1) {
			rhoPresMu.y = rhoPresMu.y - paramsD.deltaPress.y;
			rhoPresMuD[index] = rhoPresMu;
		}
		return;
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
// applies periodic BC along z
__global__ void ApplyPeriodicBoundaryZKernel(Real3* posRadD,
		Real4* rhoPresMuD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numObjectsD.numAllMarkers) {
		return;
	}
	Real4 rhoPresMu = rhoPresMuD[index];
	if (fabs(rhoPresMu.w) < .1) {
		return;
	}  // no need to do anything if it is a boundary particle
	Real3 posRad = posRadD[index];
	if (posRad.z > paramsD.cMax.z) {
		posRad.z -= (paramsD.cMax.z - paramsD.cMin.z);
		posRadD[index] = posRad;
		if (rhoPresMu.w < -.1) {
			rhoPresMu.y = rhoPresMu.y + paramsD.deltaPress.z;
			rhoPresMuD[index] = rhoPresMu;
		}
		return;
	}
	if (posRad.z < paramsD.cMin.z) {
		posRad.z += (paramsD.cMax.z - paramsD.cMin.z);
		posRadD[index] = posRad;
		if (rhoPresMu.w < -.1) {
			rhoPresMu.y = rhoPresMu.y - paramsD.deltaPress.z;
			rhoPresMuD[index] = rhoPresMu;
		}
		return;
	}
}

//%%%%%%%%%%%%%%%%%%%%%%%%
//--------------------------------------------------------------------------------------------------------------------------------
void allocateArray(void** devPtr, size_t size) {
	hipMalloc(devPtr, size);
}
//--------------------------------------------------------------------------------------------------------------------------------
void freeArray(void* devPtr) {
	hipFree(devPtr);
}

/**
 * @brief iDivUp
 * @details Round a / b to nearest higher integer value
 *
 * @param a numerator
 * @param b denominator
 *
 * @return ceil(a/b)
 */
uint iDivUp(uint a, uint b) {
	return (a % b != 0) ? (a / b + 1) : (a / b);
}

/**
 * @brief computeGridSize
 * @details Compute grid and thread block size for a given number of elements
 *
 * @param n Total number of elements. Each elements needs a thread to be computed
 * @param blockSize Number of threads per block.
 * @param numBlocks output
 * @param numThreads Output: number of threads per block
 */
void computeGridSize(uint n, uint blockSize, uint& numBlocks,
		uint& numThreads) {
	uint n2 = (n == 0) ? 1 : n;
	numThreads = min(blockSize, n2);
	numBlocks = iDivUp(n2, numThreads);
}

/**
 * @brief [brief description]
 * @details [long description]
 *
 * @param hostParams [description]
 * @param numObjects [description]
 */
void setParameters(SimParams* hostParams, NumberOfObjects* numObjects) {
	// copy parameters to constant memory
	hipMemcpyToSymbolAsync(HIP_SYMBOL(paramsD), hostParams, sizeof(SimParams));
	hipMemcpyToSymbolAsync(HIP_SYMBOL(numObjectsD), numObjects, sizeof(NumberOfObjects));
}

/**
 * @brief Wrapper function for calcHashD
 * @details See SDKCollisionSystem.cuh for more info
 */
void calcHash(thrust::device_vector<uint>& gridMarkerHash,
		thrust::device_vector<uint>& gridMarkerIndex,
		thrust::device_vector<Real3>& posRad,
		thrust::device_vector<Real4>& rhoPreMu, int numAllMarkers) {
	/* Is there a need to optimize the number of threads used at once? */
	uint numThreads, numBlocks;

	computeGridSize(numAllMarkers, 256, numBlocks, numThreads);

	/* Execute Kernel */
	calcHashD<<<numBlocks, numThreads>>>(U1CAST(gridMarkerHash),
			U1CAST(gridMarkerIndex), mR3CAST(posRad), mR4CAST(rhoPreMu),
			numAllMarkers);

	/* Check for errors in kernel execution */
	hipDeviceSynchronize();
	cudaCheckError()
	;
}

/**
 * @brief Wrapper function for reorderDataAndFindCellStartD
 * @details
 * 		See SDKCollisionSystem.cuh for brief.
 */
void reorderDataAndFindCellStart(thrust::device_vector<uint>& cellStart,
		thrust::device_vector<uint>& cellEnd,
		thrust::device_vector<Real3>& sortedPosRad,
		thrust::device_vector<Real3>& sortedVelMas,
		thrust::device_vector<Real4>& sortedRhoPreMu,

		thrust::device_vector<uint>& gridMarkerHash,
		thrust::device_vector<uint>& gridMarkerIndex,

		thrust::device_vector<uint>& mapOriginalToSorted,

		thrust::device_vector<Real3>& oldPosRad,
		thrust::device_vector<Real3>& oldVelMas,
		thrust::device_vector<Real4>& oldRhoPreMu, uint numAllMarkers,
		uint numCells) {
	uint numThreads, numBlocks;
	computeGridSize(numAllMarkers, 256, numBlocks, numThreads); //?$ 256 is blockSize

	/* Set all cells to empty */
	hipMemset(U1CAST(cellStart), 0xffffffff, numCells * sizeof(uint));

	//#if USE_TEX
	//#if 0
	//    cutilSafeCall(hipBindTexture(0, oldPosTex, oldPosRad, numAllMarkers*sizeof(Real4)));
	//    cutilSafeCall(hipBindTexture(0, oldVelTex, oldVelMas, numAllMarkers*sizeof(Real4)));
	//#endif

	uint smemSize = sizeof(uint) * (numThreads + 1);
	reorderDataAndFindCellStartD<<<numBlocks, numThreads, smemSize>>>(
			U1CAST(cellStart), U1CAST(cellEnd), mR3CAST(sortedPosRad),
			mR3CAST(sortedVelMas), mR4CAST(sortedRhoPreMu),
			U1CAST(gridMarkerHash), U1CAST(gridMarkerIndex),
			U1CAST(mapOriginalToSorted), mR3CAST(oldPosRad), mR3CAST(oldVelMas),
			mR4CAST(oldRhoPreMu), numAllMarkers);
	hipDeviceSynchronize();
	cudaCheckError()
	;

	// unroll sorted index to have the location of original particles in the sorted arrays
	thrust::device_vector<uint> dummyIndex = gridMarkerIndex;
	thrust::sort_by_key(dummyIndex.begin(), dummyIndex.end(),
			mapOriginalToSorted.begin());
	dummyIndex.clear();
	//#if USE_TEX
	//#if 0
	//    cutilSafeCall(hipUnbindTexture(oldPosTex));
	//    cutilSafeCall(hipUnbindTexture(oldVelTex));
	//#endif
}

/**
 * @brief Wrapper function for newVel_XSPH_D
 */
void RecalcVelocity_XSPH(thrust::device_vector<Real3>& vel_XSPH_Sorted_D,
		thrust::device_vector<Real3>& sortedPosRad,
		thrust::device_vector<Real3>& sortedVelMas,
		thrust::device_vector<Real4>& sortedRhoPreMu,
		thrust::device_vector<uint>& gridMarkerIndex,
		thrust::device_vector<uint>& cellStart,
		thrust::device_vector<uint>& cellEnd, uint numAllMarkers,
		uint numCells) {
	/* thread per particle */
	uint numThreads, numBlocks;
	computeGridSize(numAllMarkers, 64, numBlocks, numThreads);

	/* Execute the kernel */
	newVel_XSPH_D<<<numBlocks, numThreads>>>(mR3CAST(vel_XSPH_Sorted_D),
			mR3CAST(sortedPosRad), mR3CAST(sortedVelMas),
			mR4CAST(sortedRhoPreMu), U1CAST(gridMarkerIndex), U1CAST(cellStart),
			U1CAST(cellEnd), numAllMarkers);

	hipDeviceSynchronize();
	cudaCheckError()
	;
}
//--------------------------------------------------------------------------------------------------------------------------------
void RecalcSortedVelocityPressure_BCE(
		thrust::device_vector<Real3>& sortedVelMas,
		thrust::device_vector<Real4>& sortedRhoPreMu,
		thrust::device_vector<Real3>& sortedPosRad,
		thrust::device_vector<uint>& cellStart,
		thrust::device_vector<uint>& cellEnd, uint numAllMarkers) {
	// thread per particle
	uint numThreads, numBlocks;
	computeGridSize(numAllMarkers, 64, numBlocks, numThreads);

	// Arman modified BCE velocity version
	thrust::device_vector<Real3> sortedVelMas_ModifiedBCE = sortedVelMas;
	thrust::device_vector<Real4> sortedRhoPreMu_ModifiedBCE = sortedRhoPreMu;

	new_BCE_VelocityPressure<<<numBlocks, numThreads>>>(
			mR3CAST(sortedVelMas_ModifiedBCE),
			mR4CAST(sortedRhoPreMu_ModifiedBCE),  // input: sorted velocities
			mR3CAST(sortedPosRad), mR3CAST(sortedVelMas),
			mR4CAST(sortedRhoPreMu), U1CAST(cellStart), U1CAST(cellEnd),
			numAllMarkers);
	hipDeviceSynchronize();
	cudaCheckError()
	;

	thrust::copy(sortedVelMas_ModifiedBCE.begin(),
			sortedVelMas_ModifiedBCE.end(), sortedVelMas.begin());
	thrust::copy(sortedRhoPreMu_ModifiedBCE.begin(),
			sortedRhoPreMu_ModifiedBCE.end(), sortedRhoPreMu.begin());

	sortedVelMas_ModifiedBCE.clear();
	sortedRhoPreMu_ModifiedBCE.clear();
}
//--------------------------------------------------------------------------------------------------------------------------------
void CalcBCE_Stresses(thrust::device_vector<Real3>& devStressD,
		thrust::device_vector<Real3>& volStressD,
		thrust::device_vector<Real4>& mainStressD,
		thrust::device_vector<Real3>& sortedPosRad,
		thrust::device_vector<Real3>& sortedVelMas,
		thrust::device_vector<Real4>& sortedRhoPreMu,
		thrust::device_vector<uint>& mapOriginalToSorted,
		thrust::device_vector<uint>& cellStart,
		thrust::device_vector<uint>& cellEnd, int numBCE) {
	// thread per particle
	uint numThreads, numBlocks;
	computeGridSize(numBCE, 128, numBlocks, numThreads);
	CalcBCE_Stresses_kernel<<<numBlocks, numThreads>>>(mR3CAST(devStressD),
			mR3CAST(volStressD), mR3CAST(sortedPosRad), mR3CAST(sortedVelMas),
			mR4CAST(sortedRhoPreMu), U1CAST(mapOriginalToSorted),
			U1CAST(cellStart), U1CAST(cellEnd), numBCE);

	hipDeviceSynchronize();
	cudaCheckError()
	;

	CalcBCE_MainStresses_kernel<<<numBlocks, numThreads>>>(mR4CAST(mainStressD),
			mR3CAST(devStressD), mR3CAST(volStressD), numBCE);

	hipDeviceSynchronize();
	cudaCheckError()
	;
}

/**
 * @brief Wrapper function for collideD
 * @details
 * 		See SDKCollisionSystem.cuh for informaton on collide
 */
void collide(thrust::device_vector<Real4>& sortedDerivVelRho_fsi_D,
		thrust::device_vector<Real3>& sortedPosRad,
		thrust::device_vector<Real3>& sortedVelMas,
		thrust::device_vector<Real3>& vel_XSPH_Sorted_D,
		thrust::device_vector<Real4>& sortedRhoPreMu,

		thrust::device_vector<uint>& gridMarkerIndex,
		thrust::device_vector<uint>& cellStart,
		thrust::device_vector<uint>& cellEnd, uint numAllMarkers, uint numCells,
		Real dT) {
	//#if USE_TEX
	//    cutilSafeCall(hipBindTexture(0, oldPosTex, sortedPosRad, numAllMarkers*sizeof(Real4)));
	//    cutilSafeCall(hipBindTexture(0, oldVelTex, sortedVelMas, numAllMarkers*sizeof(Real4)));
	//    cutilSafeCall(hipBindTexture(0, cellStartTex, cellStart, numCells*sizeof(uint)));
	//    cutilSafeCall(hipBindTexture(0, cellEndTex, cellEnd, numCells*sizeof(uint)));
	//#endif

	// thread per particle
	uint numThreads, numBlocks;
	computeGridSize(numAllMarkers, 64, numBlocks, numThreads);

	// execute the kernel
	collideD<<<numBlocks, numThreads>>>(mR4CAST(sortedDerivVelRho_fsi_D),
			mR3CAST(sortedPosRad), mR3CAST(sortedVelMas),
			mR3CAST(vel_XSPH_Sorted_D), mR4CAST(sortedRhoPreMu),
			U1CAST(cellStart), U1CAST(cellEnd),
			numAllMarkers);

	hipDeviceSynchronize();
	cudaCheckError();

//					// unroll sorted index to have the location of original particles in the sorted arrays
//					thrust::device_vector<uint> dummyIndex = gridMarkerIndex;
//					thrust::sort_by_key(dummyIndex.begin(), dummyIndex.end(),
//							derivVelRhoD.begin());
//					dummyIndex.clear();


	//#if USE_TEX
	//    cutilSafeCall(hipUnbindTexture(oldPosTex));
	//    cutilSafeCall(hipUnbindTexture(oldVelTex));
	//    cutilSafeCall(hipUnbindTexture(cellStartTex));
	//    cutilSafeCall(hipUnbindTexture(cellEndTex));
	//#endif
}
//--------------------------------------------------------------------------------------------------------------------------------
void ReCalcDensity(thrust::device_vector<Real3>& oldPosRad,
		thrust::device_vector<Real3>& oldVelMas,
		thrust::device_vector<Real4>& oldRhoPreMu,
		thrust::device_vector<Real3>& sortedPosRad,
		thrust::device_vector<Real3>& sortedVelMas,
		thrust::device_vector<Real4>& sortedRhoPreMu,
		thrust::device_vector<uint>& gridMarkerIndex,
		thrust::device_vector<uint>& cellStart,
		thrust::device_vector<uint>& cellEnd, uint numAllMarkers) {
	//#if USE_TEX
	//    cutilSafeCall(hipBindTexture(0, oldPosTex, sortedPosRad, numAllMarkers*sizeof(Real4)));
	//    cutilSafeCall(hipBindTexture(0, oldVelTex, sortedVelMas, numAllMarkers*sizeof(Real4)));
	//    cutilSafeCall(hipBindTexture(0, cellStartTex, cellStart, numCells*sizeof(uint)));
	//    cutilSafeCall(hipBindTexture(0, cellEndTex, cellEnd, numCells*sizeof(uint)));
	//#endif

	// thread per particle
	uint numThreads, numBlocks;
	computeGridSize(numAllMarkers, 64, numBlocks, numThreads);

	// execute the kernel
	ReCalcDensityD_F1<<<numBlocks, numThreads>>>(mR3CAST(oldPosRad),
			mR3CAST(oldVelMas), mR4CAST(oldRhoPreMu), mR3CAST(sortedPosRad),
			mR3CAST(sortedVelMas), mR4CAST(sortedRhoPreMu),
			U1CAST(gridMarkerIndex), U1CAST(cellStart), U1CAST(cellEnd),
			numAllMarkers);

	hipDeviceSynchronize();
	cudaCheckError()
	;

	//#if USE_TEX
	//    cutilSafeCall(hipUnbindTexture(oldPosTex));
	//    cutilSafeCall(hipUnbindTexture(oldVelTex));
	//    cutilSafeCall(hipUnbindTexture(cellStartTex));
	//    cutilSafeCall(hipUnbindTexture(cellEndTex));
	//#endif
}
//--------------------------------------------------------------------------------------------------------------------------------
void ProjectDensityPressureToBCandBCE(thrust::device_vector<Real4>& oldRhoPreMu,
		thrust::device_vector<Real3>& sortedPosRad,
		thrust::device_vector<Real4>& sortedRhoPreMu,
		thrust::device_vector<uint>& gridMarkerIndex,
		thrust::device_vector<uint>& cellStart,
		thrust::device_vector<uint>& cellEnd, uint numAllMarkers) {
	//#if USE_TEX
	//    cutilSafeCall(hipBindTexture(0, oldPosTex, sortedPosRad, numAllMarkers*sizeof(Real4)));
	//    cutilSafeCall(hipBindTexture(0, oldVelTex, sortedVelMas, numAllMarkers*sizeof(Real4)));
	//    cutilSafeCall(hipBindTexture(0, cellStartTex, cellStart, numCells*sizeof(uint)));
	//    cutilSafeCall(hipBindTexture(0, cellEndTex, cellEnd, numCells*sizeof(uint)));
	//#endif

	// thread per particle
	uint numThreads, numBlocks;
	computeGridSize(numAllMarkers, 64, numBlocks, numThreads);

	// execute the kernel
	ProjectDensityPressureToBCandBCE_D<<<numBlocks, numThreads>>>(
			mR4CAST(oldRhoPreMu), mR3CAST(sortedPosRad),
			mR4CAST(sortedRhoPreMu), U1CAST(gridMarkerIndex), U1CAST(cellStart),
			U1CAST(cellEnd), numAllMarkers);

	hipDeviceSynchronize();
	cudaCheckError()
	;

	//#if USE_TEX
	//    cutilSafeCall(hipUnbindTexture(oldPosTex));
	//    cutilSafeCall(hipUnbindTexture(oldVelTex));
	//    cutilSafeCall(hipUnbindTexture(cellStartTex));
	//    cutilSafeCall(hipUnbindTexture(cellEndTex));
	//#endif
}
//--------------------------------------------------------------------------------------------------------------------------------
void CalcCartesianData(thrust::device_vector<Real4>& rho_Pres_CartD,
		thrust::device_vector<Real4>& vel_VelMag_CartD,
		thrust::device_vector<Real3>& sortedPosRad,
		thrust::device_vector<Real3>& sortedVelMas,
		thrust::device_vector<Real4>& sortedRhoPreMu,
		thrust::device_vector<uint>& gridMarkerIndex,
		thrust::device_vector<uint>& cellStart,
		thrust::device_vector<uint>& cellEnd, uint cartesianGridSize,
		int3 cartesianGridDims, Real resolution) {

	// thread per particle
	uint numThreads, numBlocks;
	computeGridSize(cartesianGridSize, 64, numBlocks, numThreads);

	// execute the kernel
	CalcCartesianDataD<<<numBlocks, numThreads>>>(mR4CAST(rho_Pres_CartD),
			mR4CAST(vel_VelMag_CartD), mR3CAST(sortedPosRad),
			mR3CAST(sortedVelMas), mR4CAST(sortedRhoPreMu),
			U1CAST(gridMarkerIndex), U1CAST(cellStart), U1CAST(cellEnd),
			cartesianGridDims, resolution);

	hipDeviceSynchronize();
	cudaCheckError()
	;

	//#if USE_TEX
	//    cutilSafeCall(hipUnbindTexture(oldPosTex));
	//    cutilSafeCall(hipUnbindTexture(oldVelTex));
	//    cutilSafeCall(hipUnbindTexture(cellStartTex));
	//    cutilSafeCall(hipUnbindTexture(cellEndTex));
	//#endif
}

//%%%%%%%%%%%%%%%%%%%%%%%%%%
//--------------------------------------------------------------------------------------------------------------------------------
// updates the fluid particles by calling UpdateFluidD
void UpdateFluid(thrust::device_vector<Real3>& posRadD,
		thrust::device_vector<Real3>& velMasD,
		thrust::device_vector<Real3>& vel_XSPH_D,
		thrust::device_vector<Real4>& rhoPresMuD,
		thrust::device_vector<Real4>& derivVelRhoD,
		const thrust::host_vector<int4>& referenceArray, Real dT) {

//	int4 referencePortion = referenceArray[0];
//	if (referencePortion.z != -1) {
//		printf("error in UpdateFluid, accessing non fluid\n");
//		return;
//	}
//	int2 updatePortion = mI2(referencePortion);
	int2 updatePortion = mI2(0, referenceArray[referenceArray.size() - 1].y);
	// int2 updatePortion = mI2(referenceArray[0].x, referenceArray[0].y);

	uint nBlock_UpdateFluid, nThreads;
	computeGridSize(updatePortion.y - updatePortion.x, 128, nBlock_UpdateFluid,
			nThreads);
	UpdateFluidD<<<nBlock_UpdateFluid, nThreads>>>(mR3CAST(posRadD),
			mR3CAST(velMasD), mR3CAST(vel_XSPH_D), mR4CAST(rhoPresMuD),
			mR4CAST(derivVelRhoD), updatePortion, dT);
	hipDeviceSynchronize();
	cudaCheckError()
	;
}
//--------------------------------------------------------------------------------------------------------------------------------
// updates the fluid particles by calling UpdateFluid_init_LF
void UpdateFluid_init_LF(thrust::device_vector<Real3>& posRadD,
		thrust::device_vector<Real3>& velMasD_half,
		thrust::device_vector<Real4>& rhoPresMuD_half,
		const thrust::device_vector<Real4>& derivVelRhoD,
		const thrust::host_vector<int4>& referenceArray, Real dT) {
	int4 referencePortion = referenceArray[0];
	if (referencePortion.z != -1) {
		printf("error in UpdateFluid, accessing non fluid\n");
		return;
	}
	int2 updatePortion = mI2(referencePortion);
	// int2 updatePortion = mI2(referenceArray[0].x, referenceArray[0].y);

	uint nBlock_UpdateFluid, nThreads;
	computeGridSize(updatePortion.y - updatePortion.x, 128, nBlock_UpdateFluid,
			nThreads);
	UpdateFluidD_init_LF<<<nBlock_UpdateFluid, nThreads>>>(mR3CAST(posRadD),
			mR3CAST(velMasD_half), mR4CAST(rhoPresMuD_half),
			mR4CAST(derivVelRhoD), updatePortion, dT);
	hipDeviceSynchronize();
	cudaCheckError()
	;
}
//--------------------------------------------------------------------------------------------------------------------------------
// updates the fluid particles by calling UpdateFluid_rho_vel_LF
void UpdateFluid_rho_vel_LF(thrust::device_vector<Real3>& velMasD,
		thrust::device_vector<Real4>& rhoPresMuD,
		const thrust::device_vector<Real3>& velMasD_old,
		const thrust::device_vector<Real4>& rhoPresMuD_old,
		const thrust::device_vector<Real4>& derivVelRhoD,
		const thrust::host_vector<int4>& referenceArray, Real dT) {
	int4 referencePortion = referenceArray[0];
	if (referencePortion.z != -1) {
		printf("error in UpdateFluid, accessing non fluid\n");
		return;
	}
	int2 updatePortion = mI2(referencePortion);
	// int2 updatePortion = mI2(referenceArray[0].x, referenceArray[0].y);

	uint nBlock_UpdateFluid, nThreads;
	computeGridSize(updatePortion.y - updatePortion.x, 128, nBlock_UpdateFluid,
			nThreads);
	UpdateFluidD_rho_vel_LF<<<nBlock_UpdateFluid, nThreads>>>(mR3CAST(velMasD),
			mR4CAST(rhoPresMuD), mR3CAST(velMasD_old), mR4CAST(rhoPresMuD_old),
			mR4CAST(derivVelRhoD), updatePortion, dT);
	hipDeviceSynchronize();
	cudaCheckError()
	;
}
//--------------------------------------------------------------------------------------------------------------------------------
// updates the fluid particles by calling UpdateFluid_EveryThing_LF
void UpdateFluid_EveryThing_LF(thrust::device_vector<Real3>& posRadD,
		thrust::device_vector<Real3>& velMasD_half,
		thrust::device_vector<Real4>& rhoPresMuD_half,
		const thrust::device_vector<Real4>& derivVelRhoD,
		const thrust::host_vector<int4>& referenceArray, Real dT) {
	int4 referencePortion = referenceArray[0];
	if (referencePortion.z != -1) {
		printf("error in UpdateFluid, accessing non fluid\n");
		return;
	}
	int2 updatePortion = mI2(referencePortion);
	// int2 updatePortion = mI2(referenceArray[0].x, referenceArray[0].y);

	uint nBlock_UpdateFluid, nThreads;
	computeGridSize(updatePortion.y - updatePortion.x, 128, nBlock_UpdateFluid,
			nThreads);
	UpdateFluidD_EveryThing_LF<<<nBlock_UpdateFluid, nThreads>>>(
			mR3CAST(posRadD), mR3CAST(velMasD_half), mR4CAST(rhoPresMuD_half),
			mR4CAST(derivVelRhoD), updatePortion, dT);
	hipDeviceSynchronize();
	cudaCheckError()
	;
}

//--------------------------------------------------------------------------------------------------------------------------------
void CopySorted_vXSPH_dVdRho_to_original(thrust::device_vector<Real3>& vel_XSPH_D,
		thrust::device_vector<Real4>& derivVelRhoD,
		thrust::device_vector<Real3>& vel_XSPH_Sorted_D,
		thrust::device_vector<Real4>& sortedDerivVelRho_fsi_D,
		thrust::device_vector<uint>& mapOriginalToSorted, int numAllMarkers) {
	uint nBlock_NumSpheres, nThreads_SphMarkers;
	computeGridSize(numAllMarkers, 256, nBlock_NumSpheres, nThreads_SphMarkers);
	CopySorted_vXSPH_dVdRho_to_original_kernel<<<nBlock_NumSpheres, nThreads_SphMarkers>>>(
			mR3CAST(vel_XSPH_D), mR4CAST(derivVelRhoD),
			mR3CAST(vel_XSPH_Sorted_D),mR4CAST(sortedDerivVelRho_fsi_D),
			U1CAST(mapOriginalToSorted));
	hipDeviceSynchronize();
	cudaCheckError()
	;
}
//--------------------------------------------------------------------------------------------------------------------------------
// updates the fluid particles by calling UpdateBoundary
void UpdateBoundary(thrust::device_vector<Real3>& posRadD,
		thrust::device_vector<Real3>& velMasD,
		thrust::device_vector<Real4>& rhoPresMuD,
		thrust::device_vector<Real4>& derivVelRhoD,
		const thrust::host_vector<int4>& referenceArray, Real dT) {
	int4 referencePortion = referenceArray[1];
	if (referencePortion.z != 0) {
		printf("error in UpdateBoundary, accessing non boundary\n");
		return;
	}
	int2 updatePortion = mI2(referencePortion);

	uint nBlock_UpdateFluid, nThreads;
	computeGridSize(updatePortion.y - updatePortion.x, 128, nBlock_UpdateFluid,
			nThreads);
	UpdateKernelBoundary<<<nBlock_UpdateFluid, nThreads>>>(mR3CAST(posRadD),
			mR3CAST(velMasD), mR4CAST(rhoPresMuD), mR4CAST(derivVelRhoD),
			updatePortion, dT);
	hipDeviceSynchronize();
	cudaCheckError()
	;
}

/**
 * @brief ApplyBoundarySPH_Markers
 * @details
 * 		See SDKCollisionSystem.cuh for more info
 */
void ApplyBoundarySPH_Markers(thrust::device_vector<Real3>& posRadD,
		thrust::device_vector<Real4>& rhoPresMuD, int numAllMarkers) {
	uint nBlock_NumSpheres, nThreads_SphMarkers;
	computeGridSize(numAllMarkers, 256, nBlock_NumSpheres, nThreads_SphMarkers);
	ApplyPeriodicBoundaryXKernel<<<nBlock_NumSpheres, nThreads_SphMarkers>>>(
			mR3CAST(posRadD), mR4CAST(rhoPresMuD));
	hipDeviceSynchronize();
	cudaCheckError()
	;
	// these are useful anyway for out of bound particles
	ApplyPeriodicBoundaryYKernel<<<nBlock_NumSpheres, nThreads_SphMarkers>>>(
			mR3CAST(posRadD), mR4CAST(rhoPresMuD));
	hipDeviceSynchronize();
	cudaCheckError()
	;
	ApplyPeriodicBoundaryZKernel<<<nBlock_NumSpheres, nThreads_SphMarkers>>>(
			mR3CAST(posRadD), mR4CAST(rhoPresMuD));
	hipDeviceSynchronize();
	cudaCheckError()
	;

	//	SetOutputPressureToZero_X<<<nBlock_NumSpheres, nThreads_SphMarkers>>>(mR3CAST(posRadD), mR4CAST(rhoPresMuD));
	//    hipDeviceSynchronize();
	//    cudaCheckError();
}
