#include "hip/hip_runtime.h"
#include "contactForces.cuh"

//--------------------------------------------------------------------------------------------------------------------------------
__device__ inline real3 DEM_Force(real_ penetration, real_ rRigidDEM1, real_ rRigidDEM2, real4 velMasRigidA, real4 velMasRigidB) {
	return R3(0);
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void Add_ContactForcesD(real3 * totalAccRigid3, real3 * posRigidD, real4 * velMassRigidD) {
	uint rigidSphereA = blockIdx.x * blockDim.x + threadIdx.x;
	if (rigidSphereA >= numObjectsD.numRigidBodies) {
		return;
	}
	real3 posRigidA = posRigidD[rigidSphereA];
	real4 dummyVelMasA = velMassRigidD[rigidSphereA];

	real3 force3 = R3(0);
	real_ penDist = ContactWith_YPlanes(posRigidA, paramsD.rigidRadius.x);
	if (penDist < 0) {
		force3 += DEM_Force(-penDist, paramsD.rigidRadius.x, 20 * paramsD.rigidRadius.x, dummyVelMasA, R4(0));
	}

	penDist = ContactWithSerpentineCurve(posRigidA, paramsD.rigidRadius.x);
	if (penDist < 0) {
		force3 += DEM_Force(-penDist, paramsD.rigidRadius.x, 20 * paramsD.rigidRadius.x, dummyVelMasA, R4(0)); //approximate the curve with straight line
	}


	for (uint rigidSphereB = 0; rigidSphereB < numObjectsD.numRigidBodies; rigidSphereB ++) { //n^2 operation
		if (rigidSphereB == rigidSphereA) {
			continue; //avoid self contact
		}
		real3 posRigidB = posRigidD[rigidSphereB];
		real4 dummyVelMasB = velMassRigidD[rigidSphereB];
		penDist = length(posRigidB - posRigidA) - 2 * paramsD.rigidRadius.x;
		if (penDist < 0) {
			force3 += DEM_Force(-penDist, paramsD.rigidRadius.x, paramsD.rigidRadius.x, dummyVelMasA, dummyVelMasB);
		}
	}

	real3 totalAcc = totalAccRigid3[rigidSphereA];
	totalAcc += force3 / dummyVelMasA.w;
	totalAccRigid3[rigidSphereA] = totalAcc;
}
//--------------------------------------------------------------------------------------------------------------------------------
void setParameters2(SimParams *hostParams, NumberOfObjects *numObjects) {
	// copy parameters to constant memory
	cutilSafeCall( hipMemcpyToSymbolAsync(HIP_SYMBOL(paramsD), hostParams, sizeof(SimParams)));
	cutilSafeCall( hipMemcpyToSymbolAsync(HIP_SYMBOL(numObjectsD), numObjects, sizeof(NumberOfObjects)));
}
//--------------------------------------------------------------------------------------------------------------------------------
void Add_ContactForces(
		real3* totalAccRigid3,
		real3* posRigidD,
		real4* velMassRigidD) {
	NumberOfObjects numObjectsH;
	hipMemcpyFromSymbolAsync(&numObjectsH, HIP_SYMBOL(numObjectsD), sizeof(NumberOfObjects));
	printf("*** numObjectsH %d\n", numObjectsH.numRigidBodies);

	//**********************************************************************
	SerpentineParams serpGeom;
	serpGeom.mm = .001;
	serpGeom.r1_2 = R2(1.351, 1.750) * serpGeom.mm;
	serpGeom.r2_2 = R2(1.341, 1.754) * serpGeom.mm;
	serpGeom.r3_2 = R2(2.413, 3.532) * serpGeom.mm;
	serpGeom.r4_2 = R2(0.279, 0.413) * serpGeom.mm;

	serpGeom.r5_2 = R2(1.675, 1.235) * serpGeom.mm; //r5_2 = R2(1.727, 1.235);  	//the smaller one
	serpGeom.r6_2 = R2(2.747, 4.272) * serpGeom.mm; //the larger one
	serpGeom.x_FirstChannel = 8 * serpGeom.mm;
	serpGeom.sPeriod = 5.384 * serpGeom.mm; //serpentine period
	serpGeom.x_SecondChannel = 2 * serpGeom.mm;

	cutilSafeCall( hipMemcpyToSymbolAsync(HIP_SYMBOL(serpGeomD), &serpGeom, sizeof(SerpentineParams)));
	//**********************************************************************

	uint nBlock_UpdateRigid;
	uint nThreads_rigidParticles;
	computeGridSize(numObjectsH.numRigidBodies, 128, nBlock_UpdateRigid, nThreads_rigidParticles);

	Add_ContactForcesD<<<nBlock_UpdateRigid, nThreads_rigidParticles>>>(totalAccRigid3, posRigidD, velMassRigidD);

}

