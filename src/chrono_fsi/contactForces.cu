#include "hip/hip_runtime.h"
#include "contactForces.cuh"

//--------------------------------------------------------------------------------------------------------------------------------
__global__ void Add_ContactForces_SerpentineD(real3 * totalAccRigid3, real3 * posRigidD, real4 * velMassRigidD) {
	uint rigidSphereA = blockIdx.x * blockDim.x + threadIdx.x;
	if (rigidSphereA >= numObjectsD.numRigidBodies) {
		return;
	}
	real3 posRigidA = posRigidD[rigidSphereA];
	real4 dummyVelMasA = velMassRigidD[rigidSphereA];
	real3 n3;

	real3 force3 = R3(0);
	real_ rad = paramsD.rigidRadius.x + paramsD.HSML;
	real_ penDist = ContactWith_YPlanes(n3, posRigidA, rad);
	if (penDist < 0) {
		force3 += DEM_Force(-penDist, n3, rad, 20 * rad, dummyVelMasA, R4(0));
	}

	penDist = ContactWithSerpentineCurve(n3, posRigidA, rad);
	if (penDist < 0) {
		force3 += DEM_Force(-penDist, n3, rad, 20 * rad, dummyVelMasA, R4(0)); //approximate the curve with straight line
	}

	ContactWithOtherSpheres(force3, n3, rigidSphereA, posRigidA, dummyVelMasA, posRigidD, velMassRigidD, rad);

	real3 totalAcc = totalAccRigid3[rigidSphereA];
	totalAcc += force3 / dummyVelMasA.w;
	totalAccRigid3[rigidSphereA] = totalAcc;
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void Add_ContactForces_StraightChD(real3 * totalAccRigid3, real3 * posRigidD, real4 * velMassRigidD) {
	uint rigidSphereA = blockIdx.x * blockDim.x + threadIdx.x;
	if (rigidSphereA >= numObjectsD.numRigidBodies) {
		return;
	}
	real3 posRigidA = posRigidD[rigidSphereA];
	real4 dummyVelMasA = velMassRigidD[rigidSphereA];
	real3 n3;

	real3 force3 = R3(0);
	real_ rad = paramsD.rigidRadius.x + paramsD.HSML;
	real_ penDist = ContactWith_YPlanes(n3, posRigidA, rad);
	if (penDist < 0) {
		force3 += DEM_Force(-penDist, n3, rad, 20 * rad, dummyVelMasA, R4(0));
	}
	penDist = ContactWith_ZPlanes(n3, posRigidA, rad);
	if (penDist < 0) {
		force3 += DEM_Force(-penDist, n3, rad, 20 * rad, dummyVelMasA, R4(0));
	}

	ContactWithOtherSpheres(force3, n3, rigidSphereA, posRigidA, dummyVelMasA, posRigidD, velMassRigidD, rad);

	real3 totalAcc = totalAccRigid3[rigidSphereA];
	totalAcc += force3 / dummyVelMasA.w;
	totalAccRigid3[rigidSphereA] = totalAcc;
}
//--------------------------------------------------------------------------------------------------------------------------------
void setParameters2(SimParams *hostParams, NumberOfObjects *numObjects) {
	// copy parameters to constant memory
	cutilSafeCall( hipMemcpyToSymbolAsync(HIP_SYMBOL(paramsD), hostParams, sizeof(SimParams)));
	cutilSafeCall( hipMemcpyToSymbolAsync(HIP_SYMBOL(numObjectsD), numObjects, sizeof(NumberOfObjects)));
}
//--------------------------------------------------------------------------------------------------------------------------------
void Add_ContactForces_Serpentine(
		real3* totalAccRigid3,
		real3* posRigidD,
		real4* velMassRigidD) {
	NumberOfObjects numObjectsH;
	hipMemcpyFromSymbolAsync(&numObjectsH, HIP_SYMBOL(numObjectsD), sizeof(NumberOfObjects));

	//**********************************************************************
	SerpentineParams serpGeom;
	serpGeom.mm = .001;
	serpGeom.r1_2 = R2(1.351, 1.750) * serpGeom.mm;
	serpGeom.r2_2 = R2(1.341, 1.754) * serpGeom.mm;
	serpGeom.r3_2 = R2(2.413, 3.532) * serpGeom.mm;
	serpGeom.r4_2 = R2(0.279, 0.413) * serpGeom.mm;

	serpGeom.r5_2 = R2(1.675, 1.235) * serpGeom.mm; //r5_2 = R2(1.727, 1.235);  	//the smaller one
	serpGeom.r6_2 = R2(2.747, 4.272) * serpGeom.mm; //the larger one
	serpGeom.x_FirstChannel = 8 * serpGeom.mm;
	serpGeom.sPeriod = 5.384 * serpGeom.mm; //serpentine period
	serpGeom.x_SecondChannel = 2 * serpGeom.mm;

	cutilSafeCall( hipMemcpyToSymbolAsync(HIP_SYMBOL(serpGeomD), &serpGeom, sizeof(SerpentineParams)));
	//**********************************************************************

	uint nBlock_UpdateRigid;
	uint nThreads_rigidParticles;
	computeGridSize(numObjectsH.numRigidBodies, 128, nBlock_UpdateRigid, nThreads_rigidParticles);

	Add_ContactForces_SerpentineD<<<nBlock_UpdateRigid, nThreads_rigidParticles>>>(totalAccRigid3, posRigidD, velMassRigidD);

}
//--------------------------------------------------------------------------------------------------------------------------------
void Add_ContactForces_StraightCh(
		real3* totalAccRigid3,
		real3* posRigidD,
		real4* velMassRigidD) {
	NumberOfObjects numObjectsH;
	hipMemcpyFromSymbolAsync(&numObjectsH, HIP_SYMBOL(numObjectsD), sizeof(NumberOfObjects));
	//**********************************************************************

	uint nBlock_UpdateRigid;
	uint nThreads_rigidParticles;
	computeGridSize(numObjectsH.numRigidBodies, 128, nBlock_UpdateRigid, nThreads_rigidParticles);

	Add_ContactForces_StraightChD<<<nBlock_UpdateRigid, nThreads_rigidParticles>>>(totalAccRigid3, posRigidD, velMassRigidD);

}

