#include "hip/hip_runtime.h"
#include "custom_cutil_math.h"
#include "SPHCudaUtils.h"
#include <sys/time.h>
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/reduce.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "SDKCollisionSystem.cuh"
#include "collideSphereSphere.cuh"
#include "FlexibleBodies.cuh"
#include "printToFile.cuh"
#include <string.h>
#include <stdio.h>
#include <math.h>
#include <sstream>
#include <fstream>
using namespace std;
//#####################################################################################
#define B_SIZE 128
//#####################################################################################
__constant__ real_ dTD;
__constant__ real_ solid_SPH_massD;
__constant__ int2 updatePortionD;
__constant__ int2 portionD;
__constant__ int flagD;
//--------------------------------------------------------------------------------------------------------------------------------
// first comp of q is rotation, last 3 components are axis of rot
__device__ __host__ inline void RotationMatirixFromQuaternion(real3 & AD1, real3 & AD2, real3 & AD3, const real4 & q) {
	AD1 = 2 * R3(0.5f - q.z * q.z - q.w * q.w, q.y * q.z - q.x * q.w, q.y * q.w + q.x * q.z);
	AD2 = 2 * R3(q.y * q.z + q.x * q.w, 0.5f - q.y * q.y - q.w * q.w, q.z * q.w - q.x * q.y);
	AD3 = 2 * R3(q.y * q.w - q.x * q.z, q.z * q.w + q.x * q.y, 0.5f - q.y * q.y - q.z * q.z);
}
//--------------------------------------------------------------------------------------------------------------------------------
// first comp of q is rotation, last 3 components are axis of rot
__device__ __host__ inline void QuaternionFromRotationMatirix(real4 & q, const real3 & AD1, const real3 & AD2, const real3 & AD3) {
	real_ q0, q1, q2, q3;
	q0 = 0.5 * sqrt(1 + AD1.x + AD2.y + AD3.z);
	q1 = 0.5 * sqrt(1 + AD1.x - AD2.y - AD3.z);
	q2 = 0.5 * sqrt(1 - AD1.x + AD2.y - AD3.z);
	q3 = 0.5 * sqrt(1 - AD1.x - AD2.y + AD3.z);
	if (fabs(q0) > .1) {
		q.x = q0;
		q.y = .25 / q0 * (AD3.y - AD2.z);
		q.z = .25 / q0 * (AD1.z - AD3.x);
		q.w = .25 / q0 * (AD2.x - AD1.y);
	} else if (fabs(q1) > .1) {
		q.x = .25 / q1 * (AD3.y - AD2.z);
		q.y = q1;
		q.z = .25 / q1 * (AD1.y + AD2.x);
		q.w = .25 / q1 * (AD1.z + AD3.x);
	} else if (fabs(q2) > .1) {
		q.x = .25 / q2 * (AD1.z - AD3.x);
		q.y = .25 / q2 * (AD1.y + AD2.x);
		q.z = q2;
		q.w = .25 / q2 * (AD2.z + AD3.y);
	} else if (fabs(q3) > .1) {
		q.x = .25 / q3 * (AD2.x - AD1.y);
		q.y = .25 / q3 * (AD1.z + AD3.x);
		q.z = .25 / q3 * (AD2.z + AD3.y);
		q.w = q3;
	} else {
		printf("\n\n\nError in quaternion! Quaterniion is not Normalized\n\n\n");
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
// first comp of q is rotation, last 3 components are axis of rot
__device__ __host__ inline void RotationMatirixFromAxisVector(real3 & AD1, real3 & AD2, real3 & AD3, const real3 & n) {
	real3 n3z = normalize(n);
	real3 n3x, n3y;
	if (length(n3z - R3(1,0,0)) > .001) {
		n3x = normalize(cross(n3z, R3(1,0,0)));
	} else {
		n3x = normalize(cross(n3z, R3(0,1,0)));
	}
	n3y = cross(n3z,n3x);
	AD1 = R3(n3x.x, n3y.x, n3z.x);
	AD2 = R3(n3x.y, n3y.y, n3z.y);
	AD3 = R3(n3x.z, n3y.z, n3z.z);
}
//--------------------------------------------------------------------------------------------------------------------------------
// first comp of q is rotation, last 3 components are axis of rot
__device__ __host__ inline void QuaternionFromAxisVector_DeviceHost(real4 & q, const real3 & n) {
	real3 aD1, aD2, aD3;
	RotationMatirixFromAxisVector(aD1, aD2, aD3, n);
	QuaternionFromRotationMatirix(q, aD1, aD2, aD3);
}
//--------------------------------------------------------------------------------------------------------------------------------
__device__ __host__ inline real3 InverseRotate_By_RotationMatrix_DeviceHost(const real3 & A1, const real3 & A2, const real3 & A3, const real3 & r3) {
	return R3(	A1.x * r3.x + A2.x * r3.y + A3.x * r3.z,
				A1.y * r3.x + A2.y * r3.y + A3.y * r3.z,
				A1.z * r3.x + A2.z * r3.y + A3.z * r3.z);
}
//--------------------------------------------------------------------------------------------------------------------------------
void QuaternionFromAxisVector(real4 & q, const real3 & n) {
	QuaternionFromAxisVector_DeviceHost(q, n);
}
//--------------------------------------------------------------------------------------------------------------------------------
// first comp of q is rotation, last 3 components are axis of rot
void CalcQuat2RotationMatrix(Rotation & rotMat, const real4 & q) {
	rotMat.a00 = 2.0 * (0.5f - q.z * q.z - q.w * q.w);
	rotMat.a01 = 2.0 * (q.y * q.z - q.x * q.w);
	rotMat.a02 = 2.0 * (q.y * q.w + q.x * q.z);

	rotMat.a10 = 2 * (q.y * q.z + q.x * q.w);
	rotMat.a11 = 2 * (0.5f - q.y * q.y - q.w * q.w);
	rotMat.a12 = 2 * (q.z * q.w - q.x * q.y);

	rotMat.a20 = 2 * (q.y * q.w - q.x * q.z);
	rotMat.a21 = 2 * (q.z * q.w + q.x * q.y);
	rotMat.a22 = 2 * (0.5f - q.y * q.y - q.z * q.z);
}
//--------------------------------------------------------------------------------------------------------------------------------
// first comp of q is rotation, last 3 components are axis of rot
real3 Rotate_By_RotationMatrix(const Rotation & rotMat, const real3 & r3) {
	return R3(rotMat.a00 * r3.x + rotMat.a01 * r3.y + rotMat.a02 * r3.z,
			rotMat.a10 * r3.x + rotMat.a11 * r3.y + rotMat.a12 * r3.z,
			rotMat.a20 * r3.x + rotMat.a21 * r3.y + rotMat.a22 * r3.z);
}
//--------------------------------------------------------------------------------------------------------------------------------
real3 InverseRotate_By_RotationMatrix(const Rotation & A, const real3 & r3) {
	real3 AD1 = R3(A.a00, A.a01, A.a02);
	real3 AD2 = R3(A.a10, A.a11, A.a12);
	real3 AD3 = R3(A.a20, A.a21, A.a22);
	return InverseRotate_By_RotationMatrix_DeviceHost(AD1, AD2, AD3, r3);
}
//--------------------------------------------------------------------------------------------------------------------------------
// first comp of q is rotation, last 3 components are axis of rot
real3 Rotate_By_Quaternion(const real4 & q4, const real3 & r3) {
	Rotation rotMat;
	CalcQuat2RotationMatrix(rotMat, q4);
	return Rotate_By_RotationMatrix(rotMat, r3);
}
//--------------------------------------------------------------------------------------------------------------------------------
__device__ __host__ inline int IndexOfClosestNode(real_ sOverBeam, real_ lBeam, int2 nodesInterval) {
	int nNodes = nodesInterval.y - nodesInterval.x;
	int maxNodeIdx = nNodes - 1;
	int indexOfClosestNodeLocal = int(sOverBeam / lBeam * maxNodeIdx);
	if (indexOfClosestNodeLocal == maxNodeIdx) indexOfClosestNodeLocal--;
	return (indexOfClosestNodeLocal + nodesInterval.x);
}
//--------------------------------------------------------------------------------------------------------------------------------
__device__ __host__ inline void Applied_Force(real_* f_a, real_ sE, real_ lE, real3 F)
{
	real_ S[4];

	shape_fun(S, sE, lE);

	f_a[0]  = F.x*S[0];
	f_a[1]  = F.y*S[0];
	f_a[2]  = F.z*S[0];
	f_a[3]  = F.x*S[1];
	f_a[4]  = F.y*S[1];
	f_a[5]  = F.z*S[1];
	f_a[6]  = F.x*S[2];
	f_a[7]  = F.y*S[2];
	f_a[8]  = F.z*S[2];
	f_a[9]  = F.x*S[3];
	f_a[10] = F.y*S[3];
	f_a[11] = F.z*S[3];
}
//--------------------------------------------------------------------------------------------------------------------------------
__device__ __host__ inline real3 Calc_ANCF_Point_Pos(
		real3 * ANCF_NodesD,
		real3 * ANCF_SlopesD,
		int indexOfClosestNode,
		real_ sE,
		real_ lE){
	real_ S[4];
	shape_fun(S, sE, lE);


	real3 r;
	real3 ni = ANCF_NodesD[indexOfClosestNode];
	real3 si = ANCF_SlopesD[indexOfClosestNode];
	real3 nj = ANCF_NodesD[indexOfClosestNode + 1];
	real3 sj = ANCF_SlopesD[indexOfClosestNode + 1];

	r.x = S[0]*ni.x + S[1]*si.x + S[2]*nj.x + S[3]*sj.x;
	r.y = S[0]*ni.y + S[1]*si.y + S[2]*nj.y + S[3]*sj.y;
	r.z = S[0]*ni.z + S[1]*si.z + S[2]*nj.z + S[3]*sj.z;

//	//ff1
//	printf("n1 %f %f %f, r %f %f %f, n2 %f %f %f\n", ni.x, ni.y, ni.z, r.x, r.y, r.z, nj.x, nj.y, nj.z);

	return r;
}
//--------------------------------------------------------------------------------------------------------------------------------
// in the calculations that use this function, we have assumed Calc_ANCF_Point_Slope returns the unit vector. theta calculation is based on this assumption. Also cross product
__device__ __host__ inline real3 Calc_ANCF_Point_Slope(
		real3 * ANCF_NodesD,
		real3 * ANCF_SlopesD,
		int indexOfClosestNode,
		real_ sE,
		real_ lE){
	real_ Sx[4];
	shape_fun_d(Sx, sE, lE);


	real3 rx;
	real3 ni = ANCF_NodesD[indexOfClosestNode];
	real3 si = ANCF_SlopesD[indexOfClosestNode];
	real3 nj = ANCF_NodesD[indexOfClosestNode + 1];
	real3 sj = ANCF_SlopesD[indexOfClosestNode + 1];

	rx.x = Sx[0]*ni.x + Sx[1]*si.x + Sx[2]*nj.x + Sx[3]*sj.x;
	rx.y = Sx[0]*ni.y + Sx[1]*si.y + Sx[2]*nj.y + Sx[3]*sj.y;
	rx.z = Sx[0]*ni.z + Sx[1]*si.z + Sx[2]*nj.z + Sx[3]*sj.z;

	return rx;
}
//--------------------------------------------------------------------------------------------------------------------------------
__device__ __host__ inline real3 Calc_ANCF_Point_Vel(
		real3 * ANCF_NodesVelD,
		real3 * ANCF_SlopesVelD,
		int indexOfClosestNode,
		real_ sE,
		real_ lE) {
	real_ S[4];
	shape_fun(S, sE, lE);


	real3 rt;
	real3 nti = ANCF_NodesVelD[indexOfClosestNode];
	real3 sti = ANCF_SlopesVelD[indexOfClosestNode];
	real3 ntj = ANCF_NodesVelD[indexOfClosestNode + 1];
	real3 stj = ANCF_SlopesVelD[indexOfClosestNode + 1];

	rt.x = S[0]*nti.x + S[1]*sti.x + S[2]*ntj.x + S[3]*stj.x;
	rt.y = S[0]*nti.y + S[1]*sti.y + S[2]*ntj.y + S[3]*stj.y;
	rt.z = S[0]*nti.z + S[1]*sti.z + S[2]*ntj.z + S[3]*stj.z;

	return rt;
}
//--------------------------------------------------------------------------------------------------------------------------------
// needs more work
__device__ __host__ inline real3 Calc_ANCF_Point_Omega(
		real3 * ANCF_NodesVelD,
		real3 * ANCF_SlopesVelD,
		int indexOfClosestNode,
		real_ sE,
		real_ lE,
		real3 rX){
	real_ Sx[4];
	shape_fun_d(Sx, sE, lE);


	real3 rxt;
	real3 nti = ANCF_NodesVelD[indexOfClosestNode];
	real3 sti = ANCF_SlopesVelD[indexOfClosestNode];
	real3 ntj = ANCF_NodesVelD[indexOfClosestNode + 1];
	real3 stj = ANCF_SlopesVelD[indexOfClosestNode + 1];

	rxt.x = Sx[0]*nti.x + Sx[1]*sti.x + Sx[2]*ntj.x + Sx[3]*stj.x;
	rxt.y = Sx[0]*nti.y + Sx[1]*sti.y + Sx[2]*ntj.y + Sx[3]*stj.y;
	rxt.z = Sx[0]*nti.z + Sx[1]*sti.z + Sx[2]*ntj.z + Sx[3]*stj.z;

	return cross(rX, rxt)/dot(rX, rX);
}
//--------------------------------------------------------------------------------------------------------------------------------
//updates the fluid particles' properties, i.e. velocity, density, pressure, position
__global__ void UpdateKernelFluid(real3 * posRadD, real4 * velMasD, real3 * vel_XSPH_D, real4 * rhoPresMuD, real4 * derivVelRhoD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	index += updatePortionD.x; // updatePortionD = [start, end] index of the update portion
	if (index >= updatePortionD.y) {
		return;
	}
	real3 vel_XSPH = vel_XSPH_D[index];
	// 1*** let's tweak a little bit :)
	if (length(vel_XSPH) > .2 * paramsD.HSML / dTD) {
		vel_XSPH *= ( .2 * paramsD.HSML / dTD ) / length(vel_XSPH);
	}
	// 1*** end tweak
	real3 posRad = posRadD[index];
	real3 updatedPositon = posRad + vel_XSPH * dTD;
	posRadD[index] = updatedPositon; //posRadD updated

	real4 derivVelRho = derivVelRhoD[index];
	real4 velMas = velMasD[index];
	real3 updatedVelocity = R3(velMas + derivVelRho * dTD);
	// 2*** let's tweak a little bit :)
	if (length(updatedVelocity) > .2 * paramsD.HSML / dTD) {
		updatedVelocity *= ( .2 * paramsD.HSML / dTD ) / length(updatedVelocity);
	}
	// 2*** end tweak
	velMasD[index] = R4(updatedVelocity, /*rho2 / rhoPresMu.x * */velMas.w); //velMasD updated

	real4 rhoPresMu = rhoPresMuD[index];
	real_ rho2 = rhoPresMu.x + derivVelRho.w * dTD; //rho update. (i.e. rhoPresMu.x), still not wriiten to global matrix
	rhoPresMu.y = Eos(rho2, rhoPresMu.w);
	rhoPresMu.x = rho2;
	rhoPresMuD[index] = rhoPresMu; //rhoPresMuD updated
}
//--------------------------------------------------------------------------------------------------------------------------------
//copies the sortedVelXSPH to velXSPH according to indexing
__global__ void Copy_SortedVelXSPH_To_VelXSPH(real3 * vel_XSPH_D, real3 * vel_XSPH_Sorted_D, uint * m_dGridMarkerIndex) {
	uint index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	if (index >= numObjectsD.numAllMarkers) return;
	vel_XSPH_D[m_dGridMarkerIndex[index]] = vel_XSPH_Sorted_D[index];
}
//--------------------------------------------------------------------------------------------------------------------------------
//updates the fluid particles' properties, i.e. velocity, density, pressure, position
__global__ void UpdateKernelBoundary(real3 * posRadD, real4 * velMasD, real4 * rhoPresMuD, real4 * derivVelRhoD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	index += updatePortionD.x; // updatePortionD = [start, end] index of the update portion
	if (index >= updatePortionD.y) {
		return;
	}

	real4 derivVelRho = derivVelRhoD[index];
	real4 rhoPresMu = rhoPresMuD[index];
	real_ rho2 = rhoPresMu.x + derivVelRho.w * dTD; //rho update. (i.e. rhoPresMu.x), still not wriiten to global matrix
	rhoPresMu.y = Eos(rho2, rhoPresMu.w);
	rhoPresMu.x = rho2;
	rhoPresMuD[index] = rhoPresMu; //rhoPresMuD updated
}
//--------------------------------------------------------------------------------------------------------------------------------
//applies periodic BC along x
__global__ void ApplyPeriodicBoundaryXKernel(real3 * posRadD, real4 * rhoPresMuD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numObjectsD.numAllMarkers) {
		return;
	}
	real4 rhoPresMu = rhoPresMuD[index];
	if (fabs(rhoPresMu.w) < .1) {
		return;
	} //no need to do anything if it is a boundary particle
	real3 posRad = posRadD[index];
	if (posRad.x > paramsD.cMax.x) {
		posRad.x -= (paramsD.cMax.x - paramsD.cMin.x);
		posRadD[index] = posRad;
		if (rhoPresMu.w < -.1) {
			rhoPresMu.y = rhoPresMu.y + paramsD.bodyForce4.x * (paramsD.cMax.x - paramsD.cMin.x);
			rhoPresMuD[index] = rhoPresMu;
		}
		return;
	}
	if (posRad.x < paramsD.cMin.x) {
		posRad.x += (paramsD.cMax.x - paramsD.cMin.x);
		posRadD[index] = posRad;
		if (rhoPresMu.w < -.1) {
			rhoPresMu.y = rhoPresMu.y - paramsD.bodyForce4.x * (paramsD.cMax.x - paramsD.cMin.x);
			rhoPresMuD[index] = rhoPresMu;
		}
		return;
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
//applies periodic BC along y
__global__ void ApplyPeriodicBoundaryYKernel(real3 * posRadD, real4 * rhoPresMuD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numObjectsD.numAllMarkers) {
		return;
	}
	real4 rhoPresMu = rhoPresMuD[index];
	if (fabs(rhoPresMu.w) < .1) {
		return;
	} //no need to do anything if it is a boundary particle
	real3 posRad = posRadD[index];
	if (posRad.y > paramsD.cMax.y) {
		posRad.y -= (paramsD.cMax.y - paramsD.cMin.y);
		posRadD[index] = posRad;
		if (rhoPresMu.w < -.1) {
			rhoPresMu.y = rhoPresMu.y + paramsD.bodyForce4.y * (paramsD.cMax.y - paramsD.cMin.y);
			rhoPresMuD[index] = rhoPresMu;
		}
		return;
	}
	if (posRad.y < paramsD.cMin.y) {
		posRad.y += (paramsD.cMax.y - paramsD.cMin.y);
		posRadD[index] = posRad;
		if (rhoPresMu.w < -.1) {
			rhoPresMu.y = rhoPresMu.y - paramsD.bodyForce4.y * (paramsD.cMax.y - paramsD.cMin.y);
			rhoPresMuD[index] = rhoPresMu;
		}
		return;
	}
}

//--------------------------------------------------------------------------------------------------------------------------------
//applies periodic BC along z
__global__ void ApplyPeriodicBoundaryZKernel(real3 * posRadD, real4 * rhoPresMuD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numObjectsD.numAllMarkers) {
		return;
	}
	real4 rhoPresMu = rhoPresMuD[index];
	if (fabs(rhoPresMu.w) < .1) {
		return;
	} //no need to do anything if it is a boundary particle
	real3 posRad = posRadD[index];
	if (posRad.z > paramsD.cMax.z) {
		posRad.z -= (paramsD.cMax.z - paramsD.cMin.z);
		posRadD[index] = posRad;
		if (rhoPresMu.w < -.1) {
			rhoPresMu.y = rhoPresMu.y + paramsD.bodyForce4.z * (paramsD.cMax.z - paramsD.cMin.z);
			rhoPresMuD[index] = rhoPresMu;
		}
		return;
	}
	if (posRad.z < paramsD.cMin.z) {
		posRad.z += (paramsD.cMax.z - paramsD.cMin.z);
		posRadD[index] = posRad;
		if (rhoPresMu.w < -.1) {
			rhoPresMu.y = rhoPresMu.y - paramsD.bodyForce4.z * (paramsD.cMax.z - paramsD.cMin.z);
			rhoPresMuD[index] = rhoPresMu;
		}
		return;
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
//applies periodic BC along x, for ridid bodies
__global__ void ApplyPeriodicBoundaryXKernel_RigidBodies(real3 * posRigidD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numObjectsD.numRigidBodies) {
		return;
	}
	real3 posRigid = posRigidD[index];
	if (posRigid.x > paramsD.cMax.x) {
		posRigid.x -= (paramsD.cMax.x - paramsD.cMin.x);
		posRigidD[index] = posRigid;
		return;
	}
	if (posRigid.x < paramsD.cMin.x) {
		posRigid.x += (paramsD.cMax.x - paramsD.cMin.x);
		posRigidD[index] = posRigid;
		return;
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
//applies periodic BC along x, for ridid bodies
__global__ void ApplyPeriodicBoundaryYKernel_RigidBodies(real3 * posRigidD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numObjectsD.numRigidBodies) {
		return;
	}
	real3 posRigid = posRigidD[index];
	if (posRigid.y > paramsD.cMax.y) {
		posRigid.y -= (paramsD.cMax.y - paramsD.cMin.y);
		posRigidD[index] = posRigid;
		return;
	}
	if (posRigid.y < paramsD.cMin.y) {
		posRigid.y += (paramsD.cMax.y - paramsD.cMin.y);
		posRigidD[index] = posRigid;
		return;
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
//applies periodic BC along x, for ridid bodies
__global__ void ApplyPeriodicBoundaryZKernel_RigidBodies(real3 * posRigidD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numObjectsD.numRigidBodies) {
		return;
	}
	real3 posRigid = posRigidD[index];
	if (posRigid.z > paramsD.cMax.z) {
		posRigid.z -= (paramsD.cMax.z - paramsD.cMin.z);
		posRigidD[index] = posRigid;
		return;
	}
	if (posRigid.z < paramsD.cMin.z) {
		posRigid.z += (paramsD.cMax.z - paramsD.cMin.z);
		posRigidD[index] = posRigid;
		return;
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
//applies periodic BC along x, for ridid bodies
__global__ void ApplyPeriodicBoundaryKernel_FlexBodies(real3* ANCF_NodesD, int2* ANCF_ReferenceArrayNodesOnBeamsD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numObjectsD.numFlexBodies) {
		return;
	}
	int2 nodesInterval = ANCF_ReferenceArrayNodesOnBeamsD[index];

	bool flagXR = true, flagYR = true, flagZR = true;
	bool flagXL = true, flagYL = true, flagZL = true;
	for (int i = nodesInterval.x; i < nodesInterval.y; i++) {
		real3 nodePos = ANCF_NodesD[i];
		//*** max boundary
		if (nodePos.x <= paramsD.cMax.x) flagXR = false;
		if (nodePos.y <= paramsD.cMax.y) flagYR = false;
		if (nodePos.z <= paramsD.cMax.z) flagZR = false;
		//*** min boundary
		if (nodePos.x >= paramsD.cMin.x) flagXL = false;
		if (nodePos.y >= paramsD.cMin.y) flagYL = false;
		if (nodePos.z >= paramsD.cMin.z) flagZL = false;
	}
	for (int i = nodesInterval.x; i < nodesInterval.y; i++) {
		real3 nodePos = ANCF_NodesD[i];
		//*** max boundary
		if (flagXR) nodePos.x = fmod(nodePos.x - paramsD.cMin.x, paramsD.boxDims.x) + paramsD.cMin.x;
		if (flagYR) nodePos.y = fmod(nodePos.y - paramsD.cMin.y, paramsD.boxDims.y) + paramsD.cMin.y;
		if (flagZR) nodePos.z = fmod(nodePos.z - paramsD.cMin.z, paramsD.boxDims.z) + paramsD.cMin.z;
		//*** min boundary
		if (flagXL) nodePos.x = fmod(nodePos.x - paramsD.cMin.x, paramsD.boxDims.x) + paramsD.boxDims.x + paramsD.cMin.x;
		if (flagYL) nodePos.y = fmod(nodePos.y - paramsD.cMin.y, paramsD.boxDims.y) + paramsD.boxDims.y + paramsD.cMin.y;
		if (flagZL) nodePos.z = fmod(nodePos.z - paramsD.cMin.z, paramsD.boxDims.z) + paramsD.boxDims.z + paramsD.cMin.z;
		//***
		ANCF_NodesD[i] = nodePos;
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
//related to post processing of Segre-Silberberg. Distribution thing!
__global__ void PassesFromTheEnd_Kernel(
		real3 * posRigidD,
		uint * radialPositions,
		uint * radialPosCounter,
		real2 pipeCenter,
		real_ dR) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numObjectsD.numRigidBodies) {
		return;
	}
	real3 posRigid = posRigidD[index];
	if ( (posRigid.x > paramsD.cMax.x) || (posRigid.x < paramsD.cMin.x) ) {													//assuming the fluid flows in the positive x direction
		real_ r = length(R2(posRigid.y, posRigid.z) - pipeCenter);
		uint radPosition = int(r / dR);
		radialPositions[index] = radPosition;
		radialPosCounter[index] = 1;
			//printf("passed. r %f  dR %f    r/dR %f    radial_pos: %d",  r, dR , r/dR, radPosition);
		return;
	}
	//syncthreads();
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void AddToCumulutaiveNumberOfPasses(
		int * distributionD,
		uint * dummy_radialPosition,
		uint * radialPosCounter_Cumulative,
		int numberOfSections) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numberOfSections) {
		return;
	}
	uint radPosition = dummy_radialPosition[index];
	uint distributionCumul = radialPosCounter_Cumulative[index];
	if (radPosition < numberOfSections) {
		//if (distributionCumul > 0) printf("radPositon %d\n", radPosition);
		distributionD[radPosition] += distributionCumul;
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void Calc_SurfaceInducedAcceleration(real3 * totalAccRigid3, real4 * totalSurfaceInteractionRigid4, real4 * velMassRigidD) {
	uint rigidSphereA = blockIdx.x * blockDim.x + threadIdx.x;
	if (rigidSphereA >= numObjectsD.numRigidBodies) {
		return;
	}
	real4 dummyVelMas = velMassRigidD[rigidSphereA];
	real3 derivRigid = solid_SPH_massD / dummyVelMas.w * R3(totalSurfaceInteractionRigid4[rigidSphereA]);
	//** tweak 3
	if (length(derivRigid) > .2 * paramsD.HSML / (dTD * dTD)) {
			derivRigid *= ( .2 * paramsD.HSML / (dTD * dTD) ) / length(derivRigid);
	}
	//** end tweak
	totalAccRigid3[rigidSphereA] = derivRigid;
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void CalcTorqueOf_SPH_Marker_Acceleration(real3* torqueMarkersD, real4* derivVelRhoD, real3* posRadD, int* rigidIdentifierD, real3* posRigidD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	uint rigidMarkerIndex = index + numObjectsD.startRigidMarkers;
	if (index >= numObjectsD.numRigid_SphMarkers) {
		return;
	}
	real3 dist3 = Distance(posRadD[rigidMarkerIndex], posRigidD[rigidIdentifierD[index]]);
	torqueMarkersD[index] = cross(dist3, R3(derivVelRhoD[rigidMarkerIndex]));
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void MapForcesOnNodes(
		real3* flexNodesForcesAllMarkers1,
		real3* flexNodesForcesAllMarkers2,
		int* flexIdentifierD,
		int2* ANCF_ReferenceArrayNodesOnBeamsD,
		int* ANCF_NumMarkers_Per_BeamD,
		int* ANCF_NumMarkers_Per_Beam_CumulD, //exclusive scan
//		int* ANCF_NumNodesMultMarkers_Per_BeamD,
		int* ANCF_NumNodesMultMarkers_Per_Beam_CumulD, //exclusive scan
		real_* flexParametricDistD,
		real_* ANCF_Beam_LengthD,
		real4* derivVelRhoD)
{
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numObjectsD.numFlex_SphMarkers) {
		return;
	}
	uint absMarkerIndex = index + numObjectsD.startFlexMarkers;
	real_ sOverBeam = flexParametricDistD[index];

	real3 derivVel = R3( derivVelRhoD[absMarkerIndex] );
	real3 markerForce = solid_SPH_massD * derivVel;

//	Map Marker Force to ANCF Nodes, gives you as many forces as the number of nodes per beam
//	F0, F1, ..., F(m-1) : Forces on nodes 0, 1, 2, ..., m-1
//	Fi ---> flexNodesForces[numSavedForcesSoFar + (i * numMarkersOnThisBeam + markerIndexOnThisBeam)];
//	...

	int flexBodyIndex = flexIdentifierD[index];
	real_ lBeam = ANCF_Beam_LengthD[flexBodyIndex];


	int numFlexMarkersPreviousBeamsTotal = ANCF_NumMarkers_Per_Beam_CumulD[flexBodyIndex];
	int markerIndexOnThisBeam = index - numFlexMarkersPreviousBeamsTotal;
	int numMarkersOnThisBeam = ANCF_NumMarkers_Per_BeamD[flexBodyIndex];
	int numSavedForcesSoFar = ANCF_NumNodesMultMarkers_Per_Beam_CumulD[flexBodyIndex];

	int2 nodesInterval = ANCF_ReferenceArrayNodesOnBeamsD[flexBodyIndex];
	int indexOfClosestNode = IndexOfClosestNode(sOverBeam, lBeam, nodesInterval);
	int indexOfClosestNodeLocal = indexOfClosestNode - nodesInterval.x;

	int nNodes = nodesInterval.y - nodesInterval.x;
	real_ lE = lBeam / (nNodes - 1); //Element length
	real_ sE = fmod(sOverBeam, lE);
	real_ f_a[12] = {0};
	Applied_Force(f_a, sE, lE, markerForce);
	//left node
	flexNodesForcesAllMarkers1[numSavedForcesSoFar + indexOfClosestNodeLocal * numMarkersOnThisBeam + markerIndexOnThisBeam] = R3(f_a[0], f_a[1], f_a[2]);
	flexNodesForcesAllMarkers2[numSavedForcesSoFar + indexOfClosestNodeLocal * numMarkersOnThisBeam + markerIndexOnThisBeam] = R3(f_a[3], f_a[4], f_a[5]);
	//right node
	flexNodesForcesAllMarkers1[numSavedForcesSoFar + (indexOfClosestNodeLocal + 1) * numMarkersOnThisBeam + markerIndexOnThisBeam] = R3(f_a[6], f_a[7], f_a[8]);
	flexNodesForcesAllMarkers2[numSavedForcesSoFar + (indexOfClosestNodeLocal + 1) * numMarkersOnThisBeam + markerIndexOnThisBeam] = R3(f_a[9], f_a[10], f_a[11]);

}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void Populate_RigidSPH_MeshPos_LRF_kernel(
		real3* rigidSPH_MeshPos_LRF_D,
		real3* posRadD,
		int* rigidIdentifierD,
		real3* posRigidD,
		real3 * AD1,
		real3 * AD2,
		real3 * AD3) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	uint rigidMarkerIndex = index + numObjectsD.startRigidMarkers; // updatePortionD = [start, end] index of the update portion
	if (index >= numObjectsD.numRigid_SphMarkers) {
		return;
	}
	int rigidIndex = rigidIdentifierD[index];
	real3 aD1 = AD1[rigidIndex];
	real3 aD2 = AD2[rigidIndex];
	real3 aD3 = AD3[rigidIndex];
	real3 dist3 = posRadD[rigidMarkerIndex] - posRigidD[rigidIndex];
	real3 dist3LF = InverseRotate_By_RotationMatrix_DeviceHost(aD1, aD2, aD3, dist3);
	rigidSPH_MeshPos_LRF_D[index] = InverseRotate_By_RotationMatrix_DeviceHost(aD1, aD2, aD3, dist3);
}
//--------------------------------------------------------------------------------------------------------------------------------

__global__ void Populate_FlexSPH_MeshPos_LRF_kernel(
		real3* flexSPH_MeshPos_LRF_D,
		real3 * posRadD,
		int* flexIdentifierD,
		real_* flexParametricDistD,
		real_* ANCF_Beam_LengthD,
		int2* ANCF_ReferenceArrayNodesOnBeamsD,
		real3 * ANCF_NodesD,
		real3 * ANCF_SlopesD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numObjectsD.numFlex_SphMarkers) {
		return;
	}
	uint absMarkerIndex = index + numObjectsD.startFlexMarkers; // updatePortionD = [start, end] index of the update portion
	real_ sOverBeam = flexParametricDistD[index];
	int flexBodyIndex = flexIdentifierD[index];
	real_ lBeam = ANCF_Beam_LengthD[flexBodyIndex];
	int2 nodesInterval = ANCF_ReferenceArrayNodesOnBeamsD[flexBodyIndex];

	int indexOfClosestNode = IndexOfClosestNode(sOverBeam, lBeam, nodesInterval);

	int nNodes = nodesInterval.y - nodesInterval.x;
	real_ lE = lBeam / (nNodes - 1); //Element length
	real_ sE = fmod(sOverBeam, lE);
	real3 beamPointPos = Calc_ANCF_Point_Pos(ANCF_NodesD, ANCF_SlopesD, indexOfClosestNode, sE, lE); //interpolation using ANCF beam, cubic hermit equation

//	//ff1
//	real3 pa = ANCF_NodesD[nodesInterval.x];
//	real3 pb = ANCF_NodesD[nodesInterval.y - 1];
//	real3 r3 = normalize(pb - pa);
//	real3 beamPointPos2 = pa + dot(posRadD[absMarkerIndex] - pa, r3) * r3;

//	//ff1
//	if (length(pb - pa) > .000001) {
//		printf("midPoint %f %f %f, midPoin2 %f %f %f\n", beamPointPos.x, beamPointPos.y, beamPointPos.z, beamPointPos2.x, beamPointPos2.y, beamPointPos2.z);
//	}

	real3 dist3 = posRadD[absMarkerIndex] - beamPointPos;
	flexSPH_MeshPos_LRF_D[index] = dist3;

//	///ff1
//		 	 real_ hh = paramsD.HSML;
//		 	 printf("dist %f \n", length(dist3)/hh);
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void Populate_FlexSPH_MeshSlope_LRF_kernel(
		real3* flexSPH_MeshSlope_Initial_D,
		int* flexIdentifierD,
		real_* flexParametricDistD,
		real_* ANCF_Beam_LengthD,
		int2* ANCF_ReferenceArrayNodesOnBeamsD,
		real3 * ANCF_NodesD,
		real3 * ANCF_SlopesD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numObjectsD.numFlex_SphMarkers) {
		return;
	}
	uint absMarkerIndex = index + numObjectsD.startFlexMarkers; // updatePortionD = [start, end] index of the update portion
	real_ sOverBeam = flexParametricDistD[index];
	int flexBodyIndex = flexIdentifierD[index];
	real_ lBeam = ANCF_Beam_LengthD[flexBodyIndex];
	int2 nodesInterval = ANCF_ReferenceArrayNodesOnBeamsD[flexBodyIndex];

	int indexOfClosestNode = IndexOfClosestNode(sOverBeam, lBeam, nodesInterval);

	int nNodes = nodesInterval.y - nodesInterval.x;
	real_ lE = lBeam / (nNodes - 1); //Element length
	real_ sE = fmod(sOverBeam, lE);
	real3 beamPointSlope = Calc_ANCF_Point_Slope(ANCF_NodesD, ANCF_SlopesD, indexOfClosestNode, sE, lE); //interpolation using ANCF beam, cubic hermit equation
	flexSPH_MeshSlope_Initial_D[index] = normalize(beamPointSlope);
}

//--------------------------------------------------------------------------------------------------------------------------------
//the rigid body torque has been calculated in global RF. This kernel maps it to local RF to be appropriate for the formulas
//local torque = T' = A' * T
__global__ void MapTorqueToLRFKernel(real3 * AD1, real3 * AD2, real3 * AD3, real3 * totalTorqueOfAcc3, real3 * LF_totalTorqueOfAcc3) {
	uint rigidSphereA = blockIdx.x * blockDim.x + threadIdx.x;
	if (rigidSphereA >= numObjectsD.numRigidBodies) {
		return;
	}
	real3 totalTorqueGRF = totalTorqueOfAcc3[rigidSphereA];
	LF_totalTorqueOfAcc3[rigidSphereA] = AD1[rigidSphereA] * totalTorqueGRF.x + AD2[rigidSphereA] * totalTorqueGRF.y
			+ AD3[rigidSphereA] * totalTorqueGRF.z;
}
//--------------------------------------------------------------------------------------------------------------------------------
//updates the rigid body particles
__global__ void UpdateKernelRigidTranstalation(
		real3 * posRigidD2, real3 * posRigidCumulativeD2, real4 * velMassRigidD2, real4 * velMassRigidD, real3 * totalAccRigid3) {
	uint rigidSphereA = blockIdx.x * blockDim.x + threadIdx.x;
	if (rigidSphereA >= numObjectsD.numRigidBodies) {
		return;
	}

	real4 dummyVelMas = velMassRigidD[rigidSphereA];
	real3 deltaPos = R3(dummyVelMas) * dTD;
	posRigidD2[rigidSphereA] += deltaPos;
	posRigidCumulativeD2[rigidSphereA] += deltaPos;

	real3 derivV_SPH = totalAccRigid3[rigidSphereA]; //in fact, totalBodyForce4 is originially sum of dV/dt of sph particles and should be multiplied by m to produce force. paramsD.gravity is applied in the force kernel
	real3 deltaVel = derivV_SPH * dTD;
	velMassRigidD2[rigidSphereA] += R4(deltaVel, 0);
}
//--------------------------------------------------------------------------------------------------------------------------------
//updates the rigid body particles
__global__ void UpdateKernelRigidTranstalationBeta(
		real3 * posRigidD2, real3 * posRigidCumulativeD2, real4 * velMassRigidD2, real4 * velMassRigidD, real3 * totalAccRigid3) {
	uint rigidSphereA = blockIdx.x * blockDim.x + threadIdx.x;
	if (rigidSphereA >= numObjectsD.numRigidBodies) {
		return;
	}

	real4 dummyVelMas = velMassRigidD[rigidSphereA];
	real3 deltaPos = R3(dummyVelMas) * dTD;
	posRigidD2[rigidSphereA] += deltaPos;
	posRigidCumulativeD2[rigidSphereA] += deltaPos;

	real3 derivV_SPH = totalAccRigid3[rigidSphereA]; //in fact, totalBodyForce4 is originially sum of dV/dt of sph particles and should be multiplied by m to produce force. paramsD.gravity is applied in the force kernel
	derivV_SPH.y = 0;
	derivV_SPH.z = 0;
	real3 deltaVel = derivV_SPH * dTD;
	velMassRigidD2[rigidSphereA] += R4(deltaVel, 0);
}
//--------------------------------------------------------------------------------------------------------------------------------
//updates the rigid body Quaternion of Rotation
// A is rotation matrix, A = [AD1; AD2; AD3]
__global__ void UpdateRigidBodyQuaternion_kernel(real4 * qD2, real4 * qD, real3 * omegaLRF_D) {
	uint rigidSphereA = blockIdx.x * blockDim.x + threadIdx.x;
	if (rigidSphereA >= numObjectsD.numRigidBodies) {
		return;
	}
	real3 omega = omegaLRF_D[rigidSphereA];
	real4 q = qD[rigidSphereA];
	real4 qDot = 0.5 * (
			omega.x * R4(-(q.y), q.x, q.w, -(q.z)) + omega.y * R4(-(q.z), -(q.w), q.x, q.y) + omega.z * R4(-(q.w), q.z, -(q.y), q.x)
	);

	real4 q2 = qD2[rigidSphereA];
	q2 += dTD * qDot;
	q2 *= (1.0f / length(q2));
	qD2[rigidSphereA] = q2;
}
//--------------------------------------------------------------------------------------------------------------------------------
//updates the rigid body Rotation
// A is rotation matrix, A = [AD1; AD2; AD3], first comp of q is rotation, last 3 components are axis of rot
// in wikipedia, last quat comp is the angle, in my version, first one is the angle.
// here is the mapping between wikipedia (g) and mine (q): [gx, gy, gz, gw] = [qy, qz, qw, qx]
__global__ void RotationMatirixFromQuaternion_kernel(real3 * AD1, real3 * AD2, real3 * AD3, real4 * qD) {
	uint rigidSphereA = blockIdx.x * blockDim.x + threadIdx.x;
	if (rigidSphereA >= numObjectsD.numRigidBodies) {
		return;
	}
	real4 q = qD[rigidSphereA];
	real3 aD1, aD2, aD3;
	RotationMatirixFromQuaternion(aD1, aD2, aD3, q);
	AD1[rigidSphereA] = aD1;
	AD2[rigidSphereA] = aD2;
	AD3[rigidSphereA] = aD3;
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void UpdateRigidBodyAngularVelocity_kernel(
		real3 * omegaLRF_D2,
		real3 * LF_totalTorqueOfAcc3,
		real3 * omegaLRF_D,
		real3 * jD1,
		real3 * jD2,
		real3 * jInvD1,
		real3 * jInvD2) {
	uint rigidSphereA = blockIdx.x * blockDim.x + threadIdx.x;
	if (rigidSphereA >= numObjectsD.numRigidBodies) {
		return;
	}

	real3 omega3 = omegaLRF_D[rigidSphereA];
//	printf("1: tt %f %f %f\n", omega3.x, omega3.y, omega3.z);
	real3 j1 = jD1[rigidSphereA];
	real3 j2 = jD2[rigidSphereA];
	//printf("j j %f %f %f %f %f %f\n", j1.x, j1.y, j1.z, j2.x, j2.y, j2.z);
	real3 torquingTerm;
	torquingTerm.x = (-omega3.z * j1.y + omega3.y * j1.z) * omega3.x + (-omega3.z * j2.x + omega3.y * j2.y) * omega3.y
			+ (-omega3.z * j2.y + omega3.y * j2.z) * omega3.z;
	torquingTerm.y = (omega3.z * j1.x - omega3.x * j1.z) * omega3.x + (omega3.z * j1.y - omega3.x * j2.y) * omega3.y
			+ (omega3.z * j1.z - omega3.x * j2.z) * omega3.z;
	torquingTerm.z = (-omega3.y * j1.x + omega3.x * j1.y) * omega3.x + (-omega3.y * j1.y + omega3.x * j2.x) * omega3.y
			+ (-omega3.y * j1.z + omega3.x * j2.y) * omega3.z;


	torquingTerm = solid_SPH_massD * LF_totalTorqueOfAcc3[rigidSphereA] - torquingTerm;
	//*** from this point j1 and j2 will represent the j_Inverse
	j1 = jInvD1[rigidSphereA];
	j2 = jInvD2[rigidSphereA];
	//printf("j j %f %f %f %f %f %f\n", j1.x, j1.y, j1.z, j2.x, j2.y, j2.z);
	real3 omegaDot3 = torquingTerm.x * j1 + torquingTerm.y * R3(j1.y, j2.x, j2.y) + torquingTerm.z * R3(j1.z, j2.y, j2.z);
//	//	*** for 2D motion
//		omegaDot3.x = 0;
//		omegaDot3.z = 0;

	omegaLRF_D2[rigidSphereA] += omegaDot3 * dTD;
//	printf("2: tt %f %f %f\n", omega3.x, omega3.y, omega3.z);
}
//--------------------------------------------------------------------------------------------------------------------------------
//updates the rigid body particles
__global__ void UpdateRigidMarkersPosition(
		real3 * posRadD,
		real4 * velMasD,
		const real3 * rigidSPH_MeshPos_LRF_D,
		const int * rigidIdentifierD,
		real3 * posRigidD,
		real4 * velMassRigidD,
		real3 * omegaLRF_D,
		real3 * AD1,
		real3 * AD2,
		real3 * AD3) {

	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numObjectsD.numRigid_SphMarkers) {
		return;
	}
	uint rigidMarkerIndex = index + numObjectsD.startRigidMarkers; // updatePortionD = [start, end] index of the update portion
	int rigidBodyIndex = rigidIdentifierD[index];

	real3 a1, a2, a3;
	a1 = AD1[rigidBodyIndex];
	a2 = AD2[rigidBodyIndex];
	a3 = AD3[rigidBodyIndex];

	real3 rigidSPH_MeshPos_LRF = rigidSPH_MeshPos_LRF_D[index];

	//position
	real3 p_Rigid = posRigidD[rigidBodyIndex];
	posRadD[rigidMarkerIndex] = p_Rigid + R3(dot(a1, rigidSPH_MeshPos_LRF), dot(a2, rigidSPH_MeshPos_LRF), dot(a3, rigidSPH_MeshPos_LRF));

	//velociy
	real4 vM_Rigid = velMassRigidD[rigidBodyIndex];
	real3 omega3 = omegaLRF_D[rigidBodyIndex];
	real3 omegaCrossS = cross(omega3, rigidSPH_MeshPos_LRF);
	real4 vM = velMasD[rigidMarkerIndex];
	velMasD[rigidMarkerIndex] = R4(R3(vM_Rigid) + R3(dot(a1, omegaCrossS), dot(a2, omegaCrossS), dot(a3, omegaCrossS)), vM.w);
}
//--------------------------------------------------------------------------------------------------------------------------------
//updates the flex body markers
__global__ void UpdateFlexMarkersPosition(
		real3 * posRadD,
		real4 * velMasD,
		int* flexIdentifierD,
		real3* flexSPH_MeshPos_LRF_D,
		real3* flexSPH_MeshSlope_Initial_D,
		real_* flexParametricDistD,
		real_* ANCF_Beam_LengthD,
		int2* ANCF_ReferenceArrayNodesOnBeamsD,
		real3 * ANCF_NodesD,
		real3 * ANCF_SlopesD,
		real3 * ANCF_NodesVelD,
		real3 * ANCF_SlopesVelD) {

	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numObjectsD.numFlex_SphMarkers) {
		return;
	}
	uint absMarkerIndex = index + numObjectsD.startFlexMarkers; // updatePortionD = [start, end] index of the update portion

	real_ sOverBeam = flexParametricDistD[index];
	int flexBodyIndex = flexIdentifierD[index];
	real_ lBeam = ANCF_Beam_LengthD[flexBodyIndex];
	int2 nodesInterval = ANCF_ReferenceArrayNodesOnBeamsD[flexBodyIndex];
	int indexOfClosestNode = IndexOfClosestNode(sOverBeam, lBeam, nodesInterval);

	int nNodes = nodesInterval.y - nodesInterval.x;
	real_ lE = lBeam / (nNodes - 1); //Element length
	real_ sE = fmod(sOverBeam, lE);

//		real3 pa = ANCF_NodesD[indexOfClosestNode];
//		real3 pb = ANCF_NodesD[indexOfClosestNode + 1];
//		printf(" pa %f %f %f\n pb %f %f %f, nodes interval %d %d\n\n", pa.x, pa.y, pa.z, pb.x, pb.y, pb.z, nodesInterval.x, nodesInterval.y);

	real3 rX = Calc_ANCF_Point_Slope(ANCF_NodesD, ANCF_SlopesD, indexOfClosestNode, sE, lE); //interpolation using ANCF beam, cubic hermit equation
	real3 beamPointSlope = normalize(rX);

	real3 dist3 = flexSPH_MeshPos_LRF_D[index];
//		real3 sphPoint = posRadD[absMarkerIndex];
//		printf(" pa %f %f %f\n pm %f %f %f\n pb %f %f %f\n\n", beamPointPos.x, beamPointPos.y, beamPointPos.z, dist3.x, dist3.y, dist3.z, sphPoint.x, sphPoint.y, sphPoint.z);
	real3 beamPointSlopeInitial = flexSPH_MeshSlope_Initial_D[index];

	real_ cosTheta = dot(beamPointSlopeInitial, beamPointSlope);
	if(cosTheta>1){
		cosTheta=1;
	} else if(cosTheta<-1){
		cosTheta=-1;
	}
//	cosTheta *= rminr(.99999999999999999, .99999999999999999/fabs(cosTheta));  //to take care of numerical error and |cosTheta| > 1 situations

	real_ theta = acos(cosTheta);
	real3 n3;
	if (fabs(theta) > 1e-6) {
		n3 = cross(beamPointSlopeInitial, beamPointSlope);
		n3 = normalize(n3);
	} else {
		n3 = R3(1, 0, 0); //does not really matter, it rotates as much as theta almost equal to zero
	}
	real4 q = R4(cos(0.5 * theta),
			n3.x * sin(0.5 * theta), n3.y * sin(0.5 * theta), n3.z * sin(0.5 * theta));
	real3 A1, A2, A3;

//	printf("theta %f q %f %f %f %f\n", theta, q.x, q.y, q.z, q.w);

	RotationMatirixFromQuaternion(A1, A2, A3, q);
//	printf("theta %f \nA1 %f %f %f \nA2 %f %f %f \nA3 %f %f %f\n\n\n", theta, A1.x, A1.y, A1.z, A2.x, A2.y, A2.z, A3.x, A3.y, A3.z);

//		real3 p1 = posRadD[absMarkerIndex];
//		real3 p2 = beamPointPos + R3(dot(A1, dist3), dot(A2, dist3), dot(A3, dist3));
//		real3 pdiff = dist3;// - p2;
//		printf("length p1 %f length p2 %f length dist3 %f theta %f cosTheta %f beamPointSlopeInitial %f beamPointSlope %f\n", length(p1), length(p2), length(dist3), theta, cosTheta, length(beamPointSlopeInitial), length(beamPointSlope));
//		if (length(pdiff) > 1e-8) {
//			printf("diff of calc and real %f %f %f\n", pdiff.x, pdiff.y, pdiff.z);
//		}

//		real3 pSPH = posRadD[absMarkerIndex];
//		real3 pa = ANCF_NodesD[0];
//		real3 pb = ANCF_NodesD[3];
//		real3 slope3 = normalize(pb-pa);
//		printf("pb-pa %f %f %f , beamPointSlope %f %f %f \n", slope3.x, slope3.y, slope3.z, beamPointSlope.x, beamPointSlope.y, beamPointSlope.z);
//		real3 r = pSPH - pa;
//		beamPointPos = pa + dot(beamPointSlopeInitial, r) * beamPointSlopeInitial;

//	printf("beamPointPos %f %f %f \n", beamPointPos.x, beamPointPos.y, beamPointPos.z);

//		///ff1
//	 	 real_ hh = paramsD.HSML;
//	 	 printf("dist %f \n", length(dist3)/hh);

	real3 beamPointPos = Calc_ANCF_Point_Pos(ANCF_NodesD, ANCF_SlopesD, indexOfClosestNode, sE, lE); //interpolation using ANCF beam, cubic hermit equation
//	//ff1
//	real3 newPos = beamPointPos + R3(dot(A1, dist3), dot(A2, dist3), dot(A3, dist3));
//	real3 oldPos = posRadD[absMarkerIndex];
//	if (length(newPos- oldPos) > .001 * paramsD.HSML) {
//		printf("pos %f %f %f and newPos %f %f %f \n", oldPos.x, oldPos.y, oldPos.z, newPos.x, newPos.y, newPos.z);
//	}

	posRadD[absMarkerIndex] = beamPointPos + R3(dot(A1, dist3), dot(A2, dist3), dot(A3, dist3));

//	//ask Radu
	real_ markerMass = velMasD[absMarkerIndex].w;
	real3 beamPointVel = Calc_ANCF_Point_Vel(ANCF_NodesVelD, ANCF_SlopesVelD, indexOfClosestNode, sE, lE); //interpolation using ANCF beam, cubic hermit equation

	real3 absOmega = Calc_ANCF_Point_Omega(ANCF_NodesVelD, ANCF_SlopesVelD, indexOfClosestNode, sE, lE, rX); //interpolation using ANCF beam, cubic hermit equation
		//ff1
	//	velMasD[absMarkerIndex] = R4(beamPointVel + cross(absOmega, dist3), markerMass); //wrong
	velMasD[absMarkerIndex] = R4(beamPointVel, markerMass);
}
////--------------------------------------------------------------------------------------------------------------------------------
void MakeRigidIdentifier(
		thrust::device_vector<int> & rigidIdentifierD,
		int numRigidBodies, int startRigidMarkers, const thrust::host_vector<int3> & referenceArray)
{
	if (numRigidBodies > 0) {
		for (int rigidSphereA = 0; rigidSphereA < numRigidBodies; rigidSphereA++) {
			int3 referencePart = referenceArray[2 + rigidSphereA];
			if (referencePart.z != 1) {
				printf("error in accessing rigid bodies. Reference array indexing is wrong\n");
				return;
			}
			int2 updatePortion = I2(referencePart); //first two component of the referenceArray denote to the fluid and boundary particles
			thrust::fill(rigidIdentifierD.begin() + (updatePortion.x - startRigidMarkers),
					rigidIdentifierD.begin() + (updatePortion.y - startRigidMarkers), rigidSphereA);
		}
	}
}
////--------------------------------------------------------------------------------------------------------------------------------

////; flexIdentifier is not of the size of total flex bodies. Here, apparently, it is of the size of total markers!!!!!!!!!!!!!!!!!!!!!!!!
void MakeFlexIdentifier(
		thrust::device_vector<int> & flexIdentifierD,
		int numFlexBodies, int numFlBcRigid, int startFlexMarkers, const thrust::host_vector<int3> & referenceArray)
{
	if (numFlexBodies > 0) {
		for (int flexIdx = 0; flexIdx < numFlexBodies; flexIdx++) {
			int3 referencePart = referenceArray[numFlBcRigid + flexIdx];
			if (referencePart.z != 2) {
				printf("error in accessing flex bodies. Reference array indexing is wrong\n");
				return;
			}
			int2 updatePortion = I2(referencePart); //first two component of the referenceArray denote to the fluid and boundary particles
			thrust::fill(flexIdentifierD.begin() + (updatePortion.x - startFlexMarkers),
					flexIdentifierD.begin() + (updatePortion.y - startFlexMarkers), flexIdx);
		}
	}
}
////--------------------------------------------------------------------------------------------------------------------------------
void Calc_NumNodesMultMarkers_Per_Beam(
		thrust::device_vector<int> & ANCF_NumNodesMultMarkers_Per_BeamD,
		const thrust::device_vector<int> & ANCF_NumMarkers_Per_BeamD,
		const thrust::host_vector<int2> & ANCF_ReferenceArrayNodesOnBeams,
		int numFlexBodies)
{
	if (numFlexBodies > 0) {
		for (int flexIdx = 0; flexIdx < numFlexBodies; flexIdx++) {
			int2 flexPortion = ANCF_ReferenceArrayNodesOnBeams[flexIdx];
			int numNodes = flexPortion.y - flexPortion.x;
			ANCF_NumNodesMultMarkers_Per_BeamD[flexIdx] = numNodes * ANCF_NumMarkers_Per_BeamD[flexIdx];
		}
	}
}
////--------------------------------------------------------------------------------------------------------------------------------
void Calc_mapEachMarkerOnAllBeamNodes_IdentifierD(
		thrust::device_vector<int2> & flexMapEachMarkerOnAllBeamNodesD,
		const thrust::device_vector<int> & ANCF_NumNodesMultMarkers_Per_Beam_CumulD,
		const thrust::device_vector<int> & ANCF_NumMarkers_Per_BeamD,
		const thrust::host_vector<int2> & ANCF_ReferenceArrayNodesOnBeams,
		int numFlexBodies)
{
	if (numFlexBodies > 0) {
		for (int flexIdx = 0; flexIdx < numFlexBodies; flexIdx++) {
			int numMarkersOnThisBeam = ANCF_NumMarkers_Per_BeamD[flexIdx];
			int2 flexPortion = ANCF_ReferenceArrayNodesOnBeams[flexIdx];
			int numNodes = flexPortion.y - flexPortion.x;

			int startWrite = ANCF_NumNodesMultMarkers_Per_Beam_CumulD[flexIdx];
			for (int i = 0; i < numNodes; i++) {
				int2 flexIdx_nodeIdx_pair = I2(flexIdx, i);
				int2 writeInterval = I2(startWrite + i * numMarkersOnThisBeam, startWrite + (i + 1) * numMarkersOnThisBeam);
				thrust::fill(flexMapEachMarkerOnAllBeamNodesD.begin() + writeInterval.x,
						flexMapEachMarkerOnAllBeamNodesD.begin() + writeInterval.y, flexIdx_nodeIdx_pair);
			}
		}
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
void MapSPH_ToGrid(
		real_ resolution,
		int3 & cartesianGridDims,
		thrust::host_vector<real4> & rho_Pres_CartH,
		thrust::host_vector<real4> & vel_VelMag_CartH,
		thrust::device_vector<real3> & posRadD,
		thrust::device_vector<real4> & velMasD,
		thrust::device_vector<real4> & rhoPresMuD,
		int numAllMarkers,
		SimParams paramsH) {
//	real3* m_dSortedPosRad;
//	real4* m_dSortedVelMas;
//	real4* m_dSortedRhoPreMu;
//	uint* m_dCellStart; // index of start of each cell in sorted list
//	uint* m_dCellEnd; // index of end of cell

	int3 SIDE = paramsH.gridSize;
	uint m_numGridCells = SIDE.x * SIDE.y * SIDE.z; //m_gridSize = SIDE
	//TODO here

	// calculate grid hash
	thrust::device_vector<real3> m_dSortedPosRad(numAllMarkers);
	thrust::device_vector<real4> m_dSortedVelMas(numAllMarkers);
	thrust::device_vector<real4> m_dSortedRhoPreMu(numAllMarkers);

	thrust::device_vector<uint> m_dGridMarkerHash(numAllMarkers);
	thrust::device_vector<uint> m_dGridMarkerIndex(numAllMarkers);

	thrust::device_vector<uint> m_dCellStart(m_numGridCells);
	thrust::device_vector<uint> m_dCellEnd(m_numGridCells);

	// calculate grid hash
	calcHash(U1CAST(m_dGridMarkerHash), U1CAST(m_dGridMarkerIndex), R3CAST(posRadD), numAllMarkers);

	thrust::sort_by_key(m_dGridMarkerHash.begin(), m_dGridMarkerHash.end(), m_dGridMarkerIndex.begin());

	// reorder particle arrays into sorted order and find start and end of each cell
	reorderDataAndFindCellStart(U1CAST(m_dCellStart), U1CAST(m_dCellEnd), R3CAST(m_dSortedPosRad), R4CAST(m_dSortedVelMas), R4CAST(m_dSortedRhoPreMu), U1CAST(m_dGridMarkerHash),
			U1CAST(m_dGridMarkerIndex), TCAST(posRadD), R4CAST(velMasD), R4CAST(rhoPresMuD), numAllMarkers, m_numGridCells);

	//real_ resolution = 8 * paramsH.markerRadius;
	cartesianGridDims = I3(paramsH.boxDims / resolution) + I3(1);
//	printf("^^^ bodDim %f %f %f, GridDim %d %d %d, resolution %f \n", paramsH.boxDims.x, paramsH.boxDims.y, paramsH.boxDims.z, cartesianGridDims.x,
//			cartesianGridDims.y, cartesianGridDims.z, resolution);
	uint cartesianGridSize = cartesianGridDims.x * cartesianGridDims.y * cartesianGridDims.z;
	thrust::device_vector<real4> rho_Pres_CartD(cartesianGridSize);
	thrust::device_vector<real4> vel_VelMag_CartD(cartesianGridSize);

	CalcCartesianData(R4CAST(rho_Pres_CartD), R4CAST(vel_VelMag_CartD), R3CAST(m_dSortedPosRad), R4CAST(m_dSortedVelMas), R4CAST(m_dSortedRhoPreMu),
			U1CAST(m_dGridMarkerIndex), U1CAST(m_dCellStart), U1CAST(m_dCellEnd), cartesianGridSize, cartesianGridDims, resolution);

//	freeArray(m_dSortedPosRad);
//	freeArray(m_dSortedVelMas);
//	freeArray(m_dSortedRhoPreMu);
	m_dSortedPosRad.clear();
	m_dSortedVelMas.clear();
	m_dSortedRhoPreMu.clear();

	m_dGridMarkerHash.clear();
	m_dGridMarkerIndex.clear();

//	freeArray(m_dCellStart);
//	freeArray(m_dCellEnd);
	m_dCellStart.clear();
	m_dCellEnd.clear();

	rho_Pres_CartH.resize(cartesianGridSize);
	vel_VelMag_CartH.resize(cartesianGridSize);
	thrust::copy(rho_Pres_CartD.begin(), rho_Pres_CartD.end(), rho_Pres_CartH.begin());
	thrust::copy(vel_VelMag_CartD.begin(), vel_VelMag_CartD.end(), vel_VelMag_CartH.begin());

	rho_Pres_CartD.clear();
	vel_VelMag_CartD.clear();
}

//*******************************************************************************************************************************
//builds the neighbors' list of each particle and finds the force on each particle
//calculates the interaction force between 1- fluid-fluid, 2- fluid-solid, 3- solid-fluid particles
//calculates forces from other SPH or solid particles, as wall as boundaries
void ForceSPH(
		thrust::device_vector<real3> & posRadD,
		thrust::device_vector<real4> & velMasD,
		thrust::device_vector<real3> & vel_XSPH_D,
		thrust::device_vector<real4> & rhoPresMuD,

		const thrust::device_vector<real3> & posRigidD,
		const thrust::device_vector<int> & rigidIdentifierD,

		thrust::device_vector<uint> & bodyIndexD,
		thrust::device_vector<real4> & derivVelRhoD,
		const thrust::host_vector<int3> & referenceArray,
		int numAllMarkers,
		SimParams paramsH,
		real_ dT) {
	// Part1: contact detection #########################################################################################################################
	// grid data for sorting method
//	real3* m_dSortedPosRad;
//	real4* m_dSortedVelMas;
//	real4* m_dSortedRhoPreMu;
//	uint* m_dCellStart; // index of start of each cell in sorted list
//	uint* m_dCellEnd; // index of end of cell

	uint m_numGridCells = paramsH.gridSize.x * paramsH.gridSize.y * paramsH.gridSize.z; //m_gridSize = SIDE
	//TODO here

	// calculate grid hash
	thrust::device_vector<real3> m_dSortedPosRad(numAllMarkers);
	thrust::device_vector<real4> m_dSortedVelMas(numAllMarkers);
	thrust::device_vector<real4> m_dSortedRhoPreMu(numAllMarkers);
	thrust::device_vector<real3> vel_XSPH_Sorted_D(numAllMarkers);

	thrust::device_vector<uint> m_dGridMarkerHash(numAllMarkers);
	thrust::device_vector<uint> m_dGridMarkerIndex(numAllMarkers);

	thrust::device_vector<uint> m_dCellStart(m_numGridCells);
	thrust::device_vector<uint> m_dCellEnd(m_numGridCells);
	// calculate grid hash
	calcHash(U1CAST(m_dGridMarkerHash), U1CAST(m_dGridMarkerIndex), R3CAST(posRadD), numAllMarkers);

//	GpuTimer myT0;
//	myT0.Start();
	thrust::sort_by_key(m_dGridMarkerHash.begin(), m_dGridMarkerHash.end(), m_dGridMarkerIndex.begin());
//	myT0.Stop();
//	real_ t0 = (real_)myT0.Elapsed();
//	printf("(0) ** Sort by key timer %f, array size %d\n", t0, m_dGridMarkerHash.size());


	// reorder particle arrays into sorted order and find start and end of each cell
	reorderDataAndFindCellStart(U1CAST(m_dCellStart), U1CAST(m_dCellEnd), R3CAST(m_dSortedPosRad), R4CAST(m_dSortedVelMas), R4CAST(m_dSortedRhoPreMu), U1CAST(m_dGridMarkerHash),
			U1CAST(m_dGridMarkerIndex), TCAST(posRadD), R4CAST(velMasD), R4CAST(rhoPresMuD), numAllMarkers, m_numGridCells);

	//process collisions
	real4 totalFluidBodyForce4 = paramsH.bodyForce4 + R4(paramsH.gravity);
	thrust::fill(derivVelRhoD.begin(), derivVelRhoD.end(), R4(0)); //initialize derivVelRhoD with zero. necessary
//	GpuTimer myT1;
//	myT1.Start();
	thrust::fill(derivVelRhoD.begin() + referenceArray[0].x, derivVelRhoD.begin() + referenceArray[0].y, totalFluidBodyForce4); //add body force to fluid particles.
//	myT1.Stop();
//	real_ t1 = (real_)myT1.Elapsed();
//	printf("(1) *** fill timer %f, array size %d\n", t1, referenceArray[0].y - referenceArray[0].x);

	RecalcVelocity_XSPH(R3CAST(vel_XSPH_Sorted_D), R3CAST(m_dSortedPosRad), R4CAST(m_dSortedVelMas), R4CAST(m_dSortedRhoPreMu), U1CAST(m_dGridMarkerIndex), U1CAST(m_dCellStart),
			U1CAST(m_dCellEnd), numAllMarkers, m_numGridCells);

	collide(R4CAST(derivVelRhoD), R3CAST(m_dSortedPosRad), R4CAST(m_dSortedVelMas), R3CAST(vel_XSPH_Sorted_D), R4CAST(m_dSortedRhoPreMu), R3CAST(posRigidD), I1CAST(rigidIdentifierD), U1CAST(m_dGridMarkerIndex), U1CAST(m_dCellStart),
			U1CAST(m_dCellEnd), numAllMarkers, m_numGridCells, dT);


	uint nBlock_NumSpheres, nThreads_SphMarkers;
	computeGridSize(numAllMarkers, 256, nBlock_NumSpheres, nThreads_SphMarkers);
	Copy_SortedVelXSPH_To_VelXSPH<<<nBlock_NumSpheres, nThreads_SphMarkers>>>(R3CAST(vel_XSPH_D), R3CAST(vel_XSPH_Sorted_D), U1CAST(m_dGridMarkerIndex));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: Copy_SortedVelXSPH_To_VelXSPH");

	////
	m_dSortedPosRad.clear();
	m_dSortedVelMas.clear();
	m_dSortedRhoPreMu.clear();
	vel_XSPH_Sorted_D.clear();

	m_dGridMarkerHash.clear();
	m_dGridMarkerIndex.clear();

	m_dCellStart.clear();
	m_dCellEnd.clear();
}
//--------------------------------------------------------------------------------------------------------------------------------
void DensityReinitialization(
		thrust::device_vector<real3> & posRadD,
		thrust::device_vector<real4> & velMasD,
		thrust::device_vector<real4> & rhoPresMuD,
		int numAllMarkers,
		int3 SIDE) {
//	real3* m_dSortedPosRad;
//	real4* m_dSortedVelMas;
//	real4* m_dSortedRhoPreMu;
//	uint* m_dCellStart; // index of start of each cell in sorted list
//	uint* m_dCellEnd; // index of end of cell

	uint m_numGridCells = SIDE.x * SIDE.y * SIDE.z; //m_gridSize = SIDE
	//TODO here

	// calculate grid hash
	thrust::device_vector<real3> m_dSortedPosRad(numAllMarkers);
	thrust::device_vector<real4> m_dSortedVelMas(numAllMarkers);
	thrust::device_vector<real4> m_dSortedRhoPreMu(numAllMarkers);

	thrust::device_vector<uint> m_dGridMarkerHash(numAllMarkers);
	thrust::device_vector<uint> m_dGridMarkerIndex(numAllMarkers);

	thrust::device_vector<uint> m_dCellStart(m_numGridCells);
	thrust::device_vector<uint> m_dCellEnd(m_numGridCells);

	// calculate grid hash
	calcHash(U1CAST(m_dGridMarkerHash), U1CAST(m_dGridMarkerIndex), R3CAST(posRadD), numAllMarkers);

	thrust::sort_by_key(m_dGridMarkerHash.begin(), m_dGridMarkerHash.end(), m_dGridMarkerIndex.begin());

	// reorder particle arrays into sorted order and find start and end of each cell
	reorderDataAndFindCellStart(U1CAST(m_dCellStart), U1CAST(m_dCellEnd), R3CAST(m_dSortedPosRad), R4CAST(m_dSortedVelMas), R4CAST(m_dSortedRhoPreMu), U1CAST(m_dGridMarkerHash),
			U1CAST(m_dGridMarkerIndex), TCAST(posRadD), R4CAST(velMasD), R4CAST(rhoPresMuD), numAllMarkers, m_numGridCells);

	ReCalcDensity(R3CAST(posRadD), R4CAST(velMasD), R4CAST(rhoPresMuD), R3CAST(m_dSortedPosRad), R4CAST(m_dSortedVelMas), R4CAST(m_dSortedRhoPreMu),
			U1CAST(m_dGridMarkerIndex), U1CAST(m_dCellStart), U1CAST(m_dCellEnd), numAllMarkers, m_numGridCells);

	m_dSortedPosRad.clear();
	m_dSortedVelMas.clear();
	m_dSortedRhoPreMu.clear();

	m_dGridMarkerHash.clear();
	m_dGridMarkerIndex.clear();

	m_dCellStart.clear();
	m_dCellEnd.clear();
}
//--------------------------------------------------------------------------------------------------------------------------------
//updates the fluid particles by calling UpdateKernelFluid 
void UpdateFluid(
		thrust::device_vector<real3> & posRadD,
		thrust::device_vector<real4> & velMasD,
		thrust::device_vector<real3> & vel_XSPH_D,
		thrust::device_vector<real4> & rhoPresMuD,
		thrust::device_vector<real4> & derivVelRhoD,
		const thrust::host_vector<int3> & referenceArray,
		real_ dT) {
	int3 referencePortion = referenceArray[0];
	if (referencePortion.z != -1) {
		printf("error in UpdateFluid, accessing non fluid\n");
		return;
	}
	int2 updatePortion = I2(referencePortion);
	//int2 updatePortion = I2(referenceArray[0].x, referenceArray[0].y);
	hipMemcpyToSymbolAsync(HIP_SYMBOL(dTD), &dT, sizeof(dT));
	hipMemcpyToSymbolAsync(HIP_SYMBOL(updatePortionD), &updatePortion, sizeof(updatePortion));

	uint nBlock_UpdateFluid, nThreads;
	computeGridSize(updatePortion.y - updatePortion.x, 128, nBlock_UpdateFluid, nThreads);
	UpdateKernelFluid<<<nBlock_UpdateFluid, nThreads>>>(R3CAST(posRadD), R4CAST(velMasD), R3CAST(vel_XSPH_D), R4CAST(rhoPresMuD), R4CAST(derivVelRhoD));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: UpdateKernelFluid");
}
//--------------------------------------------------------------------------------------------------------------------------------
//updates the fluid particles by calling UpdateBoundary
void UpdateBoundary(
		thrust::device_vector<real3> & posRadD,
		thrust::device_vector<real4> & velMasD,
		thrust::device_vector<real4> & rhoPresMuD,
		thrust::device_vector<real4> & derivVelRhoD,
		const thrust::host_vector<int3> & referenceArray,
		real_ dT) {
	int3 referencePortion = referenceArray[1];
	if (referencePortion.z != 0) {
		printf("error in UpdateBoundary, accessing non boundary\n");
		return;
	}
	int2 updatePortion = I2(referencePortion);
	hipMemcpyToSymbolAsync(HIP_SYMBOL(dTD), &dT, sizeof(dT));
	hipMemcpyToSymbolAsync(HIP_SYMBOL(updatePortionD), &updatePortion, sizeof(updatePortion));

	uint nBlock_UpdateFluid, nThreads;
	computeGridSize(updatePortion.y - updatePortion.x, 128, nBlock_UpdateFluid, nThreads);
	UpdateKernelBoundary<<<nBlock_UpdateFluid, nThreads>>>(R3CAST(posRadD), R4CAST(velMasD), R4CAST(rhoPresMuD), R4CAST(derivVelRhoD));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: UpdateKernelBoundary");
}
//--------------------------------------------------------------------------------------------------------------------------------
void ApplyBoundarySPH_Markers(
		thrust::device_vector<real3> & posRadD,
		thrust::device_vector<real4> & rhoPresMuD,
		int numAllMarkers) {
	uint nBlock_NumSpheres, nThreads_SphMarkers;
	computeGridSize(numAllMarkers, 256, nBlock_NumSpheres, nThreads_SphMarkers);
	ApplyPeriodicBoundaryXKernel<<<nBlock_NumSpheres, nThreads_SphMarkers>>>(R3CAST(posRadD), R4CAST(rhoPresMuD));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: ApplyPeriodicBoundaryXKernel");
	// these are useful anyway for out of bound particles
	ApplyPeriodicBoundaryYKernel<<<nBlock_NumSpheres, nThreads_SphMarkers>>>(R3CAST(posRadD), R4CAST(rhoPresMuD));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: ApplyPeriodicBoundaryYKernel");
	ApplyPeriodicBoundaryZKernel<<<nBlock_NumSpheres, nThreads_SphMarkers>>>(R3CAST(posRadD), R4CAST(rhoPresMuD));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: ApplyPeriodicBoundaryZKernel");
}
//--------------------------------------------------------------------------------------------------------------------------------
void ApplyBoundaryRigid(
		thrust::device_vector<real3> & posRigidD,
		int numRigidBodies) {
	uint nBlock_NumRigids, nThreads_RigidBodies;
	computeGridSize(numRigidBodies, 128, nBlock_NumRigids, nThreads_RigidBodies);
	ApplyPeriodicBoundaryXKernel_RigidBodies<<<nBlock_NumRigids, nThreads_RigidBodies>>>(R3CAST(posRigidD));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: UpdateKernelRigid");
	ApplyPeriodicBoundaryYKernel_RigidBodies<<<nBlock_NumRigids, nThreads_RigidBodies>>>(R3CAST(posRigidD));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: UpdateKernelRigid");
	ApplyPeriodicBoundaryZKernel_RigidBodies<<<nBlock_NumRigids, nThreads_RigidBodies>>>(R3CAST(posRigidD));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: UpdateKernelRigid");
}

//--------------------------------------------------------------------------------------------------------------------------------
void ApplyBoundaryFlex(
		thrust::device_vector<real3> & ANCF_NodesD,
		thrust::device_vector<int2> & ANCF_ReferenceArrayNodesOnBeamsD,
		int numFlex) {
	uint nBlock_NumFlex, nThreads_Flex;
	computeGridSize(numFlex, 128, nBlock_NumFlex, nThreads_Flex);
	ApplyPeriodicBoundaryKernel_FlexBodies<<<nBlock_NumFlex, nThreads_Flex>>>(R3CAST(ANCF_NodesD), I2CAST(ANCF_ReferenceArrayNodesOnBeamsD)); // x,y,z all implemented in a single kernel
	hipDeviceSynchronize();
}
//--------------------------------------------------------------------------------------------------------------------------------
void ApplyBoundary(
		thrust::device_vector<real3> & posRadD,
		thrust::device_vector<real4> & rhoPresMuD,
		thrust::device_vector<real3> & posRigidD,
		thrust::device_vector<real3> & ANCF_NodesD,
		thrust::device_vector<int2> & ANCF_ReferenceArrayNodesOnBeamsD,
		NumberOfObjects numObjects) {

	ApplyBoundarySPH_Markers(posRadD, rhoPresMuD, numObjects.numAllMarkers);
	ApplyBoundaryRigid(posRigidD, numObjects.numRigidBodies);
	ApplyBoundaryFlex(ANCF_NodesD, ANCF_ReferenceArrayNodesOnBeamsD, numObjects.numFlexBodies);

}
//--------------------------------------------------------------------------------------------------------------------------------
void FindPassesFromTheEnd(
		thrust::device_vector<real3> & posRigidD,
		thrust::device_vector<int> & distributionD,
		int numRigidBodies,
		real2 pipeCenter,
		real_ pipeRadius,
		int numberOfSections) {
//	real3 posRigid = posRigidD[0];
//	printf("xRigid %f\n", posRadRigid.x);cutil_math deprecate
	real_ dR = pipeRadius / numberOfSections;
	thrust::device_vector<uint> radialPositions(numRigidBodies);
	thrust::device_vector<uint> radialPosCounter(numRigidBodies);
	thrust::fill(radialPositions.begin(), radialPositions.end(), 10000); //10000 as a large number
	thrust::fill(radialPosCounter.begin(), radialPosCounter.end(), 0);

	uint nBlock_NumRigids, nThreads_RigidBodies;
	computeGridSize(numRigidBodies, 128, nBlock_NumRigids, nThreads_RigidBodies);
	PassesFromTheEnd_Kernel<<<nBlock_NumRigids, nThreads_RigidBodies>>>(R3CAST(posRigidD), U1CAST(radialPositions), U1CAST(radialPosCounter), pipeCenter, dR);
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: PassesFromTheEnd_Kernel");

	thrust::sort_by_key(radialPositions.begin(), radialPositions.end(), radialPosCounter.begin());
	thrust::device_vector<uint> radialPosCounter_Cumulative(numberOfSections + 2); //+2 for safety, specially when the particle goes outside of the pipe
	thrust::device_vector<uint> dummy_radialPosition(numberOfSections + 2);
	(void) thrust::reduce_by_key(radialPositions.begin(), radialPositions.end(), radialPosCounter.begin(), dummy_radialPosition.begin(),
			radialPosCounter_Cumulative.begin());
//	radialPosCounter_Cumulative.resize(numberOfSections);
//	dummy_radialPosition.resize(numberOfSections);

	//printf("%$%$%$%$%$%$ dummy_radialPosition[0] %d")

	uint nBlock_NumSections, nThreads_numSections;
	computeGridSize(numberOfSections, 128, nBlock_NumSections, nThreads_numSections);
	AddToCumulutaiveNumberOfPasses<<<nBlock_NumSections, nThreads_numSections>>>(I1CAST(distributionD), U1CAST(dummy_radialPosition), U1CAST(radialPosCounter_Cumulative), numberOfSections);
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: AddToCumulutaiveNumberOfPasses");

	radialPosCounter_Cumulative.clear();
	dummy_radialPosition.clear();
	radialPositions.clear();
	radialPosCounter.clear();
}
//--------------------------------------------------------------------------------------------------------------------------------
// applies the time step to the current quantities and saves the new values into variable with the same name and '2' and the end
// precondition: for the first step of RK2, all variables with '2' at the end have the values the same as those without '2' at the end.
void UpdateRigidBody(
		thrust::device_vector<real3> & posRadD2,
		thrust::device_vector<real4> & velMasD2,
		thrust::device_vector<real3> & posRigidD2,
		thrust::device_vector<real3> & posRigidCumulativeD2,
		thrust::device_vector<real4> & velMassRigidD2,
		thrust::device_vector<real4> & qD2,
		thrust::device_vector<real3> & AD1_2,
		thrust::device_vector<real3> & AD2_2,
		thrust::device_vector<real3> & AD3_2,
		thrust::device_vector<real3> & omegaLRF_D2,

		thrust::device_vector<real3> & posRadD,
		const thrust::device_vector<real3> & posRigidD,
		const thrust::device_vector<real3> & posRigidCumulativeD,
		const thrust::device_vector<real4> & velMassRigidD,
		const thrust::device_vector<real4> & qD,
		const thrust::device_vector<real3> & AD1,
		const thrust::device_vector<real3> & AD2,
		const thrust::device_vector<real3> & AD3,
		const thrust::device_vector<real3> & omegaLRF_D,

		const thrust::device_vector<real4> & derivVelRhoD,
		const thrust::device_vector<int> & rigidIdentifierD,

		const thrust::device_vector<real3> & rigidSPH_MeshPos_LRF_D,
		const thrust::host_vector<int3> & referenceArray,
		const thrust::device_vector<real3> & jD1,
		const thrust::device_vector<real3> & jD2,
		const thrust::device_vector<real3> & jInvD1,
		const thrust::device_vector<real3> & jInvD2,
		real3 gravity,
		NumberOfObjects numObjects,
		float fracSimulation,
		real_ dT) {
	if (numObjects.numRigidBodies == 0) {
		return;
	}
	hipMemcpyToSymbolAsync(HIP_SYMBOL(dTD), &dT, sizeof(dT));
	//################################################### make force and torque arrays
	//####### Force (Acceleration)
	thrust::device_vector<real4> totalSurfaceInteractionRigid4(numObjects.numRigidBodies);
	thrust::device_vector<real3> totalTorqueOfAcc3(numObjects.numRigidBodies);
	thrust::fill(totalSurfaceInteractionRigid4.begin(), totalSurfaceInteractionRigid4.end(), R4(0));
	thrust::device_vector<int> dummyIdentify(numObjects.numRigidBodies);
	thrust::equal_to<int> binary_pred;

	//** forces on BCE markers of each rigid body are accumulated at center. "totalSurfaceInteractionRigid4" is got built.
	(void) thrust::reduce_by_key(rigidIdentifierD.begin(), rigidIdentifierD.end(), derivVelRhoD.begin() + numObjects.startRigidMarkers, dummyIdentify.begin(),
			totalSurfaceInteractionRigid4.begin(), binary_pred, thrust::plus<real4>());
	thrust::device_vector<real3> totalAccRigid3(numObjects.numRigidBodies);
	thrust::fill(totalAccRigid3.begin(), totalAccRigid3.end(), R3(0));

	uint nBlock_UpdateRigid;
	uint nThreads_rigidParticles;
	computeGridSize(numObjects.numRigidBodies, 128, nBlock_UpdateRigid, nThreads_rigidParticles);

	//** accumulated BCE forces at center are transformed to acceleration of rigid body "totalAccRigid3". "totalAccRigid3" gets built.
	Calc_SurfaceInducedAcceleration<<<nBlock_UpdateRigid, nThreads_rigidParticles>>>(
			R3CAST(totalAccRigid3), R4CAST(totalSurfaceInteractionRigid4), R4CAST(velMassRigidD));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: Calc_SurfaceInducedAcceleration");
	totalSurfaceInteractionRigid4.clear();

	//add gravity from flex
	thrust::device_vector<real3> gravityForces3(numObjects.numRigidBodies);

	thrust::fill(gravityForces3.begin(), gravityForces3.end(), gravity);

	//** gravity is added to total acceleration of rigid body (so far it only contained FSI forces). "totalAccRigid3" gets modified.
//	GpuTimer myT2;
//	myT2.Start();
	thrust::transform(totalAccRigid3.begin(), totalAccRigid3.end(), gravityForces3.begin(), totalAccRigid3.begin(), thrust::plus<real3>());
//	myT2.Stop();
//	real_ t2 = (real_)myT2.Elapsed();
//	printf("(2) ** transform timer %f, array size %d\n", t2, totalAccRigid3.size());



	gravityForces3.clear();

	//####### Torque
	uint nBlocks_numRigid_SphMarkers;
	uint nThreads_SphMarkers;
	computeGridSize(numObjects.numRigid_SphMarkers, 256, nBlocks_numRigid_SphMarkers, nThreads_SphMarkers);
	thrust::device_vector<real3> torqueMarkersD(numObjects.numRigid_SphMarkers);

	//** the current position of the rigid, 'posRigidD', is used to calculate the moment of BCE acceleration at the rigid
	//*** body center (i.e. torque/mass). "torqueMarkersD" gets built.
	CalcTorqueOf_SPH_Marker_Acceleration<<<nBlocks_numRigid_SphMarkers, nThreads_SphMarkers>>>(
			R3CAST(torqueMarkersD), R4CAST(derivVelRhoD), R3CAST(posRadD), I1CAST(rigidIdentifierD), R3CAST(posRigidD));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: CalcTorqueOf_SPH_Marker_Acceleration");

//	GpuTimer myT3;
//	myT3.Start();
	(void) thrust::reduce_by_key(rigidIdentifierD.begin(), rigidIdentifierD.end(), torqueMarkersD.begin(), dummyIdentify.begin(),
			totalTorqueOfAcc3.begin(), binary_pred, thrust::plus<real3>());
//	myT3.Stop();
//	real_ t3 = (real_)myT3.Elapsed();
//	printf("(3) ** reduce_by_key timer %f, array size %d\n", t3, rigidIdentifierD.size());


	torqueMarkersD.clear();
	dummyIdentify.clear();

	thrust::device_vector<real3> LF_totalTorqueOfAcc3(numObjects.numRigidBodies);

	//** current rotation of the rigid body, 'AD', is used to convert torque to LRF.
	MapTorqueToLRFKernel<<<nBlock_UpdateRigid, nThreads_rigidParticles>>>(
			R3CAST(AD1), R3CAST(AD2), R3CAST(AD3), R3CAST(totalTorqueOfAcc3), R3CAST(LF_totalTorqueOfAcc3));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: MapTorqueToLRFKernel");
	totalTorqueOfAcc3.clear();

	//################################################### update rigid body motion
	//####### Translation

	//** posRigidD2, posRigidCumulativeD2, velMassRigidD2, are updated based on their current value and dT.
	if (fracSimulation <-.01) {
		UpdateKernelRigidTranstalationBeta<<<nBlock_UpdateRigid, nThreads_rigidParticles>>>(
				R3CAST(posRigidD2), R3CAST(posRigidCumulativeD2), R4CAST(velMassRigidD2), R4CAST(velMassRigidD), R3CAST(totalAccRigid3));
	} else {
		UpdateKernelRigidTranstalation<<<nBlock_UpdateRigid, nThreads_rigidParticles>>>(
				R3CAST(posRigidD2), R3CAST(posRigidCumulativeD2), R4CAST(velMassRigidD2), R4CAST(velMassRigidD), R3CAST(totalAccRigid3));
	}
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: UpdateKernelRigid");
	totalAccRigid3.clear();

	//####### Rotation
	//** "qD2" is updated based on its current value and dTD.
	UpdateRigidBodyQuaternion_kernel<<<nBlock_UpdateRigid, nThreads_rigidParticles>>>(R4CAST(qD2), R4CAST(qD), R3CAST(omegaLRF_D));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: UpdateRotation");

	//** "qD2" is tranlated into rotation matrix, "AD_2"
	RotationMatirixFromQuaternion_kernel<<<nBlock_UpdateRigid, nThreads_rigidParticles>>>(R3CAST(AD1_2), R3CAST(AD2_2), R3CAST(AD3_2), R4CAST(qD2));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: UpdateRotation");

	//** "omegaLRF_D2" is updated based on its current value and dT
	UpdateRigidBodyAngularVelocity_kernel<<<nBlock_UpdateRigid, nThreads_rigidParticles>>>(
			R3CAST(omegaLRF_D2), R3CAST(LF_totalTorqueOfAcc3), R3CAST(omegaLRF_D), R3CAST(jD1), R3CAST(jD2), R3CAST(jInvD1), R3CAST(jInvD2));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: UpdateKernelRigid");

	LF_totalTorqueOfAcc3.clear();
	//################################################### update BCE markers position
	//** "posRadD2"/"velMasD2" associated to BCE markers are updated based on new rigid body (position, orientation)/(velocity, angular velocity)
	UpdateRigidMarkersPosition<<<nBlocks_numRigid_SphMarkers, nThreads_SphMarkers>>>(
			R3CAST(posRadD2), R4CAST(velMasD2),
			R3CAST(rigidSPH_MeshPos_LRF_D),
			I1CAST(rigidIdentifierD), R3CAST(posRigidD2), R4CAST(velMassRigidD2), R3CAST(omegaLRF_D2), R3CAST(AD1_2), R3CAST(AD2_2), R3CAST(AD3_2));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: UpdateKernelRigid");
}

//--------------------------------------------------------------------------------------------------------------------------------
void UpdateFlexibleBody(
		thrust::device_vector<real3> & posRadD2,
		thrust::device_vector<real4> & velMasD2,

		thrust::device_vector<real3> & ANCF_NodesD2,
		thrust::device_vector<real3> & ANCF_SlopesD2,
		thrust::device_vector<real3> & ANCF_NodesVelD2,
		thrust::device_vector<real3> & ANCF_SlopesVelD2,

		thrust::device_vector<real4> & derivVelRhoD,

		const thrust::device_vector<real3> & ANCF_NodesD,
		const thrust::device_vector<real3> & ANCF_SlopesD,
		const thrust::device_vector<real3> & ANCF_NodesVelD,
		const thrust::device_vector<real3> & ANCF_SlopesVelD,

		thrust::device_vector<int2> & ANCF_ReferenceArrayNodesOnBeamsD,
		thrust::device_vector<int> & ANCF_NumMarkers_Per_BeamD,
		thrust::device_vector<int> & ANCF_NumMarkers_Per_Beam_CumulD,
//		thrust::device_vector<int> & ANCF_NumNodesMultMarkers_Per_BeamD,
		thrust::device_vector<int> & ANCF_NumNodesMultMarkers_Per_Beam_CumulD,

		const thrust::device_vector<int> & flexIdentifierD,
		const thrust::device_vector<int2> & flexMapEachMarkerOnAllBeamNodesD,
		const thrust::device_vector<real3> & flexSPH_MeshPos_LRF_D,
		const thrust::device_vector<real3> & flexSPH_MeshSlope_Initial_D,
		const thrust::device_vector<real_> & flexParametricDistD,
		const thrust::device_vector<real_> & ANCF_Beam_LengthD,
		const thrust::device_vector<bool> & ANCF_IsCantileverD,
		const thrust::host_vector<int3> & referenceArray,

		const ANCF_Params & flexParams,
		NumberOfObjects numObjects,
		float fracSimulation,
		real_ dT) {
	if (numObjects.numFlexBodies == 0) {
		return;
	}
	hipMemcpyToSymbolAsync(HIP_SYMBOL(dTD), &dT, sizeof(dT));

	//################################################### make force arrays
	int2 totalNumberOfFlexNodes2 = ANCF_ReferenceArrayNodesOnBeamsD[ANCF_ReferenceArrayNodesOnBeamsD.size() - 1];
	int totalNumberOfFlexNodes = totalNumberOfFlexNodes2.y;
	int totalNumberOfFlexBCEMultNodes = flexMapEachMarkerOnAllBeamNodesD.size();

	thrust::device_vector<real3> flex_FSI_NodesForces1(totalNumberOfFlexNodes);
	thrust::device_vector<real3> flex_FSI_NodesForces2(totalNumberOfFlexNodes);
	thrust::fill(flex_FSI_NodesForces1.begin(), flex_FSI_NodesForces1.end(),R3(0));
	thrust::fill(flex_FSI_NodesForces2.begin(), flex_FSI_NodesForces2.end(),R3(0));

	//**
	thrust::device_vector<real3> flexNodesForcesAllMarkers1(totalNumberOfFlexBCEMultNodes);
	thrust::device_vector<real3> flexNodesForcesAllMarkers2(totalNumberOfFlexBCEMultNodes);

	uint nBlocks_numFlex_SphMarkers;
	uint nThreads_SphMarkers;
	computeGridSize(numObjects.numFlex_SphMarkers, 256, nBlocks_numFlex_SphMarkers, nThreads_SphMarkers);
	//** maps each BCE marker FORCE onto all beam nodes. Does not sum them up, writes them next to each other. Assumes the BCE forces
	//*** at the beam center line. "flexNodesForcesAllMarkers1" and "flexNodesForcesAllMarkers1" are built.
	MapForcesOnNodes<<<nBlocks_numFlex_SphMarkers, nThreads_SphMarkers>>>(
			R3CAST(flexNodesForcesAllMarkers1),
			R3CAST(flexNodesForcesAllMarkers2),
			I1CAST(flexIdentifierD),
			I2CAST(ANCF_ReferenceArrayNodesOnBeamsD),
			I1CAST(ANCF_NumMarkers_Per_BeamD),
			I1CAST(ANCF_NumMarkers_Per_Beam_CumulD),
			I1CAST(ANCF_NumNodesMultMarkers_Per_Beam_CumulD),
			R1CAST(flexParametricDistD),
			R1CAST(ANCF_Beam_LengthD),
			R4CAST(derivVelRhoD));

	if (flexMapEachMarkerOnAllBeamNodesD.size() != flexNodesForcesAllMarkers1.size()) {
		printf("we have size inconsistency between flex nodesForces and nodesPair identifier");
	}
	thrust::device_vector<int2> dummyNodesFlexIdentify(totalNumberOfFlexNodes);
	thrust::equal_to<int2> binary_pred_int2; //if binary_pred int2 does not work, you have to either add operator == to custom_cutil_math, or you have to map nodes identifiers from int2 to int

	//** Superposes the forces of all BCE markers on nodes onto single force on each node. First 3 nodal coordinates.
	(void) thrust::reduce_by_key(flexMapEachMarkerOnAllBeamNodesD.begin(), flexMapEachMarkerOnAllBeamNodesD.end(), flexNodesForcesAllMarkers1.begin(),
			dummyNodesFlexIdentify.begin(), flex_FSI_NodesForces1.begin(),
			binary_pred_int2, thrust::plus<real3>());
	//** Superposes the forces of all BCE markers on nodes onto single force on each node. Second 3 nodal coordinates.
	(void) thrust::reduce_by_key(flexMapEachMarkerOnAllBeamNodesD.begin(), flexMapEachMarkerOnAllBeamNodesD.end(), flexNodesForcesAllMarkers2.begin(),
			dummyNodesFlexIdentify.begin(), flex_FSI_NodesForces2.begin(),
			binary_pred_int2, thrust::plus<real3>());
	dummyNodesFlexIdentify.clear();
	flexNodesForcesAllMarkers1.clear();
	flexNodesForcesAllMarkers2.clear();

	//################################################### Update nodal coordinates (integrate in time)
	//** uses the force values on nodal coordinates to update nodal position and slope and velocities ('2' at the end).
	//*** it uses the current un-updated nodal coordinates (without '2' at the end) to calculate elastic forces.
	//################################### multi-rate
	thrust::device_vector<real3> ANCF_NodesD3(ANCF_NodesD2.size());
	thrust::device_vector<real3> ANCF_SlopesD3(ANCF_SlopesD2.size());
	thrust::device_vector<real3> ANCF_NodesVelD3(ANCF_NodesVelD2.size());
	thrust::device_vector<real3> ANCF_SlopesVelD3(ANCF_SlopesVelD2.size());
	int n = 50;
	for (int i = 0; i < n; i++) {
		thrust::copy(ANCF_NodesD2.begin(), ANCF_NodesD2.end(), ANCF_NodesD3.begin());
		thrust::copy(ANCF_SlopesD2.begin(), ANCF_SlopesD2.end(), ANCF_SlopesD3.begin());
		thrust::copy(ANCF_NodesVelD2.begin(), ANCF_NodesVelD2.end(), ANCF_NodesVelD3.begin());
		thrust::copy(ANCF_SlopesVelD2.begin(), ANCF_SlopesVelD2.end(), ANCF_SlopesVelD3.begin());

		Update_ANCF_Beam(
				ANCF_NodesD3, ANCF_SlopesD3, ANCF_NodesVelD3, ANCF_SlopesVelD3,
				ANCF_NodesD2, ANCF_SlopesD2, ANCF_NodesVelD2, ANCF_SlopesVelD2,
				flex_FSI_NodesForces1, flex_FSI_NodesForces2,
				ANCF_ReferenceArrayNodesOnBeamsD, ANCF_Beam_LengthD, ANCF_IsCantileverD,
				numObjects.numFlexBodies, flexParams, dT/(2*n)
				);

		Update_ANCF_Beam(
				ANCF_NodesD2, ANCF_SlopesD2, ANCF_NodesVelD2, ANCF_SlopesVelD2,
				ANCF_NodesD3, ANCF_SlopesD3, ANCF_NodesVelD3, ANCF_SlopesVelD3,
				flex_FSI_NodesForces1, flex_FSI_NodesForces2,
				ANCF_ReferenceArrayNodesOnBeamsD, ANCF_Beam_LengthD, ANCF_IsCantileverD,
				numObjects.numFlexBodies, flexParams, dT/(n)
				);
	}
	ANCF_NodesD3.clear();
	ANCF_SlopesD3.clear();
	ANCF_NodesVelD3.clear();
	ANCF_SlopesVelD3.clear();
	//################################### not multi-rate
//		Update_ANCF_Beam(
//				ANCF_NodesD2, ANCF_SlopesD2, ANCF_NodesVelD2, ANCF_SlopesVelD2,
//				ANCF_NodesD, ANCF_SlopesD, ANCF_NodesVelD, ANCF_SlopesVelD,
//				flex_FSI_NodesForces1, flex_FSI_NodesForces2,
//				ANCF_ReferenceArrayNodesOnBeamsD, ANCF_Beam_LengthD, ANCF_IsCantileverD,
//				numFlexBodies, flexParams, dT
//				);
	//################################################### update BCE markers position
	//** new nodal velocity and positions are used to update BCE markers position and velocities. "posRadD", "velMasD" are updated (only the flex portion)
	computeGridSize(numObjects.numFlex_SphMarkers, 256, nBlocks_numFlex_SphMarkers, nThreads_SphMarkers);
	UpdateFlexMarkersPosition<<<nBlocks_numFlex_SphMarkers, nThreads_SphMarkers>>>(
			R3CAST(posRadD2), R4CAST(velMasD2),
			I1CAST(flexIdentifierD),
			R3CAST(flexSPH_MeshPos_LRF_D),
			R3CAST(flexSPH_MeshSlope_Initial_D),
			R1CAST(flexParametricDistD),
			R1CAST(ANCF_Beam_LengthD),
			I2CAST(ANCF_ReferenceArrayNodesOnBeamsD),
			R3CAST(ANCF_NodesD2),
			R3CAST(ANCF_SlopesD2),
			R3CAST(ANCF_NodesVelD2),
			R3CAST(ANCF_SlopesVelD2)
			);

	hipDeviceSynchronize();

	CUT_CHECK_ERROR("Kernel execution failed: UpdateKernelRigid");


	//------------------------ delete stuff
	flex_FSI_NodesForces1.clear();
	flex_FSI_NodesForces2.clear();
}
//##############################################################################################################################################
// the main function, which updates the particles and implements BC
void cudaCollisions(
		thrust::host_vector<real3> & mPosRad,
		thrust::host_vector<real4> & mVelMas,
		thrust::host_vector<real4> & mRhoPresMu,
		const thrust::host_vector<uint> & bodyIndex,
		const thrust::host_vector<int3> & referenceArray,

		thrust::host_vector<real3> & posRigidH,
		thrust::host_vector<real4> & mQuatRot,
		thrust::host_vector<real4> & velMassRigidH,
		thrust::host_vector<real3> omegaLRF_H,
		thrust::host_vector<real3> jH1,
		thrust::host_vector<real3> jH2,
		thrust::host_vector<real3> jInvH1,
		thrust::host_vector<real3> jInvH2,

		const thrust::host_vector<real3> & ANCF_Nodes,
		const thrust::host_vector<real3> & ANCF_Slopes,
		const thrust::host_vector<real3> & ANCF_NodesVel,
		const thrust::host_vector<real3> & ANCF_SlopesVel,
		const thrust::host_vector<real_> & ANCF_Beam_Length,
		const thrust::host_vector<bool> & ANCF_IsCantilever,
		const thrust::host_vector<int2> & ANCF_ReferenceArrayNodesOnBeams,
		const thrust::host_vector<real_> & flexParametricDist,

		real_ channelRadius,
		real2 channelCenterYZ,
		SimParams paramsH,
		const ANCF_Params & flexParams,
		NumberOfObjects & numObjects) {
	//****************************** bin size adjustement and contact detection stuff *****************************
	int3 SIDE = I3(int((paramsH.cMax.x - paramsH.cMin.x) / paramsH.binSize0 + .1), int((paramsH.cMax.y - paramsH.cMin.y) / paramsH.binSize0 + .1),
			int((paramsH.cMax.z - paramsH.cMin.z) / paramsH.binSize0 + .1));
	real_ mBinSize = paramsH.binSize0; //Best solution in that case may be to change cMax or cMin such that periodic sides be a multiple of binSize

	printf("SIDE: %d, %d, %d\n", SIDE.x, SIDE.y, SIDE.z);
	//**********************************************************************************************************
	paramsH.gridSize = SIDE;
	//paramsH.numCells = SIDE.x * SIDE.y * SIDE.z;
	paramsH.worldOrigin = paramsH.cMin;
	paramsH.cellSize = R3(mBinSize, mBinSize, mBinSize);
	paramsH.boxDims = paramsH.cMax - paramsH.cMin;
	printf("boxDims: %f, %f, %f\n", paramsH.boxDims.x, paramsH.boxDims.y, paramsH.boxDims.z);

	setParameters(&paramsH, &numObjects); 														// sets paramsD in SDKCollisionSystem
	cutilSafeCall( hipMemcpyToSymbolAsync(HIP_SYMBOL(paramsD), &paramsH, sizeof(SimParams))); 	//sets paramsD for this file
	cutilSafeCall( hipMemcpyToSymbolAsync(HIP_SYMBOL(numObjectsD), &numObjects, sizeof(NumberOfObjects)));
	//*************************************************************************************************************
	//--------- initialization ---------------
	//hipError_t dumDevErr = hipSetDevice(2);
	GpuTimer myTotalTime;
	myTotalTime.Start();
	//printf("cMin.x, y, z, CMAx.x, y, z, binSize %f %f %f , %f %f %f, %f\n", paramsH.cMin.x, paramsH.cMin.y, paramsH.cMin.z, paramsH.cMax.x, paramsH.cMax.y, paramsH.cMax.z, paramsH.binSize0);
	hipDeviceSetCacheConfig(hipFuncCachePreferL1);

	thrust::device_vector<real3> posRadD=mPosRad;
	//thrust::copy(mPosRad.begin(), mPosRad.end(), posRadD.begin());
	thrust::device_vector<real4> velMasD=mVelMas;
	//thrust::copy(mVelMas.begin(), mVelMas.end(), velMasD.begin());
	thrust::device_vector<real4> rhoPresMuD=mRhoPresMu;
	//thrust::copy(mRhoPresMu.begin(), mRhoPresMu.end(), rhoPresMuD.begin());

	thrust::device_vector<real3> posRigidD=posRigidH;
	//thrust::copy(posRigidH.begin(), posRigidH.end(), posRigidD.begin());
	thrust::device_vector<real3> posRigidCumulativeD=posRigidH;
	//thrust::copy(posRigidH.begin(), posRigidH.end(), posRigidCumulativeD.begin());
	thrust::device_vector<real4> velMassRigidD=velMassRigidH;
	//thrust::copy(velMassRigidH.begin(), velMassRigidH.end(), velMassRigidD.begin());
	thrust::device_vector<real3> omegaLRF_D=omegaLRF_H;
	//thrust::copy(omegaLRF_H.begin(), omegaLRF_H.end(), omegaLRF_D.begin());

	thrust::device_vector<real3> jD1=jH1;
	thrust::device_vector<real3> jD2=jH2;
	thrust::device_vector<real3> jInvD1=jInvH1;
	thrust::device_vector<real3> jInvD2=jInvH2;
	//thrust::copy(jH1.begin(), jH1.end(), jD1.begin());
	//thrust::copy(jH2.begin(), jH2.end(), jD2.begin());
	//thrust::copy(jInvH1.begin(), jInvH1.end(), jInvD1.begin());
	//thrust::copy(jInvH2.begin(), jInvH2.end(), jInvD2.begin());

	thrust::device_vector<uint> bodyIndexD=bodyIndex;
	//thrust::copy(bodyIndex.begin(), bodyIndex.end(), bodyIndexD.begin());
	thrust::device_vector<real4> derivVelRhoD(numObjects.numAllMarkers);
		//******************** rigid body some initialization
	real_ solid_SPH_mass;
	printf("ff1, reference array [0]: %d %d, [1]: %d %d, [2]: %d %d, size %d\n", (referenceArray[0]).x, (referenceArray[0]).y, (referenceArray[1]).x, (referenceArray[1]).y, (referenceArray[2]).x, (referenceArray[2]).y, referenceArray.size());
	thrust::device_vector<int> rigidIdentifierD(numObjects.numRigid_SphMarkers);
	if (referenceArray.size() > 2) {
		real4 typical_BCE_MarkerVelMass = mVelMas[referenceArray[2].x];
		solid_SPH_mass = typical_BCE_MarkerVelMass.w;
	} else {
		real4 dummyFluid = mVelMas[referenceArray[0].x];
		solid_SPH_mass = dummyFluid.w;
	}
	cutilSafeCall( hipMemcpyToSymbolAsync(HIP_SYMBOL(solid_SPH_massD), &solid_SPH_mass, sizeof(solid_SPH_mass)));

	MakeRigidIdentifier(rigidIdentifierD, numObjects.numRigidBodies, numObjects.startRigidMarkers, referenceArray);

	//******************************************************************************
	thrust::device_vector<real4> qD1 = mQuatRot;
	thrust::device_vector<real3> AD1(numObjects.numRigidBodies);
	thrust::device_vector<real3> AD2(numObjects.numRigidBodies);
	thrust::device_vector<real3> AD3(numObjects.numRigidBodies);
	uint nBlock_UpdateRigid;
	uint nThreads_rigidParticles;
	computeGridSize(numObjects.numRigidBodies, 128, nBlock_UpdateRigid, nThreads_rigidParticles);
	RotationMatirixFromQuaternion_kernel<<<nBlock_UpdateRigid, nThreads_rigidParticles>>>(R3CAST(AD1), R3CAST(AD2), R3CAST(AD3), R4CAST(qD1));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: UpdateRotation");

	//******************************************************************************
	thrust::device_vector<real3> rigidSPH_MeshPos_LRF_D(numObjects.numRigid_SphMarkers);
	uint nBlocks_numRigid_SphMarkers;
	uint nThreads_SphMarkers;
	computeGridSize(numObjects.numRigid_SphMarkers, 256, nBlocks_numRigid_SphMarkers, nThreads_SphMarkers);

	Populate_RigidSPH_MeshPos_LRF_kernel<<<nBlocks_numRigid_SphMarkers, nThreads_SphMarkers>>>(R3CAST(rigidSPH_MeshPos_LRF_D), R3CAST(posRadD), I1CAST(rigidIdentifierD), R3CAST(posRigidD),
			R3CAST(AD1), R3CAST(AD2), R3CAST(AD3));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: CalcTorqueOf_SPH_Marker_Acceleration");

	//******************************************************************************
	//******************** flex body some initialization
//	int totalNumberOfFlexNodes = ANCF_ReferenceArrayNodesOnBeamsD[ANCF_ReferenceArrayNodesOnBeamsD.size() - 1].y;

	//******************************************************************************
	thrust::device_vector<real_> flexParametricDistD = flexParametricDist;
	thrust::device_vector<int> flexIdentifierD(numObjects.numFlex_SphMarkers);

	MakeFlexIdentifier(flexIdentifierD, numObjects.numFlexBodies, numObjects.numFlBcRigid, numObjects.startFlexMarkers, referenceArray);

	thrust::device_vector<real3> ANCF_NodesD = ANCF_Nodes;
	thrust::device_vector<real3> ANCF_SlopesD = ANCF_Slopes;
	thrust::device_vector<real3> ANCF_NodesVelD = ANCF_NodesVel;
	thrust::device_vector<real3> ANCF_SlopesVelD = ANCF_SlopesVel;
	thrust::device_vector<real_> ANCF_Beam_LengthD = ANCF_Beam_Length;
	thrust::device_vector<bool> ANCF_IsCantileverD = ANCF_IsCantilever;
	thrust::device_vector<int2> ANCF_ReferenceArrayNodesOnBeamsD = ANCF_ReferenceArrayNodesOnBeams;  //each element refer to a beam. and contains the start and end
																									// index of nodes in the array of beams nodes (like referenceArray)

	//*******************
	thrust::device_vector<int> ANCF_NumMarkers_Per_BeamD(numObjects.numFlexBodies);  //num BCE markers per beam
	thrust::device_vector<int> ANCF_NumMarkers_Per_Beam_CumulD(numObjects.numFlexBodies); // exclusive scan of ANCF_NumMarkers_Per_BeamD
	thrust::device_vector<int> ANCF_NumNodesMultMarkers_Per_BeamD(numObjects.numFlexBodies); //i_th component is equal to nN*nM (N and M denote nodes and markers per beam) of beam i
	thrust::device_vector<int> ANCF_NumNodesMultMarkers_Per_Beam_CumulD(numObjects.numFlexBodies); //exclusive scan of ANCF_NumNodesMultMarkers_Per_BeamD
	thrust::device_vector<int2> flexMapEachMarkerOnAllBeamNodesD(0); //assume beam i has nN nodes and nM markers. lets j denote the nodes. This array includes
																	// concequtive chunks of pairs I2(i,j). Each chunk has a length of nM. The total number of chunks
																	// per beam is nN. In summary, nN chuncks of I2(i, j) pairs (j changes from 0 to nN), Each chunk with
																	// with the length of nM

	printf("ff1, flexIdsize %d, num flex bodies %d\n", flexIdentifierD.size(), numObjects.numFlexBodies);
	thrust::device_vector<int> dummySum(flexIdentifierD.size());
	thrust::device_vector<int> dummyIdentifier(numObjects.numFlexBodies);
	thrust::fill(dummySum.begin(), dummySum.end(), 1);





	printf("\n\n\n");
	(void) thrust::reduce_by_key(flexIdentifierD.begin(), flexIdentifierD.end(), dummySum.begin(), dummyIdentifier.begin(), ANCF_NumMarkers_Per_BeamD.begin());
	thrust::exclusive_scan(ANCF_NumMarkers_Per_BeamD.begin(), ANCF_NumMarkers_Per_BeamD.end(), ANCF_NumMarkers_Per_Beam_CumulD.begin());
	dummySum.clear();
	dummyIdentifier.clear();
	Calc_NumNodesMultMarkers_Per_Beam(ANCF_NumNodesMultMarkers_Per_BeamD, ANCF_NumMarkers_Per_BeamD, ANCF_ReferenceArrayNodesOnBeams, numObjects.numFlexBodies);
	thrust::exclusive_scan(ANCF_NumNodesMultMarkers_Per_BeamD.begin(), ANCF_NumNodesMultMarkers_Per_BeamD.end(), ANCF_NumNodesMultMarkers_Per_Beam_CumulD.begin());
	int total_NumNodesMultMarkers_Per_Beam = (numObjects.numFlexBodies > 0) ? (ANCF_NumNodesMultMarkers_Per_Beam_CumulD[numObjects.numFlexBodies - 1] + ANCF_NumNodesMultMarkers_Per_BeamD[numObjects.numFlexBodies - 1]) : 0;
	flexMapEachMarkerOnAllBeamNodesD.resize(total_NumNodesMultMarkers_Per_Beam);


	printf("total_NumNodesMultMarkers_Per_Beam %d\n", total_NumNodesMultMarkers_Per_Beam);

	Calc_mapEachMarkerOnAllBeamNodes_IdentifierD(flexMapEachMarkerOnAllBeamNodesD, ANCF_NumNodesMultMarkers_Per_Beam_CumulD, ANCF_NumMarkers_Per_BeamD, ANCF_ReferenceArrayNodesOnBeams, numObjects.numFlexBodies);

	//*******************

	thrust::device_vector<real3> flexSPH_MeshPos_LRF_D(numObjects.numFlex_SphMarkers);
	thrust::device_vector<real3> flexSPH_MeshSlope_Initial_D(numObjects.numFlex_SphMarkers);  //slope of the beam at BCE marker (associated to BCE marker)
	uint nBlocks_numFlex_SphMarkers;
	computeGridSize(numObjects.numFlex_SphMarkers, 256, nBlocks_numFlex_SphMarkers, nThreads_SphMarkers);

	Populate_FlexSPH_MeshPos_LRF_kernel<<<nBlocks_numFlex_SphMarkers, nThreads_SphMarkers>>>(R3CAST(flexSPH_MeshPos_LRF_D), R3CAST(posRadD), I1CAST(flexIdentifierD), R1CAST(flexParametricDistD), R1CAST(ANCF_Beam_LengthD),
			I2CAST(ANCF_ReferenceArrayNodesOnBeamsD), R3CAST(ANCF_NodesD), R3CAST(ANCF_SlopesD));
	hipDeviceSynchronize();
		CUT_CHECK_ERROR("Kernel execution failed: Populate_FlexSPH_MeshPos_LRF_kernel");

	Populate_FlexSPH_MeshSlope_LRF_kernel<<<nBlocks_numFlex_SphMarkers, nThreads_SphMarkers>>>(R3CAST(flexSPH_MeshSlope_Initial_D), I1CAST(flexIdentifierD), R1CAST(flexParametricDistD), R1CAST(ANCF_Beam_LengthD),
				I2CAST(ANCF_ReferenceArrayNodesOnBeamsD), R3CAST(ANCF_NodesD), R3CAST(ANCF_SlopesD));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: Populate_FlexSPH_MeshSlope_LRF_kernel");

	//int i =  rigidIdentifierD[429];
	//printf("rigid body coord %d %f %f\n", i, posRigidH[i].x, posRigidH[i].z);
	//printf("length %f\n", length(R2(posRigidH[i].x - .003474, posRigidH[i].z - .000673)));

	int numberOfSections = 20; //number of sections for measuring the distribution
	thrust::device_vector<int>  distributionD(numberOfSections);

	FILE *outFileMultipleZones;

	int povRayCounter = 0;
	int stepEnd = int(paramsH.tFinal/paramsH.dT);//1.0e6;//2.4e6;//600000;//2.4e6 * (.02 * paramsH.sizeScale) / currentParamsH.dT ; //1.4e6 * (.02 * paramsH.sizeScale) / currentParamsH.dT ;//0.7e6 * (.02 * paramsH.sizeScale) / currentParamsH.dT ;//0.7e6;//2.5e6; //200000;//10000;//50000;//100000;
	printf("stepEnd %d\n", stepEnd);

	real_ delTOrig = paramsH.dT;
	real_ realTime = 0;

	real_ timePause = .001 * paramsH.tFinal; // keep it as small as possible. the time step will be 1/10 * dT
	real_ timePauseRigidFlex = .02 * paramsH.tFinal;
	SimParams paramsH_B = paramsH;
	paramsH_B.bodyForce4 = R4(0);
	paramsH_B.gravity = R3(0);
	paramsH_B.dT = .1 * paramsH.dT;

	printf("\ntimePause %f, numPause %d\n", timePause, int(timePause/paramsH_B.dT));
	printf("timePauseRigidFlex %f, numPauseRigidFlex %d\n\n", timePauseRigidFlex, int((timePauseRigidFlex-timePause)/paramsH.dT + timePause/paramsH_B.dT));

	SimParams currentParamsH = paramsH;

	real_ timeSlice = real_(paramsH.tFinal)/7;
	for (int tStep = 0; tStep < stepEnd + 1; tStep++) {
		//************************************************
		//edit  since yu deleted cyliderRotOmegaJD
		PrintToFile(posRadD, velMasD, rhoPresMuD,
				referenceArray, rigidIdentifierD,
				posRigidD, posRigidCumulativeD, velMassRigidD, qD1, AD1, AD2, AD3, omegaLRF_D,
				ANCF_NodesD, ANCF_SlopesD, ANCF_NodesVelD, ANCF_SlopesVelD, ANCF_ReferenceArrayNodesOnBeamsD,
				currentParamsH,
				realTime, tStep, channelRadius, channelCenterYZ, numObjects.numRigidBodies, numObjects.numFlexBodies);

////		PrintToFileDistribution(distributionD, channelRadius, numberOfSections, tStep);
		//************

		//edit  since yu deleted cyliderRotOmegaJD

		GpuTimer myGpuTimer;
		myGpuTimer.Start();

		struct timeval cpuT_start, cpuT_end;
		struct timezone cpuT_timezone;
		gettimeofday(&cpuT_start, &cpuT_timezone);

		//***********
		if (realTime <= timePause) 	{
			currentParamsH = paramsH_B;
		} else {
			currentParamsH = paramsH;
		}
		//***********
//		if (realTime <= timeSlice) {
//			currentParamsH = paramsH_B;
//		} else if (realTime <= 2 * timeSlice) {
//			currentParamsH.bodyForce4.x = paramsH.bodyForce4.x;
//			currentParamsH.bodyForce4.y = 0;
//		} else if (realTime <= 3 * timeSlice) {
//			currentParamsH.bodyForce4.x = 0;
//			currentParamsH.bodyForce4.y = .5 * paramsH.bodyForce4.x;
//		} else if (realTime <= 4 * timeSlice) {
//			currentParamsH.bodyForce4.x = -.7 * paramsH.bodyForce4.x;
//			currentParamsH.bodyForce4.y = -.5 * paramsH.bodyForce4.x;
//		} else if (realTime <= 5 * timeSlice) {
//			currentParamsH.bodyForce4.x = 1.0 * paramsH.bodyForce4.x;
//			currentParamsH.bodyForce4.y = 0;
//		} else if (realTime <= 5.5 * timeSlice) {
//			currentParamsH.bodyForce4.x = -.5 * paramsH.bodyForce4.x;
//			currentParamsH.bodyForce4.y = -.5 * paramsH.bodyForce4.x;
//		} else {
//			currentParamsH.bodyForce4.x = 1.0 * paramsH.bodyForce4.x;
//			currentParamsH.bodyForce4.y = 0;
//		}
		//***********
		setParameters(&currentParamsH, &numObjects); 														// sets paramsD in SDKCollisionSystem
		cutilSafeCall( hipMemcpyToSymbolAsync(HIP_SYMBOL(paramsD), &currentParamsH, sizeof(SimParams))); 	//sets paramsD for this file

		//computations
				//markers
		thrust::device_vector<real3> posRadD2 = posRadD;
		thrust::device_vector<real4> velMasD2 = velMasD;
		thrust::device_vector<real4> rhoPresMuD2 = rhoPresMuD;
		thrust::device_vector<real3> vel_XSPH_D(numObjects.numAllMarkers);
				//rigids
		thrust::device_vector<real3> posRigidD2 = posRigidD;
		thrust::device_vector<real3> posRigidCumulativeD2 = posRigidCumulativeD;
		thrust::device_vector<real4> velMassRigidD2 = velMassRigidD;
		thrust::device_vector<real3> omegaLRF_D2 = omegaLRF_D;
				thrust::device_vector<real3> AD1_2 = AD1;
		thrust::device_vector<real3> AD2_2 = AD2;
		thrust::device_vector<real3> AD3_2 = AD3;
		thrust::device_vector<real4> qD2 = qD1;
				//flex
		thrust::device_vector<real3> ANCF_NodesD2 = ANCF_NodesD;
		thrust::device_vector<real3> ANCF_SlopesD2 = ANCF_SlopesD;
		thrust::device_vector<real3> ANCF_NodesVelD2 = ANCF_NodesVelD;
		thrust::device_vector<real3> ANCF_SlopesVelD2 = ANCF_SlopesVelD;

		//******** RK2
		ForceSPH(posRadD, velMasD, vel_XSPH_D, rhoPresMuD, posRigidD, rigidIdentifierD, bodyIndexD, derivVelRhoD, referenceArray, numObjects.numAllMarkers, currentParamsH, 0.5 * currentParamsH.dT); //?$ right now, it does not consider paramsH.gravity or other stuff on rigid bodies. they should be applied at rigid body solver
		UpdateFluid(posRadD2, velMasD2, vel_XSPH_D, rhoPresMuD2, derivVelRhoD, referenceArray, 0.5 * currentParamsH.dT); //assumes ...D2 is a copy of ...D
		//UpdateBoundary(posRadD2, velMasD2, rhoPresMuD2, derivVelRhoD, referenceArray, 0.5 * currentParamsH.dT);		//assumes ...D2 is a copy of ...D

		if (realTime > timePauseRigidFlex) {
			UpdateRigidBody(
					posRadD2, velMasD2,
					posRigidD2, posRigidCumulativeD2, velMassRigidD2, qD2, AD1_2, AD2_2, AD3_2, omegaLRF_D2,
					posRadD,
					posRigidD, posRigidCumulativeD, velMassRigidD, qD1, AD1, AD2, AD3, omegaLRF_D,
					derivVelRhoD, rigidIdentifierD,
					rigidSPH_MeshPos_LRF_D, referenceArray, jD1, jD2, jInvD1, jInvD2, flexParams.gravity, numObjects, realTime / (paramsH.tFinal), 0.5 * currentParamsH.dT);

			UpdateFlexibleBody(posRadD2, velMasD2,
					ANCF_NodesD2, ANCF_SlopesD2, ANCF_NodesVelD2, ANCF_SlopesVelD2,
					derivVelRhoD,
									ANCF_NodesD, ANCF_SlopesD, ANCF_NodesVelD, ANCF_SlopesVelD,
									ANCF_ReferenceArrayNodesOnBeamsD,
									ANCF_NumMarkers_Per_BeamD,
									ANCF_NumMarkers_Per_Beam_CumulD,
									ANCF_NumNodesMultMarkers_Per_Beam_CumulD,

									flexIdentifierD,
									flexMapEachMarkerOnAllBeamNodesD,
									flexSPH_MeshPos_LRF_D,
									flexSPH_MeshSlope_Initial_D,
									flexParametricDistD,
									ANCF_Beam_LengthD,
									ANCF_IsCantileverD,
									referenceArray,

									flexParams,
									numObjects,
									realTime / (paramsH.tFinal),
									0.5 * currentParamsH.dT);
		}
		ApplyBoundary(posRadD2, rhoPresMuD2, posRigidD2, ANCF_NodesD2, ANCF_ReferenceArrayNodesOnBeamsD, numObjects);
//		//*****
		ForceSPH(posRadD2, velMasD2, vel_XSPH_D, rhoPresMuD2, posRigidD2, rigidIdentifierD, bodyIndexD, derivVelRhoD, referenceArray, numObjects.numAllMarkers, currentParamsH, currentParamsH.dT);
		UpdateFluid(posRadD, velMasD, vel_XSPH_D, rhoPresMuD, derivVelRhoD, referenceArray, currentParamsH.dT);
		//UpdateBoundary(posRadD, velMasD, rhoPresMuD, derivVelRhoD, referenceArray, currentParamsH.dT);

		if (realTime > timePauseRigidFlex) {
			UpdateRigidBody(
					posRadD, velMasD,
					posRigidD, posRigidCumulativeD, velMassRigidD, qD1, AD1, AD2, AD3, omegaLRF_D,
					posRadD2,
					posRigidD2, posRigidCumulativeD2, velMassRigidD2, qD2, AD1_2, AD2_2, AD3_2, omegaLRF_D2,
					derivVelRhoD, rigidIdentifierD,
					rigidSPH_MeshPos_LRF_D, referenceArray, jD1, jD2, jInvD1, jInvD2, flexParams.gravity, numObjects, realTime / (paramsH.tFinal), currentParamsH.dT);

			UpdateFlexibleBody(posRadD, velMasD,
					ANCF_NodesD, ANCF_SlopesD, ANCF_NodesVelD, ANCF_SlopesVelD,
					derivVelRhoD,
							ANCF_NodesD2, ANCF_SlopesD2, ANCF_NodesVelD2, ANCF_SlopesVelD2,
							ANCF_ReferenceArrayNodesOnBeamsD,
							ANCF_NumMarkers_Per_BeamD,
							ANCF_NumMarkers_Per_Beam_CumulD,
							ANCF_NumNodesMultMarkers_Per_Beam_CumulD,

							flexIdentifierD,
							flexMapEachMarkerOnAllBeamNodesD,
							flexSPH_MeshPos_LRF_D,
							flexSPH_MeshSlope_Initial_D,
							flexParametricDistD,
							ANCF_Beam_LengthD,
							ANCF_IsCantileverD,
							referenceArray,

							flexParams,
							numObjects,
							realTime / (paramsH.tFinal),
							currentParamsH.dT);
		}
		ApplyBoundary(posRadD, rhoPresMuD, posRigidD, ANCF_NodesD, ANCF_ReferenceArrayNodesOnBeamsD, numObjects);
		//************




					//			/* post_process for Segre-Silberberg */ goes before ApplyBoundary
					//			if(tStep >= 0) {
					//				real2 channelCenter = .5 * R2(currentParamsH.cMax.y + currentParamsH.cMin.y, currentParamsH.cMax.z + currentParamsH.cMin.z);
					//				FindPassesFromTheEnd(posRigidD, distributionD, numRigidBodies, channelCenter, channelRadius, numberOfSections);
					//			}


		posRadD2.clear();
		velMasD2.clear();
		rhoPresMuD2.clear();
		vel_XSPH_D.clear();

		posRigidD2.clear();
		posRigidCumulativeD2.clear();
		velMassRigidD2.clear();
		qD2.clear();
		AD1_2.clear();
		AD2_2.clear();
		AD3_2.clear();
		omegaLRF_D2.clear();

		ANCF_NodesD2.clear();
		ANCF_SlopesD2.clear();
		ANCF_NodesVelD2.clear();
		ANCF_SlopesVelD2.clear();

		//density re-initialization
		if (tStep % 10 == 0) {
			DensityReinitialization(posRadD, velMasD, rhoPresMuD, numObjects.numAllMarkers, SIDE); //does not work for analytical boundaries (non-meshed) and free surfaces
		}

		myGpuTimer.Stop();
		real_ time2 = (real_)myGpuTimer.Elapsed();

		//hipDeviceSynchronize();
		gettimeofday(&cpuT_end, &cpuT_timezone);
		double t1 = double(cpuT_start.tv_sec)+double(cpuT_start.tv_usec)/(1000*1000);
		double t2 = double(cpuT_end.tv_sec)+double(cpuT_end.tv_usec)/(1000*1000);


		if (tStep % 50 == 0) {
			printf("step: %d, realTime: %f, step Time (CUDA): %f, step Time (CPU): %f\n ", tStep, realTime, time2, 1000 * (t2 - t1));
			//printf("a \n");
		}
		fflush(stdout);

		realTime += currentParamsH.dT;

		//_CrtDumpMemoryLeaks(); //for memory leak detection (msdn suggestion for VS) apparently does not work in conjunction with cuda

	}

	//you may copy back to host
	posRadD.clear();
	velMasD.clear();
	rhoPresMuD.clear();
	posRigidD.clear();

	ANCF_NodesD.clear();
	ANCF_SlopesD.clear();
	ANCF_NodesVelD.clear();
	ANCF_SlopesVelD.clear();
	ANCF_Beam_LengthD.clear();
	ANCF_IsCantileverD.clear();
	ANCF_ReferenceArrayNodesOnBeamsD.clear();

	ANCF_NumMarkers_Per_BeamD.clear();
	ANCF_NumMarkers_Per_Beam_CumulD.clear();
	ANCF_NumNodesMultMarkers_Per_BeamD.clear();
	ANCF_NumNodesMultMarkers_Per_Beam_CumulD.clear();
	flexMapEachMarkerOnAllBeamNodesD.clear();


	posRigidCumulativeD.clear();
	velMassRigidD.clear();
	omegaLRF_D.clear();
	bodyIndexD.clear();
	derivVelRhoD.clear();
	rigidIdentifierD.clear();
	rigidSPH_MeshPos_LRF_D.clear();
	flexParametricDistD.clear();
	flexIdentifierD.clear();
	flexSPH_MeshPos_LRF_D.clear();
	flexSPH_MeshSlope_Initial_D.clear();
	qD1.clear();
	AD1.clear();
	AD2.clear();
	AD3.clear();
	distributionD.clear();

	jD1.clear();
	jD2.clear();
	jInvD1.clear();
	jInvD2.clear();

	myTotalTime.Stop();
	real_ time = (real_)myTotalTime.Elapsed();
	printf("total Time: %f\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n ", time);
}
