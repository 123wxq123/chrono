#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2014 projectchrono.org
// All right reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Author: Arman Pazouki
// =============================================================================
//
// Base class for managing data in chrono_fsi, aka fluid system.//
// =============================================================================

#include <thrust/sort.h>
#include "chrono_fsi/ChFsiDataManager.cuh"
#include "chrono_fsi/ChDeviceUtils.cuh" 

namespace chrono {
namespace fsi {

ChFsiDataManager::ChFsiDataManager() {

}

ChFsiDataManager::~ChFsiDataManager() {}

void ChFsiDataManager::AddSphMarker(Real3 pos, Real3 vel, Real4 rhoPresMu) {
	sphMarkersH.posRadH.push_back(pos);
	sphMarkersH.velMasH.push_back(vel);
	sphMarkersH.rhoPresMuH.push_back(rhoPresMu);
}

void ChFsiDataManager::ArrangeDataManager() {
	thrust::host_vector<Real4> dummyRhoPresMuH = sphMarkersH.rhoPresMuH;

	// arrange data based on type: fluid, boundary, bce1, bce2, ....
	thrust::sort_by_key(dummyRhoPresMuH.begin(), dummyRhoPresMuH.end(), sphMarkersH.iterator(), sphTypeComp());
	dummyRhoPresMuH.clear();

	ConstructReferenceArray();
}

void ChFsiDataManager::ConstructReferenceArray() {
	ArrangeDataManager();

	// determine the number of each component
	int numMarkers = sphMarkersH.rhoPresMuH.size();
	thrust::host_vector<int> numComponentMarkers(numMarkers);
	thrust::fill(numComponentMarkers.begin(), numComponentMarkers.end(), 1);
	thrust::host_vector<Real4> dummyRhoPresMuH = sphMarkersH.rhoPresMuH;
	thrust::copy(sphMarkersH.rhoPresMuH.begin(), sphMarkersH.rhoPresMuH.end(), dummyRhoPresMuH.begin());
	int numberOfComponents = (thrust::reduce_by_key(dummyRhoPresMuH.begin(), dummyRhoPresMuH.end(), numComponentMarkers.begin(), 
			dummyRhoPresMuH.begin(), numComponentMarkers.begin(), sphTypeComp())).second 
			- numComponentMarkers.begin();

	// if (numberOfComponents == 0) {
	// 	std::cout << "Error! no marker found! Thrown from ConstructReferenceArray\n";
	// 	return;
	// }
	fsiGeneralData.referenceArray.resize(numberOfComponents);
	dummyRhoPresMuH.resize(numberOfComponents);
	numComponentMarkers.resize(numberOfComponents);
	int savedNumber = 0;
	for (int i = 0; i < numberOfComponents; numberOfComponents++) {
		int compType = std::floor(dummyRhoPresMuH[i].w + .1);
		int phaseType = -1;
		if (compType < 0) {
			phaseType = -1;
		} else if (compType == 0) {
			phaseType = 0;
		} else { // Arman : TODO for flex
			phaseType = 1;
		}
		fsiGeneralData.referenceArray[i] = mI4(savedNumber, savedNumber + numComponentMarkers[i], phaseType, compType);
	}
	dummyRhoPresMuH.clear();	
	numComponentMarkers.clear();
}

void ChFsiDataManager::FinalizeDataManager() {
	ConstructReferenceArray();
	int numMarkers = sphMarkersH.rhoPresMuH.size(); // Arman : do it with numObjects and such
	sphMarkersD1.resize(numMarkers);
	sphMarkersD2.resize(numMarkers);
	fsiGeneralData.derivVelRhoD.resize(numMarkers);
	fsiGeneralData.vel_XSPH_D.resize(numMarkers);

	thrust::copy(sphMarkersH.posRadH.begin(), sphMarkersH.posRadH.end(), sphMarkersD1.posRadD.begin());
	thrust::copy(sphMarkersH.velMasH.begin(), sphMarkersH.velMasH.end(), sphMarkersD1.velMasD.begin());
	thrust::copy(sphMarkersH.rhoPresMuH.begin(), sphMarkersH.rhoPresMuH.end(), sphMarkersD1.rhoPresMuD.begin());

	thrust::copy(sphMarkersD1.posRadD.begin(), sphMarkersD1.posRadD.end(), sphMarkersD2.posRadD.begin());
	thrust::copy(sphMarkersD1.velMasD.begin(), sphMarkersD1.velMasD.end(), sphMarkersD2.velMasD.begin());
	thrust::copy(sphMarkersD1.rhoPresMuD.begin(), sphMarkersD1.rhoPresMuD.end(), sphMarkersD2.rhoPresMuD.begin());

	// copy rigids
	int numFsiBodies = fsiBodiesH.posRigid_fsiBodies_H.size(); // Arman : do it with numObjects and such or do it externally
	fsiBodiesD1.resize(numFsiBodies);
	fsiBodiesD2.resize(numFsiBodies);
	fsiGeneralData.rigid_FSI_ForcesD.resize(numFsiBodies);
	fsiGeneralData.rigid_FSI_TorquesD.resize(numFsiBodies);

	// Arman: do it with zip iterator
	thrust::copy(fsiBodiesH.posRigid_fsiBodies_H.begin(), fsiBodiesH.posRigid_fsiBodies_H.end(), fsiBodiesD1.posRigid_fsiBodies_D.begin());
	thrust::copy(fsiBodiesH.velMassRigid_fsiBodies_H.begin(), fsiBodiesH.velMassRigid_fsiBodies_H.end(), fsiBodiesD1.velMassRigid_fsiBodies_D.begin());
	thrust::copy(fsiBodiesH.accRigid_fsiBodies_H.begin(), fsiBodiesH.accRigid_fsiBodies_H.end(), fsiBodiesD1.accRigid_fsiBodies_D.begin());
	thrust::copy(fsiBodiesH.q_fsiBodies_H.begin(), fsiBodiesH.q_fsiBodies_H.end(), fsiBodiesD1.q_fsiBodies_D.begin());
	thrust::copy(fsiBodiesH.omegaVelLRF_fsiBodies_H.begin(), fsiBodiesH.omegaVelLRF_fsiBodies_H.end(), fsiBodiesD1.omegaVelLRF_fsiBodies_D.begin());
	thrust::copy(fsiBodiesH.omegaAccLRF_fsiBodies_H.begin(), fsiBodiesH.omegaAccLRF_fsiBodies_H.end(), fsiBodiesD1.omegaAccLRF_fsiBodies_D.begin());

	thrust::copy(fsiBodiesD1.posRigid_fsiBodies_H.begin(), fsiBodiesD1.posRigid_fsiBodies_H.end(), fsiBodiesD2.posRigid_fsiBodies_D.begin());
	thrust::copy(fsiBodiesD1.velMassRigid_fsiBodies_H.begin(), fsiBodiesD1.velMassRigid_fsiBodies_H.end(), fsiBodiesD2.velMassRigid_fsiBodies_D.begin());
	thrust::copy(fsiBodiesD1.accRigid_fsiBodies_H.begin(), fsiBodiesD1.accRigid_fsiBodies_H.end(), fsiBodiesD2.accRigid_fsiBodies_D.begin());
	thrust::copy(fsiBodiesD1.q_fsiBodies_H.begin(), fsiBodiesD1.q_fsiBodies_H.end(), fsiBodiesD1.fsiBodiesD2.begin());
	thrust::copy(fsiBodiesD1.omegaVelLRF_fsiBodies_H.begin(), fsiBodiesD1.omegaVelLRF_fsiBodies_H.end(), fsiBodiesD2.omegaVelLRF_fsiBodies_D.begin());
	thrust::copy(fsiBodiesD1.omegaAccLRF_fsiBodies_H.begin(), fsiBodiesD1.omegaAccLRF_fsiBodies_H.end(), fsiBodiesD2.omegaAccLRF_fsiBodies_D.begin());
}

} // end namespace fsi
} // end namespace chrono

