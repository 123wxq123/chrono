#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2014 projectchrono.org
// All right reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Author: Arman Pazouki
// =============================================================================
//
// Class for fsi properties and functions.//
// =============================================================================

#include "ChFsiGeneral.h"
using namespace fsi;

uint ChFsiGeneral::iDivUp(uint a, uint b) {
	return (a % b != 0) ? (a / b + 1) : (a / b);
}

ChFsiGeneral::computeGridSize(uint n, uint blockSize, uint& numBlocks,
		uint& numThreads) {
	uint n2 = (n == 0) ? 1 : n;
	numThreads = min(blockSize, n2);
	numBlocks = iDivUp(n2, numThreads);
}

/**
 * @brief [brief description]
 * @details [long description]
 *
 * @param hostParams [description]
 * @param numObjects [description]
 */
void ChFsiGeneral::setParameters(SimParams* hostParams, NumberOfObjects* numObjects) {
	// copy parameters to constant memory
	hipMemcpyToSymbolAsync(HIP_SYMBOL(paramsD), hostParams, sizeof(SimParams));
	hipMemcpyToSymbolAsync(HIP_SYMBOL(numObjectsD), numObjects, sizeof(NumberOfObjects));
}